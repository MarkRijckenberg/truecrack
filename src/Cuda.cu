#include "hip/hip_runtime.h"
/*
 * Copyright (C)  2011  Luca Vaccaro
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "Tcdefs.h"
#include "Volumes.cuh"
#include <stdio.h>
#include <memory.h>
#include "Crypto.cuh"
#include "Core.cuh"
#include "Pkcs5.cuh"
#include "Xts.cuh"

/* Constat gpu memory data */
__device__ __constant__ unsigned char cHeaderEncrypted[TC_VOLUME_HEADER_EFFECTIVE_SIZE];
__device__ __constant__ unsigned char cSalt[SALT_LENGTH];

/* Header key size */
#define MAXPKCS5OUTSIZE 64

/* The max number of block grid; number of max parallel gpu blocks. */
int blockGridSizeMax;

/* The number of the current block grid; number of current parallel gpu blocks. */
int blockGridSizeCurrent;


/* Pointer of structures to pass to Cuda Kernel. */
unsigned char *dev_salt, *dev_blockPwd, *dev_header, *dev_headerKey;
int *dev_blockPwd_init, *dev_blockPwd_length;
short int *dev_result;
/* With Stream
#define NSTREAM 6
unsigned char *dev_salt, *dev_blockPwd[NSTREAM], *dev_header, *dev_headerKey[NSTREAM];
int *dev_blockPwd_init[NSTREAM], *dev_blockPwd_length[NSTREAM];
short int *dev_result;
*/

int getMultiprocessorCount (void){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.multiProcessorCount;
}

//#define RESIDENTTHREADS		1536
//#define NUMBLOCKS		12
#define NUMTHREADSXBLOCK	256

// Handle GPU error
static void HandleError( hipError_t err, const char *file,  int line ) {
        if (err != hipSuccess) {
                printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
                exit( EXIT_FAILURE );
        }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// GPU kernel: build word from an alphabet
__global__ void cuKernel_generate(unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, int offset, uint32_t maxsize, int charsetlength, unsigned char *charset, int wordlength){
    int number=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
    if (number>=maxsize) {blockPwd_init[number]=1;return;}
	
    blockPwd_init[number]=number*wordlength;//(number==0)?0:blockPwd_init[number-1]+wordlength;
    blockPwd_length[number]=wordlength;
    
    unsigned char *word; word= &blockPwd[number*wordlength];
    unsigned short i=0;
    for (i=0;i<wordlength;i++)
        word[i]=0;
    i=0;
    number+=offset;
    while(number>0){
        word[i]=number%charsetlength;
        number=(number-word[i])/charsetlength;
        i++;
    }
    
    for (i=0;i<wordlength;i++)
    	word[i]=charset[word[i]];
}

// GPU kernel: ripemd160 hash
__global__ void cuKernel_ripemd160 (unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	cuda_derive_key_ripemd160 (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 2000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: sha512 hash
__global__ void cuKernel_sha512 ( unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	cuda_derive_key_sha512 (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 1000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: whirlpool hash
__global__ void cuKernel_whirlpool ( unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
       int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
        if (numData>=max) return;
        cuda_derive_key_whirlpool (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 1000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: aes xts decryption
__global__ void cuKernel_aes ( unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (AES,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}
// GPU kernel: serpent xts decryption
__global__ void cuKernel_serpent ( unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (SERPENT,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}
// GPU kernel: twofish xts decryption
__global__ void cuKernel_twofish (unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (TWOFISH,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}

// Perform the bruteforce on dictionary mode
float cuda_Core_dictionary ( int encryptionAlgorithm, int bsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int keyDerivationFunction) {
	// Initialization
	int lengthpwd=0;
	for (int j=0;j<bsize;j++) {
		lengthpwd+=blockPwd_length[j];
		result[j]=0;
	}
	// Copy memory datas from host to gpu
	HANDLE_ERROR(hipMemcpy(dev_blockPwd, 		blockPwd, 		lengthpwd * sizeof(unsigned char) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_blockPwd_init, 	blockPwd_init, 	bsize * sizeof(int) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_blockPwd_length,blockPwd_length,bsize * sizeof(int) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_result, 		result,			bsize * sizeof(short int) , hipMemcpyHostToDevice));

	// Calculate the block grid and threads for block
	int numBlocks=bsize/NUMTHREADSXBLOCK+1;
	int numThreads=NUMTHREADSXBLOCK;
	if (bsize<NUMTHREADSXBLOCK)
		numThreads=bsize;

	// Start timer
	hipEvent_t tstart,tstop;
	float time;
	hipEventCreate(&tstart);
	hipEventCreate(&tstop);
	hipEventRecord(tstart, 0);

	// GPU Kernel: Key derivation function
	switch(keyDerivationFunction){
		case RIPEMD160:
			cuKernel_ripemd160 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey, bsize);
			break;
		case SHA512:
			cuKernel_sha512 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
		case WHIRLPOOL:
			cuKernel_whirlpool <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
	}
	
	// GPU Kernel: Encryption algorithms
	switch(encryptionAlgorithm){
		case AES:
			cuKernel_aes<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case SERPENT:
			cuKernel_serpent<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case TWOFISH:
			cuKernel_twofish<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
	}

	// Stop timer
	hipEventRecord(tstop, 0);
	hipEventSynchronize(tstop);
	hipEventElapsedTime(&time, tstart, tstop);

	// Copy memory result from gpu to host
	HANDLE_ERROR(hipMemcpy(result, dev_result,bsize* sizeof(short int) , hipMemcpyDeviceToHost));
	return time;
}


// Perform the bruteforce on charset mode
float cuda_Core_charset ( int encryptionAlgorithm, uint64_t bsize, uint64_t start, unsigned short int charset_length, unsigned char *charset, unsigned short int password_length, short int *result, int keyDerivationFunction)
{
	// Initialization
	int numBlocks=(int)(bsize/NUMTHREADSXBLOCK)+1;
	int numThreads=NUMTHREADSXBLOCK;
	if (bsize<NUMTHREADSXBLOCK)
		numThreads=(int)bsize;
		
	// Copy memory datas from host to gpu
	unsigned char *dev_charset = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_charset, charset_length*sizeof(unsigned char)));
	HANDLE_ERROR(hipMemcpy(dev_charset, charset, charset_length*sizeof(unsigned char), hipMemcpyHostToDevice));
	/*
	char host_blockPwd[bsize*PASSWORD_MAXSIZE];
	int host_blockPwd_init[bsize];
	int host_blockPwd_length[bsize];
	*/
	// Start timer
    hipEvent_t tstart,tstop;
    float time;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    hipEventRecord(tstart, 0); 	

	// GPU Kernel: generate passwords
	cuKernel_generate <<<numBlocks,numThreads>>>(dev_blockPwd,dev_blockPwd_init,dev_blockPwd_length,(int)start,bsize,charset_length,dev_charset,password_length);
	
	// GPU Kernel: Key derivation function
	switch(keyDerivationFunction){
		case RIPEMD160:
			cuKernel_ripemd160 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey, bsize);
			break;
		case SHA512:
			cuKernel_sha512 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
		case WHIRLPOOL:
			cuKernel_whirlpool <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
	}
	
	// GPU Kernel: Encryption algorithms
	switch(encryptionAlgorithm){
		case AES:
			cuKernel_aes<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case SERPENT:
			cuKernel_serpent<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case TWOFISH:
			cuKernel_twofish<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
	}
	
	// Stop timer
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&time, tstart, tstop);
	/*
	HANDLE_ERROR( hipMemcpy(host_blockPwd, dev_blockPwd, bsize*PASSWORD_MAXSIZE*sizeof(unsigned char), hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(host_blockPwd_init, dev_blockPwd_init, bsize*sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(host_blockPwd_length, dev_blockPwd_length, bsize*sizeof(int), hipMemcpyDeviceToHost));
	printf("host_blockPwd_init: ");
	for (int i=0;i<bsize;i++)
	  printf("%d",host_blockPwd_init[i]);
	printf("\nhost_blockPwd_length: ");
	for (int i=0;i<bsize;i++)
	  printf("%d",host_blockPwd_length[i]);
	printf("\nhost_blockPwd: ");	
	for (int i=0;i<bsize*PASSWORD_MAXSIZE;i++)
	  printf("%c",host_blockPwd[i]);
	printf("\n");
	*/
	// Copy memory result from gpu to host
	HANDLE_ERROR( hipMemcpy(result, dev_result, bsize*sizeof(short int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(dev_charset));
	return time;
}
   
// GPU memory initialization: malloc anc constant symbols
void cuda_Init (int bsize, unsigned char *salt, unsigned char *header) {
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd, 		bsize*PASSWORD_MAXSIZE* sizeof(unsigned char))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd_init,		bsize * sizeof(int))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd_length, 	bsize * sizeof(int))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_headerKey, 		MAXPKCS5OUTSIZE * bsize * sizeof(unsigned char))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_result, 		bsize * sizeof(short int)))  ;
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( cSalt), 		salt , SALT_LENGTH* sizeof(unsigned char),0,hipMemcpyHostToDevice)) ;
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( cHeaderEncrypted), 	header , TC_VOLUME_HEADER_EFFECTIVE_SIZE* sizeof(unsigned char),0,hipMemcpyHostToDevice)) ;
  
}

// GPU memory free and reset
void cuda_Free(void) {
	hipFree(dev_salt);
	hipFree(dev_blockPwd);
	hipFree(dev_blockPwd_init);
	hipFree(dev_blockPwd_length);
	hipFree(dev_result);
	hipFree(dev_headerKey);
	hipDeviceReset();
}


// LEGACY: 
/*
void cuda_Core_dictionary ( int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int keyDerivationFunction) {

	int size_block=block_currentsize;
	int size_stream=block_currentsize/NSTREAM;
	
	int numBlocks=size_stream/NUMTHREADSXBLOCK+1;
	int numThread=NUMTHREADSXBLOCK;
	if (size_stream<NUMTHREADSXBLOCK)
		numThread=size_stream;

	hipStream_t stream[NSTREAM];
	for (int i = 0; i < NSTREAM; ++i)
		hipStreamCreate(&stream[i]);
	
	int lengthpwd[NSTREAM]={0};
	for (int i=0;i<NSTREAM;i++){
	  for (int j=0;j<size_stream;j++) {
		lengthpwd[i]+=blockPwd_length[j+i*size_stream];
	  }
	}
	printf("1-%d 2-%d \n",lengthpwd[0],lengthpwd[1]);
	
	hipMalloc ( &dev_result, size_block* sizeof(short int)) ;
	hipMemcpy ( dev_result, result, size_block* sizeof(short int),hipMemcpyHostToDevice);
	
	unsigned char *host_blockPwd[NSTREAM];
	int *host_blockPwd_init[NSTREAM];
	int *host_blockPwd_length[NSTREAM];
	short int *host_result[NSTREAM];
	
	for (int i =0; i<NSTREAM; i++){
	
		hipMalloc ( (void **)&dev_blockPwd[i], 	size_stream * PASSWORD_MAXSIZE * sizeof(unsigned char)) ;
		hipMalloc ( (void **)&dev_blockPwd_init[i], 	size_stream * sizeof(int)) ;
		hipMalloc ( (void **)&dev_blockPwd_length[i], 	size_stream * sizeof(int)) ;
		hipMalloc ( (void **)&dev_headerKey[i], 	256 * size_stream * sizeof(unsigned char)) ;
	
	        hipHostAlloc(&host_blockPwd[i], 	lengthpwd[i]* sizeof(unsigned char), 	hipHostMallocDefault);
		hipHostAlloc(&host_blockPwd_init[i], 	size_stream * sizeof(int), 		hipHostMallocDefault);
		hipHostAlloc(&host_blockPwd_length[i], size_stream * sizeof(int),	 	hipHostMallocDefault);
		hipHostAlloc(&host_result[i], 		size_stream * sizeof(int),	 	hipHostMallocDefault);
	
		memcpy(host_blockPwd[i], 	blockPwd+((i==0)?0:lengthpwd[i-1]),	lengthpwd[i]*sizeof(unsigned char));
		memcpy(host_blockPwd_init[i], 	blockPwd_init+i*size_stream, 		size_stream*sizeof(int));
		memcpy(host_blockPwd_length[i], blockPwd_length+i*size_stream, 		size_stream*sizeof(int));
			
	}
	
	for (int i = 0; i < NSTREAM; i++){
	  
		hipMemcpyAsync(dev_blockPwd[i], 	host_blockPwd[i],		lengthpwd[i] * sizeof(unsigned char) , hipMemcpyHostToDevice, stream[i]) ;
		hipMemcpyAsync(dev_blockPwd_init[i], 	host_blockPwd_init[i], 		size_stream * sizeof(int) , hipMemcpyHostToDevice,stream[i]);
		hipMemcpyAsync(dev_blockPwd_length[i],	host_blockPwd_length[i], 	size_stream * sizeof(int) , hipMemcpyHostToDevice,stream[i]) ;
		hipMemcpyAsync(dev_result, 		host_result[0], 		size_stream * sizeof(short int) , hipMemcpyHostToDevice,stream[0]) ;
		
		
		cuda_Kernel_ripemd160<<<numBlocks,numThread, 0, stream[i]>>>(dev_blockPwd[i], dev_blockPwd_init[i], dev_blockPwd_length[i], dev_headerKey[i], size_stream);
		cuda_Kernel_aes<<<numBlocks,numThread, 0, stream[i]>>>(dev_headerKey[i], dev_result, size_stream);
			
		hipError_t err=hipMemcpy(result+i*size_stream, dev_result,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost) ;
	//	hipError_t err=hipMemcpyAsync(host_result[0], dev_result,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost,stream[0]) ;
		if (err!=hipSuccess){
			printf("->%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
		}printf("ok %d\n",i);
		hipDeviceSynchronize();
//	memcpy(result, 	host_result[0], 		size_stream*sizeof(int));
	
	
		
		//cuda_Kernel_ripemd160<<<numBlocks,numThread, 0, stream[i]>>>(dev_blockPwd+ i * size_stream, dev_blockPwd_init+ i * size_stream, dev_blockPwd_length+ i * size_stream, dev_headerKey, size_stream);
		//cuda_Kernel_aes<<<numBlocks,numThread, 0, stream[i]>>>(dev_headerKey, dev_result+ i * size_stream, size_stream);
		
		
		//hipError_t err=hipMemcpyAsync(host_result+i*size_stream, 	dev_result+i*size_stream,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost, stream[i]) ;
		
	}
	
	for (int i = 0; i < NSTREAM; i++)
		hipStreamDestroy(stream[i]);
    
    
	hipFree(dev_result);
}
*/

/*
 __global__ void cuda_Kernel_charset (
    	unsigned char *salt,
    	unsigned char *headerEncrypted,
    	unsigned short int charset_length,
    	unsigned char *charset,
    	unsigned short int password_length,
    	uint64_t maxcombination,
    	 short int *result, 
	 int keyDerivationFunction)
 {
	uint64_t numData = blockIdx.x*blockDim.x+threadIdx.x;
	__align__(8) unsigned char headerkey[192];
	__align__(8) unsigned char headerDecrypted[512];
	__align__(8) unsigned char pwd[8];

	//__device__ void computePwd (int number, int maxcombination, int charsetlength, unsigned char *charset, int wordlength, unsigned char *word){
	computePwd(numData,maxcombination,charset_length,charset,password_length,pwd);
	pwd[password_length]='\0';
	
	//__device__ void cuda_Pbkdf2_charset_ ( unsigned char *salt, unsigned char *pwd, int pwd_len, unsigned char *headerkey) {
//	cuda_Pbkdf2 ( salt, pwd, password_length, headerkey);

	int value=cuda_Xts (headerEncrypted, headerkey, headerDecrypted);
	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}*/

/*	
__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int max, int keyDerivationFunction) {
	int value;
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;

	if (numData>=max) return;

	// Array of unsigned char in the shared memory
	__align__(8) unsigned char headerKey[192];
	__align__(8) unsigned char headerDecrypted[512];

	// Calculate the hash header key
	unsigned char *pwd=blockPwd+blockPwd_init[numData];
	int pwd_len = blockPwd_length[numData];


	if(keyDerivationFunction==RIPEMD160)
		cuda_Pbkdf2 ( salt, pwd, pwd_len, headerKey);
	else if(keyDerivationFunction==SHA512)
		cuda_derive_key_sha512 (  pwd, pwd_len, salt, PKCS5_SALT_SIZE, 1000, headerKey, 64);
	else if(keyDerivationFunction==WHIRLPOOL)
		cuda_derive_key_whirlpool (  pwd, pwd_len, salt, PKCS5_SALT_SIZE, 1000, headerKey, 64);
	else
		;
	
	// Decrypt the header and compare the key
	value=cuda_Xts (headerEncrypted, headerKey,headerDecrypted);

	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}
*/
/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
#include "Pkcs5.cuh"


/*
__device__ void cuda_Pbkdf2 ( unsigned char *salt, unsigned char *pwd, int pwd_len, unsigned char *headerkey) {
	SupportPkcs5 support;
	SupportPkcs5 *sup;
	sup = &support;
	int numBlock=0;
	int c, i;

	for(numBlock=0;numBlock<10;numBlock++){
		//  cuda_Pbkdf2 (salt, blockPwd, blockPwd_init, blockPwd_length, headerkey, numData, i);
		
		//INCLUDE: void derive_u_ripemd160 (char *pwd, int pwd_len, char *salt, int salt_len, int iterations, char *u, int b)		
		int b=numBlock;
		unsigned char *u=headerkey+RIPEMD160_DIGESTSIZE*b;

		// iteration 1 
		memset (sup->ccounter, 0, 4);
		sup->ccounter[3] = (char) b+1;
		memcpy (sup->cinit, salt, SALT_LENGTH);	// salt 
		memcpy (&sup->cinit[SALT_LENGTH],sup->ccounter, 4);	// big-endian block number 
		
		cuda_hmac_ripemd160 (pwd, pwd_len, sup->cinit, SALT_LENGTH + 4, sup->cj, sup);
		memcpy (u, sup->cj, RIPEMD160_DIGESTSIZE);
		
		//remaining iterations 
		for (c = 1; c < ITERATIONS; c++)
		{
			cuda_hmac_ripemd160 (pwd, pwd_len, sup->cj, RIPEMD160_DIGESTSIZE, sup->ck,sup);
			for (i = 0; i < RIPEMD160_DIGESTSIZE; i++)
			{
				u[i] ^= sup->ck[i];
				sup->cj[i] = sup->ck[i];
			}
		}
	}
}*/

__device__ void cuda_hmac_ripemd160 (unsigned char *key, int keylen, unsigned char *input, int len, unsigned char *digest)
{
    SupportPkcs5 support;
	SupportPkcs5 *sup=&support;
    int i;
    // If the key is longer than the hash algorithm block size,
    //	   let key = ripemd160(key), as per HMAC specifications.
    if (keylen > RIPEMD160_BLOCKSIZE)
	{
		//RMD160Init(&tctx);
        //RMD160Update(&tctx, (const unsigned char *) key, keylen);
        //RMD160Final(tk, &tctx);
		cuda_RMD160(&sup->ctctx,(unsigned char *) key, keylen,(unsigned char *)NULL,0,sup->ctk);
        key = (unsigned char *) sup->ctk;
        keylen = RIPEMD160_DIGESTSIZE;
		//burn (&ctctx, sizeof(ctctx));	// Prevent leaks
    }
	 /*
	 RMD160(K XOR opad, RMD160(K XOR ipad, text))
	 where K is an n byte key
	 ipad is the byte 0x36 repeated RIPEMD160_BLOCKSIZE times
	 opad is the byte 0x5c repeated RIPEMD160_BLOCKSIZE times
	 and text is the data being protected*/
	 // start out by storing key in pads
	 // XOR key with ipad and opad values
	 for (i=0; i<sizeof(sup->cpad); i++)
		 sup->cpad[i]=0x36;
	 for (i=0; i<keylen; i++)
		 sup->cpad[i] ^= key[i];
	 
	 cuda_RMD160(&sup->ccontext,sup->cpad,RIPEMD160_BLOCKSIZE,(const unsigned char *) input, len, (unsigned char *) digest);
	 
	 for (i=0; i<sizeof(sup->cpad); i++)
		 sup->cpad[i]=0x5c;
	 for (i=0; i<keylen; i++)
		 sup->cpad[i] ^= key[i];
	 cuda_RMD160(&sup->ccontext,sup->cpad,RIPEMD160_BLOCKSIZE,(const unsigned char *) digest, RIPEMD160_DIGESTSIZE, (unsigned char *) digest);
	 
}
/*
__device__ void cuda_hmac_ripemd160 (unsigned char *key, int keylen, unsigned char *input, int len, unsigned char *digest)
{
    RMD160_CTX context;
    unsigned char k_ipad[65];  //inner padding - key XORd with ipad 
    unsigned char k_opad[65];  //outer padding - key XORd with opad
    unsigned char tk[RIPEMD160_DIGESTSIZE];
    int i;
	
    // If the key is longer than the hash algorithm block size, let key = ripemd160(key), as per HMAC specifications. 
    if (keylen > RIPEMD160_BLOCKSIZE)
	{
        RMD160_CTX      tctx;
		
        RMD160Init(&tctx);
        RMD160Update(&tctx, (const unsigned char *) key, keylen);
        RMD160Final(tk, &tctx);
		
        key = ( unsigned char *) tk;
        keylen = RIPEMD160_DIGESTSIZE;
		
		burn (&tctx, sizeof(tctx));	// Prevent leaks
    }
	
	/*
	 
	 RMD160(K XOR opad, RMD160(K XOR ipad, text))
	 
	 where K is an n byte key
	 ipad is the byte 0x36 repeated RIPEMD160_BLOCKSIZE times
	 opad is the byte 0x5c repeated RIPEMD160_BLOCKSIZE times
	 and text is the data being protected 
	
	
	// start out by storing key in pads
	memset(k_ipad, 0x36, sizeof(k_ipad));
    memset(k_opad, 0x5c, sizeof(k_opad));
	
    // XOR key with ipad and opad values
    for (i=0; i<keylen; i++)
	{
        k_ipad[i] ^= key[i];
        k_opad[i] ^= key[i];
    }
	
    //perform inner RIPEMD-160
	
    RMD160Init(&context);           // init context for 1st pass
    RMD160Update(&context, k_ipad, RIPEMD160_BLOCKSIZE);  // start with inner pad
    RMD160Update(&context, (const unsigned char *) input, len); // then text of datagram
    RMD160Final((unsigned char *) digest, &context);         // finish up 1st pass
	
    // perform outer RIPEMD-160
    RMD160Init(&context);           // init context for 2nd pass
    RMD160Update(&context, k_opad, RIPEMD160_BLOCKSIZE);  // start with outer pad 
    // results of 1st hash
    RMD160Update(&context, (const unsigned char *) digest, RIPEMD160_DIGESTSIZE);
    RMD160Final((unsigned char *) digest, &context);         // finish up 2nd pass
	
	// Prevent possible leaks. 
    burn (k_ipad, sizeof(k_ipad));
    burn (k_opad, sizeof(k_opad));
	burn (tk, sizeof(tk));
	burn (&context, sizeof(context));
}
*/

__device__ void cuda_derive_u_ripemd160 (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *u, int b)
{
	unsigned char j[RIPEMD160_DIGESTSIZE], k[RIPEMD160_DIGESTSIZE];
	unsigned char init[128];
	unsigned char counter[4];
	int c, i;
	
	/* iteration 1 */
	memset (counter, 0, 4);
	counter[3] = (char) b;
	memcpy (init, salt, salt_len);	/* salt */
	memcpy (&init[salt_len], counter, 4);	/* big-endian block number */
	cuda_hmac_ripemd160 (pwd, pwd_len, init, salt_len + 4, j);
	memcpy (u, j, RIPEMD160_DIGESTSIZE);
	
	/* remaining iterations */
	for (c = 1; c < iterations; c++)
	{
		cuda_hmac_ripemd160 (pwd, pwd_len, j, RIPEMD160_DIGESTSIZE, k);
		for (i = 0; i < RIPEMD160_DIGESTSIZE; i++)
		{
			u[i] ^= k[i];
			j[i] = k[i];
		}
	}
	
	/* Prevent possible leaks. */
	burn (j, sizeof(j));
	burn (k, sizeof(k));
}


__device__ void cuda_derive_key_ripemd160 (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *dk, int dklen)
{
	unsigned char u[RIPEMD160_DIGESTSIZE];
	int b, l, r;
	
	if (dklen % RIPEMD160_DIGESTSIZE)
	{
		l = 1 + dklen / RIPEMD160_DIGESTSIZE;
	}
	else
	{
		l = dklen / RIPEMD160_DIGESTSIZE;
	}
	
	r = dklen - (l - 1) * RIPEMD160_DIGESTSIZE;
	
	// first l - 1 blocks 
	for (b = 1; b < l; b++)
	{
		cuda_derive_u_ripemd160 (pwd, pwd_len, salt, salt_len, iterations, u, b);
		memcpy (dk, u, RIPEMD160_DIGESTSIZE);
		dk += RIPEMD160_DIGESTSIZE;
	}
	
	// last block
	cuda_derive_u_ripemd160 (pwd, pwd_len, salt, salt_len, iterations, u, b);
	memcpy (dk, u, r);
	
	// Prevent possible leaks. 
	burn (u, sizeof(u));
	
}













__device__ void cuda_hmac_truncate
  (
	  unsigned char *d1,		/* data to be truncated */
	  unsigned char *d2,		/* truncated data */
	  int len		/* length in bytes to keep */
)
{
	int i;
	for (i = 0; i < len; i++)
		d2[i] = d1[i];
}






__device__ void cuda_hmac_sha512
(
	  unsigned char *k,		/* secret key */
	  int lk,		/* length of the key in bytes */
	  unsigned char *d,		/* data */
	  int ld,		/* length of data in bytes */
	  unsigned char *out,		/* output buffer, at least "t" bytes */
	  int t
)
{
	sha512_ctx ictx, octx;
	unsigned char isha[SHA512_DIGESTSIZE], osha[SHA512_DIGESTSIZE];
	unsigned char key[SHA512_DIGESTSIZE];
	unsigned char buf[SHA512_BLOCKSIZE];
	int i;

    /* If the key is longer than the hash algorithm block size,
	   let key = sha512(key), as per HMAC specifications. */
	if (lk > SHA512_BLOCKSIZE)
	{
		sha512_ctx tctx;

		sha512_begin (&tctx);
		sha512_hash ((unsigned char *) k, lk, &tctx);
		sha512_end ((unsigned char *) key, &tctx);

		k = key;
		lk = SHA512_DIGESTSIZE;

		burn (&tctx, sizeof(tctx));		// Prevent leaks
	}

	/**** Inner Digest ****/

	sha512_begin (&ictx);

	/* Pad the key for inner digest */
	for (i = 0; i < lk; ++i)
		buf[i] = (unsigned char) (k[i] ^ 0x36);
	for (i = lk; i < SHA512_BLOCKSIZE; ++i)
		buf[i] = (unsigned char) 0x36;

	sha512_hash ((unsigned char *) buf, SHA512_BLOCKSIZE, &ictx);
	sha512_hash ((unsigned char *) d, ld, &ictx);

	sha512_end ((unsigned char *) isha, &ictx);

	/**** Outer Digest ****/

	sha512_begin (&octx);

	for (i = 0; i < lk; ++i)
		buf[i] = (unsigned char) (k[i] ^ 0x5C);
	for (i = lk; i < SHA512_BLOCKSIZE; ++i)
		buf[i] = (unsigned char) 0x5C;

	sha512_hash ((unsigned char *) buf, SHA512_BLOCKSIZE, &octx);
	sha512_hash ((unsigned char *) isha, SHA512_DIGESTSIZE, &octx);

	sha512_end ((unsigned char *) osha, &octx);

	/* truncate and print the results */
	t = t > SHA512_DIGESTSIZE ? SHA512_DIGESTSIZE : t;
	cuda_hmac_truncate (osha, out, t);

	/* Prevent leaks */
	burn (&ictx, sizeof(ictx));
	burn (&octx, sizeof(octx));
	burn (isha, sizeof(isha));
	burn (osha, sizeof(osha));
	burn (buf, sizeof(buf));
	burn (key, sizeof(key));
}


__device__ void cuda_derive_u_sha512 (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *u, int b)
{
	unsigned char j[SHA512_DIGESTSIZE], k[SHA512_DIGESTSIZE];
	unsigned char init[128];
	unsigned char counter[4];
	int c, i;

	/* iteration 1 */
	memset (counter, 0, 4);
	counter[3] = (char) b;
	memcpy (init, salt, salt_len);	/* salt */
	memcpy (&init[salt_len], counter, 4);	/* big-endian block number */
	cuda_hmac_sha512 (pwd, pwd_len, init, salt_len + 4, j, SHA512_DIGESTSIZE);
	memcpy (u, j, SHA512_DIGESTSIZE);

	/* remaining iterations */
	for (c = 1; c < iterations; c++)
	{
		cuda_hmac_sha512 (pwd, pwd_len, j, SHA512_DIGESTSIZE, k, SHA512_DIGESTSIZE);
		for (i = 0; i < SHA512_DIGESTSIZE; i++)
		{
			u[i] ^= k[i];
			j[i] = k[i];
		}
	}

	/* Prevent possible leaks. */
	burn (j, sizeof(j));
	burn (k, sizeof(k));
}


__device__ void cuda_derive_key_sha512 (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *dk, int dklen)
{
	unsigned char u[SHA512_DIGESTSIZE];
	int b, l, r;

	if (dklen % SHA512_DIGESTSIZE)
	{
		l = 1 + dklen / SHA512_DIGESTSIZE;
	}
	else
	{
		l = dklen / SHA512_DIGESTSIZE;
	}

	r = dklen - (l - 1) * SHA512_DIGESTSIZE;

	/* first l - 1 blocks */
	for (b = 1; b < l; b++)
	{
		cuda_derive_u_sha512 (pwd, pwd_len, salt, salt_len, iterations, u, b);
		memcpy (dk, u, SHA512_DIGESTSIZE);
		dk += SHA512_DIGESTSIZE;
	}

	/* last block */
	cuda_derive_u_sha512 (pwd, pwd_len, salt, salt_len, iterations, u, b);
	memcpy (dk, u, r);


	/* Prevent possible leaks. */
	burn (u, sizeof(u));
}









__device__ void cuda_hmac_whirlpool
(
	  unsigned char *k,		/* secret key */
	  int lk,		/* length of the key in bytes */
	  unsigned char *d,		/* data */
	  int ld,		/* length of data in bytes */
	  unsigned char *out,	/* output buffer, at least "t" bytes */
	  int t
)
{
	WHIRLPOOL_CTX ictx, octx;
	unsigned char iwhi[WHIRLPOOL_DIGESTSIZE], owhi[WHIRLPOOL_DIGESTSIZE];
	unsigned char key[WHIRLPOOL_DIGESTSIZE];
	unsigned char buf[WHIRLPOOL_BLOCKSIZE];
	int i;

    /* If the key is longer than the hash algorithm block size,
	   let key = whirlpool(key), as per HMAC specifications. */
	if (lk > WHIRLPOOL_BLOCKSIZE)
	{
		WHIRLPOOL_CTX tctx;

		WHIRLPOOL_init (&tctx);
		WHIRLPOOL_add ((unsigned char *) k, lk * 8, &tctx);
		WHIRLPOOL_finalize (&tctx, (unsigned char *) key);

		k = key;
		lk = WHIRLPOOL_DIGESTSIZE;

		burn (&tctx, sizeof(tctx));		// Prevent leaks
	}

	/**** Inner Digest ****/

	WHIRLPOOL_init (&ictx);

	/* Pad the key for inner digest */
	for (i = 0; i < lk; ++i)
		buf[i] = (unsigned char) (k[i] ^ 0x36);
	for (i = lk; i < WHIRLPOOL_BLOCKSIZE; ++i)
		buf[i] = (unsigned char) 0x36;

	WHIRLPOOL_add ((unsigned char *) buf, WHIRLPOOL_BLOCKSIZE * 8, &ictx);
	WHIRLPOOL_add ((unsigned char *) d, ld * 8, &ictx);

	WHIRLPOOL_finalize (&ictx, (unsigned char *) iwhi);

	/**** Outer Digest ****/

	WHIRLPOOL_init (&octx);

	for (i = 0; i < lk; ++i)
		buf[i] = (unsigned char) (k[i] ^ 0x5C);
	for (i = lk; i < WHIRLPOOL_BLOCKSIZE; ++i)
		buf[i] = (unsigned char) 0x5C;

	WHIRLPOOL_add ((unsigned char *) buf, WHIRLPOOL_BLOCKSIZE * 8, &octx);
	WHIRLPOOL_add ((unsigned char *) iwhi, WHIRLPOOL_DIGESTSIZE * 8, &octx);

	WHIRLPOOL_finalize (&octx, (unsigned char *) owhi);

	/* truncate and print the results */
	t = t > WHIRLPOOL_DIGESTSIZE ? WHIRLPOOL_DIGESTSIZE : t;
	cuda_hmac_truncate (owhi, out, t);

	/* Prevent possible leaks. */
	burn (&ictx, sizeof(ictx));
	burn (&octx, sizeof(octx));
	burn (owhi, sizeof(owhi));
	burn (iwhi, sizeof(iwhi));
	burn (buf, sizeof(buf));
	burn (key, sizeof(key));
}

__device__ void cuda_derive_u_whirlpool (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *u, int b)
{
	unsigned char j[WHIRLPOOL_DIGESTSIZE], k[WHIRLPOOL_DIGESTSIZE];
	unsigned char init[128];
	unsigned char counter[4];
	int c, i;

	/* iteration 1 */
	memset (counter, 0, 4);
	counter[3] = (char) b;
	memcpy (init, salt, salt_len);	/* salt */
	memcpy (&init[salt_len], counter, 4);	/* big-endian block number */
	cuda_hmac_whirlpool (pwd, pwd_len, init, salt_len + 4, j, WHIRLPOOL_DIGESTSIZE);
	memcpy (u, j, WHIRLPOOL_DIGESTSIZE);

	/* remaining iterations */
	for (c = 1; c < iterations; c++)
	{
		cuda_hmac_whirlpool (pwd, pwd_len, j, WHIRLPOOL_DIGESTSIZE, k, WHIRLPOOL_DIGESTSIZE);
		for (i = 0; i < WHIRLPOOL_DIGESTSIZE; i++)
		{
			u[i] ^= k[i];
			j[i] = k[i];
		}
	}

	/* Prevent possible leaks. */
	burn (j, sizeof(j));
	burn (k, sizeof(k));
}

__device__ void cuda_derive_key_whirlpool (unsigned char *pwd, int pwd_len, unsigned char *salt, int salt_len, int iterations, unsigned char *dk, int dklen)
{
	unsigned char u[WHIRLPOOL_DIGESTSIZE];
	int b, l, r;

	if (dklen % WHIRLPOOL_DIGESTSIZE)
	{
		l = 1 + dklen / WHIRLPOOL_DIGESTSIZE;
	}
	else
	{
		l = dklen / WHIRLPOOL_DIGESTSIZE;
	}

	r = dklen - (l - 1) * WHIRLPOOL_DIGESTSIZE;

	/* first l - 1 blocks */
	for (b = 1; b < l; b++)
	{
		cuda_derive_u_whirlpool (pwd, pwd_len, salt, salt_len, iterations, u, b);
		memcpy (dk, u, WHIRLPOOL_DIGESTSIZE);
		dk += WHIRLPOOL_DIGESTSIZE;
	}

	/* last block */
	cuda_derive_u_whirlpool (pwd, pwd_len, salt, salt_len, iterations, u, b);
	memcpy (dk, u, r);


	/* Prevent possible leaks. */
	burn (u, sizeof(u));
}/*
Collection of source files for AES encryption algorithm
- Aescrypt.c
- Aeskey.c
- Aestab.c
*/

//#include "Aes.h"


#include "Aestab.cu"
#include "Aeskey.cu"
#include "Aescrypt.cu"
// serpent.cpp - written and placed in the public domain by Wei Dai

/* Adapted for TrueCrypt */

#ifdef TC_WINDOWS_BOOT
#pragma optimize ("t", on)
#endif

#include "Serpent.cuh"
#include "Common/Endian.h"

#include <memory.h>

#if defined(_WIN32) && !defined(_DEBUG)
#include <stdlib.h>
#define rotlFixed _rotl
#define rotrFixed _rotr
#else
#define rotlFixed(x,n)   (((x) << (n)) | ((x) >> (32 - (n))))
#define rotrFixed(x,n)   (((x) >> (n)) | ((x) << (32 - (n))))
#endif



#define TC_MINIMIZE_CODE_SIZE


// linear transformation
#define LT(i,a,b,c,d,e)	{\
	a = rotlFixed(a, 13);	\
	c = rotlFixed(c, 3); 	\
	d = rotlFixed(d ^ c ^ (a << 3), 7); 	\
	b = rotlFixed(b ^ a ^ c, 1); 	\
	a = rotlFixed(a ^ b ^ d, 5); 		\
	c = rotlFixed(c ^ d ^ (b << 7), 22);}

// inverse linear transformation
#define ILT(i,a,b,c,d,e)	{\
	c = rotrFixed(c, 22);	\
	a = rotrFixed(a, 5); 	\
	c ^= d ^ (b << 7);	\
	a ^= b ^ d; 		\
	b = rotrFixed(b, 1); 	\
	d = rotrFixed(d, 7) ^ c ^ (a << 3);	\
	b ^= a ^ c; 		\
	c = rotrFixed(c, 3); 	\
	a = rotrFixed(a, 13);}

// order of output from S-box functions
#define beforeS0(f) f(0,a,b,c,d,e)
#define afterS0(f) f(1,b,e,c,a,d)
#define afterS1(f) f(2,c,b,a,e,d)
#define afterS2(f) f(3,a,e,b,d,c)
#define afterS3(f) f(4,e,b,d,c,a)
#define afterS4(f) f(5,b,a,e,c,d)
#define afterS5(f) f(6,a,c,b,e,d)
#define afterS6(f) f(7,a,c,d,b,e)
#define afterS7(f) f(8,d,e,b,a,c)

// order of output from inverse S-box functions
#define beforeI7(f) f(8,a,b,c,d,e)
#define afterI7(f) f(7,d,a,b,e,c)
#define afterI6(f) f(6,a,b,c,e,d)
#define afterI5(f) f(5,b,d,e,c,a)
#define afterI4(f) f(4,b,c,e,a,d)
#define afterI3(f) f(3,a,b,e,c,d)
#define afterI2(f) f(2,b,d,e,c,a)
#define afterI1(f) f(1,a,b,c,e,d)
#define afterI0(f) f(0,a,d,b,e,c)

// The instruction sequences for the S-box functions 
// come from Dag Arne Osvik's paper "Speeding up Serpent".

#define S0(i, r0, r1, r2, r3, r4) \
       {           \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r4 ^= r2;   \
    r1 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r4;   \
    r4 ^= r3;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 ^= r4;   \
    r4 = ~r4;      \
    r4 |= r1;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r3 |= r0;   \
    r1 ^= r3;   \
    r4 ^= r3;   \
            }

#define I0(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r1;   \
    r1 |= r0;   \
    r4 = ~r4;      \
    r1 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r4 ^= r0;   \
    r0 |= r1;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r2 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r1;   \
    r2 &= r3;   \
    r4 ^= r2;   \
            }

#define S1(i, r0, r1, r2, r3, r4) \
       {           \
    r0 = ~r0;      \
    r2 = ~r2;      \
    r4 = r0;   \
    r0 &= r1;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r3 ^= r2;   \
    r1 ^= r0;   \
    r0 ^= r4;   \
    r4 |= r1;   \
    r1 ^= r3;   \
    r2 |= r0;   \
    r2 &= r4;   \
    r0 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r2;   \
    r0 ^= r4;   \
            }

#define I1(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r1;   \
    r1 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r3 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r3;   \
    r0 ^= r4;   \
    r0 |= r2;   \
    r1 ^= r3;   \
    r0 ^= r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
    r4 = ~r4;      \
    r4 ^= r1;   \
    r1 |= r0;   \
    r1 ^= r0;   \
    r1 |= r4;   \
    r3 ^= r1;   \
            }

#define S2(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 &= r2;   \
    r0 ^= r3;   \
    r2 ^= r1;   \
    r2 ^= r0;   \
    r3 |= r4;   \
    r3 ^= r1;   \
    r4 ^= r2;   \
    r1 = r3;   \
    r3 |= r4;   \
    r3 ^= r0;   \
    r0 &= r1;   \
    r4 ^= r0;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r4 = ~r4;      \
            }

#define I2(i, r0, r1, r2, r3, r4) \
       {           \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r3;   \
    r3 &= r2;   \
    r3 ^= r1;   \
    r1 |= r2;   \
    r1 ^= r4;   \
    r4 &= r3;   \
    r2 ^= r3;   \
    r4 &= r0;   \
    r4 ^= r2;   \
    r2 &= r1;   \
    r2 |= r0;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r0 ^= r3;   \
    r0 &= r1;   \
    r3 ^= r4;   \
    r3 ^= r0;   \
            }

#define S3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 |= r3;   \
    r3 ^= r1;   \
    r1 &= r4;   \
    r4 ^= r2;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r4 |= r1;   \
    r3 ^= r4;   \
    r0 ^= r1;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r1 |= r0;   \
    r1 ^= r2;   \
    r0 ^= r3;   \
    r2 = r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
            }

#define I3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r4;   \
    r4 ^= r3;   \
    r3 |= r1;   \
    r3 ^= r2;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r1;   \
    r4 ^= r2;   \
    r2 &= r3;   \
    r1 |= r3;   \
    r1 ^= r2;   \
    r4 ^= r0;   \
    r2 ^= r4;   \
            }

#define S4(i, r0, r1, r2, r3, r4) \
       {           \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r1 ^= r2;   \
    r4 ^= r3;   \
    r0 ^= r4;   \
    r2 &= r4;   \
    r2 ^= r0;   \
    r0 &= r1;   \
    r3 ^= r0;   \
    r4 |= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r2 &= r3;   \
    r0 = ~r0;      \
    r4 ^= r2;   \
            }

#define I4(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r3;   \
    r2 ^= r1;   \
    r1 |= r3;   \
    r1 &= r0;   \
    r4 ^= r2;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r0 = ~r0;      \
    r3 ^= r4;   \
    r1 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r2;   \
    r0 ^= r1;   \
    r2 &= r0;   \
    r3 ^= r0;   \
    r2 ^= r4;   \
    r2 |= r3;   \
    r3 ^= r0;   \
    r2 ^= r1;   \
            }

#define S5(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r1;   \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r4 = r1;   \
    r1 &= r0;   \
    r2 ^= r3;   \
    r1 ^= r2;   \
    r2 |= r4;   \
    r4 ^= r3;   \
    r3 &= r1;   \
    r3 ^= r0;   \
    r4 ^= r1;   \
    r4 ^= r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r0 ^= r4;   \
    r4 |= r3;   \
    r2 ^= r4;   \
            }

#define I5(i, r0, r1, r2, r3, r4) \
       {           \
    r1 = ~r1;      \
    r4 = r3;   \
    r2 ^= r1;   \
    r3 |= r0;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 ^= r4;   \
    r4 |= r0;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r3 &= r4;   \
    r4 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r4;   \
    r4 = ~r4;      \
            }

#define S6(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r3;   \
    r3 &= r0;   \
    r0 ^= r4;   \
    r3 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r2 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r4 ^= r3;   \
    r4 ^= r0;   \
    r3 = ~r3;      \
    r2 &= r4;   \
    r2 ^= r3;   \
            }

#define I6(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r2;   \
    r4 = r2;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 = ~r2;      \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r4 |= r0;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r4 ^= r1;   \
    r1 &= r3;   \
    r1 ^= r0;   \
    r0 ^= r3;   \
    r0 |= r2;   \
    r3 ^= r1;   \
    r4 ^= r0;   \
            }

#define S7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r1;   \
    r2 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r2 ^= r1;   \
    r1 ^= r0;   \
    r0 |= r4;   \
    r0 ^= r2;   \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r4;   \
    r4 ^= r2;   \
    r2 &= r0;   \
    r4 = ~r4;      \
    r2 ^= r4;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r1;   \
            }

#define I7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r4 |= r3;   \
    r3 ^= r1;   \
    r1 |= r0;   \
    r0 ^= r2;   \
    r2 &= r4;   \
    r1 ^= r2;   \
    r2 ^= r0;   \
    r0 |= r2;   \
    r3 &= r4;   \
    r0 ^= r3;   \
    r4 ^= r1;   \
    r3 ^= r4;   \
    r4 |= r0;   \
    r3 ^= r2;   \
    r4 ^= r2;   \
            }

// key xor
#define KX(r, a, b, c, d, e)	{\
	a ^= k[4 * r + 0]; \
	b ^= k[4 * r + 1]; \
	c ^= k[4 * r + 2]; \
	d ^= k[4 * r + 3];}


#ifdef TC_MINIMIZE_CODE_SIZE

__device__  void S0f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{
	*r3 ^= *r0;
	*r4 = *r1;
	*r1 &= *r3;
	*r4 ^= *r2;
	*r1 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r4;
	*r4 ^= *r3;
	*r3 ^= *r2;
	*r2 |= *r1;
	*r2 ^= *r4;
	*r4 = ~*r4;
	*r4 |= *r1;
	*r1 ^= *r3;
	*r1 ^= *r4;
	*r3 |= *r0;
	*r1 ^= *r3;
	*r4 ^= *r3;
}

__device__  void S1f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
    *r0 = ~*r0;   
    *r2 = ~*r2;   
    *r4 = *r0;
    *r0 &= *r1;
    *r2 ^= *r0;
    *r0 |= *r3;
    *r3 ^= *r2;
    *r1 ^= *r0;
    *r0 ^= *r4;
    *r4 |= *r1;
    *r1 ^= *r3;
    *r2 |= *r0;
    *r2 &= *r4;
    *r0 ^= *r1;
    *r1 &= *r2;
    *r1 ^= *r0;
    *r0 &= *r2;
    *r0 ^= *r4;
}

__device__  void S2f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r0;
	*r0 &= *r2;
	*r0 ^= *r3;
	*r2 ^= *r1;
	*r2 ^= *r0;
	*r3 |= *r4;
	*r3 ^= *r1;
	*r4 ^= *r2;
	*r1 = *r3;
	*r3 |= *r4;
	*r3 ^= *r0;
	*r0 &= *r1;
	*r4 ^= *r0;
	*r1 ^= *r3;
	*r1 ^= *r4;
	*r4 = ~*r4;   
}

__device__  void S3f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r0;
	*r0 |= *r3;
	*r3 ^= *r1;
	*r1 &= *r4;
	*r4 ^= *r2;
	*r2 ^= *r3;
	*r3 &= *r0;
	*r4 |= *r1;
	*r3 ^= *r4;
	*r0 ^= *r1;
	*r4 &= *r0;
	*r1 ^= *r3;
	*r4 ^= *r2;
	*r1 |= *r0;
	*r1 ^= *r2;
	*r0 ^= *r3;
	*r2 = *r1;
	*r1 |= *r3;
	*r1 ^= *r0;
}

__device__  void S4f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r1 ^= *r3;
	*r3 = ~*r3;   
	*r2 ^= *r3;
	*r3 ^= *r0;
	*r4 = *r1;
	*r1 &= *r3;
	*r1 ^= *r2;
	*r4 ^= *r3;
	*r0 ^= *r4;
	*r2 &= *r4;
	*r2 ^= *r0;
	*r0 &= *r1;
	*r3 ^= *r0;
	*r4 |= *r1;
	*r4 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r2;
	*r2 &= *r3;
	*r0 = ~*r0;   
	*r4 ^= *r2;
}

__device__  void S5f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r0 ^= *r1;
	*r1 ^= *r3;
	*r3 = ~*r3;   
	*r4 = *r1;
	*r1 &= *r0;
	*r2 ^= *r3;
	*r1 ^= *r2;
	*r2 |= *r4;
	*r4 ^= *r3;
	*r3 &= *r1;
	*r3 ^= *r0;
	*r4 ^= *r1;
	*r4 ^= *r2;
	*r2 ^= *r0;
	*r0 &= *r3;
	*r2 = ~*r2;   
	*r0 ^= *r4;
	*r4 |= *r3;
	*r2 ^= *r4;
}

__device__  void S6f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r2 = ~*r2;   
	*r4 = *r3;
	*r3 &= *r0;
	*r0 ^= *r4;
	*r3 ^= *r2;
	*r2 |= *r4;
	*r1 ^= *r3;
	*r2 ^= *r0;
	*r0 |= *r1;
	*r2 ^= *r1;
	*r4 ^= *r0;
	*r0 |= *r3;
	*r0 ^= *r2;
	*r4 ^= *r3;
	*r4 ^= *r0;
	*r3 = ~*r3;   
	*r2 &= *r4;
	*r2 ^= *r3;
}

__device__  void S7f (unsigned __int32 *r0, unsigned __int32 *r1, unsigned __int32 *r2, unsigned __int32 *r3, unsigned __int32 *r4)
{        
	*r4 = *r2;
	*r2 &= *r1;
	*r2 ^= *r3;
	*r3 &= *r1;
	*r4 ^= *r2;
	*r2 ^= *r1;
	*r1 ^= *r0;
	*r0 |= *r4;
	*r0 ^= *r2;
	*r3 ^= *r1;
	*r2 ^= *r3;
	*r3 &= *r0;
	*r3 ^= *r4;
	*r4 ^= *r2;
	*r2 &= *r0;
	*r4 = ~*r4;   
	*r2 ^= *r4;
	*r4 &= *r0;
	*r1 ^= *r3;
	*r4 ^= *r1;
}

__device__  void KXf (const unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	*a ^= k[r];
	*b ^= k[r + 1];
	*c ^= k[r + 2];
	*d ^= k[r + 3];
}

#endif // TC_MINIMIZE_CODE_SIZE

#ifndef TC_MINIMIZE_CODE_SIZE

__device__ void serpent_set_key(const unsigned __int8 userKey[], int keylen, unsigned __int8 *ks)
{
	unsigned __int32 a,b,c,d,e;
	unsigned __int32 *k = (unsigned __int32 *)ks;
	unsigned __int32 t;
	int i;

	for (i = 0; i < keylen / (int)sizeof(__int32); i++)
		k[i] = LE32(((unsigned __int32*)userKey)[i]);

	if (keylen < 32)
		k[keylen/4] |= (unsigned __int32)1 << ((keylen%4)*8);

	k += 8;
	t = k[-1];
	for (i = 0; i < 132; ++i)
		k[i] = t = rotlFixed(k[i-8] ^ k[i-5] ^ k[i-3] ^ t ^ 0x9e3779b9 ^ i, 11);
	k -= 20;

#define LK(r, a, b, c, d, e)	{\
	a = k[(8-r)*4 + 0];		\
	b = k[(8-r)*4 + 1];		\
	c = k[(8-r)*4 + 2];		\
	d = k[(8-r)*4 + 3];}

#define SK(r, a, b, c, d, e)	{\
	k[(8-r)*4 + 4] = a;		\
	k[(8-r)*4 + 5] = b;		\
	k[(8-r)*4 + 6] = c;		\
	k[(8-r)*4 + 7] = d;}	\

	for (i=0; i<4; i++)
	{
		afterS2(LK); afterS2(S3); afterS3(SK);
		afterS1(LK); afterS1(S2); afterS2(SK);
		afterS0(LK); afterS0(S1); afterS1(SK);
		beforeS0(LK); beforeS0(S0); afterS0(SK);
		k += 8*4;
		afterS6(LK); afterS6(S7); afterS7(SK);
		afterS5(LK); afterS5(S6); afterS6(SK);
		afterS4(LK); afterS4(S5); afterS5(SK);
		afterS3(LK); afterS3(S4); afterS4(SK);
	}
	afterS2(LK); afterS2(S3); afterS3(SK);
}

#else // TC_MINIMIZE_CODE_SIZE

__device__  void LKf (unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	*a = k[r];
	*b = k[r + 1];
	*c = k[r + 2];
	*d = k[r + 3];
}

__device__  void SKf (unsigned __int32 *k, unsigned int r, unsigned __int32 *a, unsigned __int32 *b, unsigned __int32 *c, unsigned __int32 *d)
{
	k[r + 4] = *a;
	k[r + 5] = *b;
	k[r + 6] = *c;
	k[r + 7] = *d;
}

__device__ void serpent_set_key(const unsigned __int8 userKey[], int keylen, unsigned __int8 *ks)
{
	unsigned __int32 a,b,c,d,e;
	unsigned __int32 *k = (unsigned __int32 *)ks;
	unsigned __int32 t;	
	int i;

	for (i = 0; i < keylen / (int)sizeof(__int32); i++)
		k[i] = LE32(((unsigned __int32*)userKey)[i]);

	if (keylen < 32)
		k[keylen/4] |= (unsigned __int32)1 << ((keylen%4)*8);

	k += 8;
	t = k[-1];
	for (i = 0; i < 132; ++i)
		k[i] = t = rotlFixed(k[i-8] ^ k[i-5] ^ k[i-3] ^ t ^ 0x9e3779b9 ^ i, 11);
	k -= 20;

	for (i=0; i<4; i++)
	{
		LKf (k, 20, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); SKf (k, 16, &e, &b, &d, &c);
		LKf (k, 24, &c, &b, &a, &e); S2f (&c, &b, &a, &e, &d); SKf (k, 20, &a, &e, &b, &d);
		LKf (k, 28, &b, &e, &c, &a); S1f (&b, &e, &c, &a, &d); SKf (k, 24, &c, &b, &a, &e);
		LKf (k, 32, &a, &b, &c, &d); S0f (&a, &b, &c, &d, &e); SKf (k, 28, &b, &e, &c, &a);
		k += 8*4;
		LKf (k,  4, &a, &c, &d, &b); S7f (&a, &c, &d, &b, &e); SKf (k,  0, &d, &e, &b, &a);
		LKf (k,  8, &a, &c, &b, &e); S6f (&a, &c, &b, &e, &d); SKf (k,  4, &a, &c, &d, &b);
		LKf (k, 12, &b, &a, &e, &c); S5f (&b, &a, &e, &c, &d); SKf (k,  8, &a, &c, &b, &e);
		LKf (k, 16, &e, &b, &d, &c); S4f (&e, &b, &d, &c, &a); SKf (k, 12, &b, &a, &e, &c);
	}
	LKf (k, 20, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); SKf (k, 16, &e, &b, &d, &c);
	
}

#endif // TC_MINIMIZE_CODE_SIZE


#ifndef TC_MINIMIZE_CODE_SIZE

__device__ void serpent_encrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	unsigned int i=1;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 8;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	do
	{
		beforeS0(KX); beforeS0(S0); afterS0(LT);
		afterS0(KX); afterS0(S1); afterS1(LT);
		afterS1(KX); afterS1(S2); afterS2(LT);
		afterS2(KX); afterS2(S3); afterS3(LT);
		afterS3(KX); afterS3(S4); afterS4(LT);
		afterS4(KX); afterS4(S5); afterS5(LT);
		afterS5(KX); afterS5(S6); afterS6(LT);
		afterS6(KX); afterS6(S7);

		if (i == 4)
			break;

		++i;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		k += 32;
		beforeS0(LT);
	}
	while (1);

	afterS7(KX);
	
    out[0] = LE32(d);
	out[1] = LE32(e);
	out[2] = LE32(b);
	out[3] = LE32(a);
}

#else // TC_MINIMIZE_CODE_SIZE

typedef unsigned __int32 uint32;

__device__  void LTf (uint32 *a, uint32 *b, uint32 *c, uint32 *d)
{
	*a = rotlFixed(*a, 13);
	*c = rotlFixed(*c, 3);
	*d = rotlFixed(*d ^ *c ^ (*a << 3), 7);
	*b = rotlFixed(*b ^ *a ^ *c, 1);
	*a = rotlFixed(*a ^ *b ^ *d, 5);
	*c = rotlFixed(*c ^ *d ^ (*b << 7), 22);
}

__device__ void serpent_encrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	unsigned int i=1;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 8;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	do
	{
		KXf (k,  0, &a, &b, &c, &d); S0f (&a, &b, &c, &d, &e); LTf (&b, &e, &c, &a);
		KXf (k,  4, &b, &e, &c, &a); S1f (&b, &e, &c, &a, &d); LTf (&c, &b, &a, &e);
		KXf (k,  8, &c, &b, &a, &e); S2f (&c, &b, &a, &e, &d); LTf (&a, &e, &b, &d);
		KXf (k, 12, &a, &e, &b, &d); S3f (&a, &e, &b, &d, &c); LTf (&e, &b, &d, &c);
		KXf (k, 16, &e, &b, &d, &c); S4f (&e, &b, &d, &c, &a); LTf (&b, &a, &e, &c);
		KXf (k, 20, &b, &a, &e, &c); S5f (&b, &a, &e, &c, &d); LTf (&a, &c, &b, &e);
		KXf (k, 24, &a, &c, &b, &e); S6f (&a, &c, &b, &e, &d); LTf (&a, &c, &d, &b);
		KXf (k, 28, &a, &c, &d, &b); S7f (&a, &c, &d, &b, &e);

		if (i == 4)
			break;

		++i;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		k += 32;
		LTf (&a,&b,&c,&d);
	}
	while (1);

	KXf (k, 32, &d, &e, &b, &a);
	
    out[0] = LE32(d);
	out[1] = LE32(e);
	out[2] = LE32(b);
	out[3] = LE32(a);
}

#endif // TC_MINIMIZE_CODE_SIZE

#if !defined (TC_MINIMIZE_CODE_SIZE) || defined (TC_WINDOWS_BOOT_SERPENT)

__device__ void serpent_decrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 104;
	unsigned int i=4;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	beforeI7(KX);
	goto start;

	do
	{
		c = b;
		b = d;
		d = e;
		k -= 32;
		beforeI7(ILT);
start:
		beforeI7(I7); afterI7(KX); 
		afterI7(ILT); afterI7(I6); afterI6(KX); 
		afterI6(ILT); afterI6(I5); afterI5(KX); 
		afterI5(ILT); afterI5(I4); afterI4(KX); 
		afterI4(ILT); afterI4(I3); afterI3(KX); 
		afterI3(ILT); afterI3(I2); afterI2(KX); 
		afterI2(ILT); afterI2(I1); afterI1(KX); 
		afterI1(ILT); afterI1(I0); afterI0(KX);
	}
	while (--i != 0);
	
    out[0] = LE32(a);
	out[1] = LE32(d);
	out[2] = LE32(b);
	out[3] = LE32(e);
}

#else // TC_MINIMIZE_CODE_SIZE && !TC_WINDOWS_BOOT_SERPENT

__device__  void ILTf (uint32 *a, uint32 *b, uint32 *c, uint32 *d)
{ 
	*c = rotrFixed(*c, 22);
	*a = rotrFixed(*a, 5);
	*c ^= *d ^ (*b << 7);
	*a ^= *b ^ *d;
	*b = rotrFixed(*b, 1);
	*d = rotrFixed(*d, 7) ^ *c ^ (*a << 3);
	*b ^= *a ^ *c;
	*c = rotrFixed(*c, 3);
	*a = rotrFixed(*a, 13);
}

__device__ void serpent_decrypt(const unsigned __int8 *inBlock, unsigned __int8 *outBlock, unsigned __int8 *ks)
{
	unsigned __int32 a, b, c, d, e;
	const unsigned __int32 *k = (unsigned __int32 *)ks + 104;
	unsigned int i=4;
	unsigned __int32 *in = (unsigned __int32 *) inBlock;
	unsigned __int32 *out = (unsigned __int32 *) outBlock;

    a = LE32(in[0]);
	b = LE32(in[1]);
	c = LE32(in[2]);
	d = LE32(in[3]);

	KXf (k, 32, &a, &b, &c, &d);
	goto start;

	do
	{
		c = b;
		b = d;
		d = e;
		k -= 32;
		beforeI7(ILT);
start:
		beforeI7(I7); KXf (k, 28, &d, &a, &b, &e);
		ILTf (&d, &a, &b, &e); afterI7(I6); KXf (k, 24, &a, &b, &c, &e); 
		ILTf (&a, &b, &c, &e); afterI6(I5); KXf (k, 20, &b, &d, &e, &c); 
		ILTf (&b, &d, &e, &c); afterI5(I4); KXf (k, 16, &b, &c, &e, &a); 
		ILTf (&b, &c, &e, &a); afterI4(I3); KXf (k, 12, &a, &b, &e, &c);
		ILTf (&a, &b, &e, &c); afterI3(I2); KXf (k, 8,  &b, &d, &e, &c);
		ILTf (&b, &d, &e, &c); afterI2(I1); KXf (k, 4,  &a, &b, &c, &e);
		ILTf (&a, &b, &c, &e); afterI1(I0); KXf (k, 0,  &a, &d, &b, &e);
	}
	while (--i != 0);
	
    out[0] = LE32(a);
	out[1] = LE32(d);
	out[2] = LE32(b);
	out[3] = LE32(e);
}

#endif // TC_MINIMIZE_CODE_SIZE && !TC_WINDOWS_BOOT_SERPENT
/*
 ---------------------------------------------------------------------------
 Copyright (c) 1999, Dr Brian Gladman, Worcester, UK.   All rights reserved.

 LICENSE TERMS

 The free distribution and use of this software is allowed (with or without
 changes) provided that:

  1. source code distributions include the above copyright notice, this
     list of conditions and the following disclaimer;

  2. binary distributions include the above copyright notice, this list
     of conditions and the following disclaimer in their documentation;

  3. the name of the copyright holder is not used to endorse products
     built using this software without specific written permission.

 DISCLAIMER

 This software is provided 'as is' with no explicit or implied warranties
 in respect of its properties, including, but not limited to, correctness
 and/or fitness for purpose.
 ---------------------------------------------------------------------------

 My thanks to Doug Whiting and Niels Ferguson for comments that led
 to improvements in this implementation.

 Issue Date: 14th January 1999
*/

/* Adapted for TrueCrypt */


#ifdef TC_WINDOWS_BOOT
#pragma optimize ("tl", on)
#endif

#include "Twofish.cuh"
#include "Common/Endian.h"

#define Q_TABLES
#define M_TABLE

#if !defined (TC_MINIMIZE_CODE_SIZE) || defined (TC_WINDOWS_BOOT_TWOFISH)
#	define MK_TABLE
#	define ONE_STEP
#endif

/* finite field arithmetic for GF(2**8) with the modular    */
/* polynomial x^8 + x^6 + x^5 + x^3 + 1 (0x169)             */

#define G_M 0x0169

__device__ static u1byte  tab_5b[4] = { 0, G_M >> 2, G_M >> 1, (G_M >> 1) ^ (G_M >> 2) };
__device__ static u1byte  tab_ef[4] = { 0, (G_M >> 1) ^ (G_M >> 2), G_M >> 1, G_M >> 2 };

#define ffm_01(x)    (x)
#define ffm_5b(x)   ((x) ^ ((x) >> 2) ^ tab_5b[(x) & 3])
#define ffm_ef(x)   ((x) ^ ((x) >> 1) ^ ((x) >> 2) ^ tab_ef[(x) & 3])

__device__ static u1byte ror4[16] = { 0, 8, 1, 9, 2, 10, 3, 11, 4, 12, 5, 13, 6, 14, 7, 15 };
__device__ static u1byte ashx[16] = { 0, 9, 2, 11, 4, 13, 6, 15, 8, 1, 10, 3, 12, 5, 14, 7 };

__device__ static u1byte qt0[2][16] = 
{   { 8, 1, 7, 13, 6, 15, 3, 2, 0, 11, 5, 9, 14, 12, 10, 4 },
    { 2, 8, 11, 13, 15, 7, 6, 14, 3, 1, 9, 4, 0, 10, 12, 5 }
};

__device__ static u1byte qt1[2][16] =
{   { 14, 12, 11, 8, 1, 2, 3, 5, 15, 4, 10, 6, 7, 0, 9, 13 }, 
    { 1, 14, 2, 11, 4, 12, 3, 7, 6, 13, 10, 5, 15, 9, 0, 8 }
};

__device__ static u1byte qt2[2][16] = 
{   { 11, 10, 5, 14, 6, 13, 9, 0, 12, 8, 15, 3, 2, 4, 7, 1 },
    { 4, 12, 7, 5, 1, 6, 9, 10, 0, 14, 13, 8, 2, 11, 3, 15 }
};

__device__ static u1byte qt3[2][16] = 
{   { 13, 7, 15, 4, 1, 2, 6, 14, 9, 11, 3, 0, 8, 5, 12, 10 },
    { 11, 9, 5, 1, 12, 3, 13, 14, 6, 4, 7, 15, 2, 0, 8, 10 }
};
 
__device__ static u1byte qp(const u4byte n, const u1byte x)
{   u1byte  a0, a1, a2, a3, a4, b0, b1, b2, b3, b4;

    a0 = x >> 4; b0 = x & 15;
    a1 = a0 ^ b0; b1 = ror4[b0] ^ ashx[a0];
    a2 = qt0[n][a1]; b2 = qt1[n][b1];
    a3 = a2 ^ b2; b3 = ror4[b2] ^ ashx[a2];
    a4 = qt2[n][a3]; b4 = qt3[n][b3];
    return (b4 << 4) | a4;
};

#ifdef  Q_TABLES

__device__ static u4byte  qt_gen = 0;
__device__ static u1byte  q_tab[2][256];

#define q(n,x)  q_tab[n][x]

__device__ static void gen_qtab(void)
{   u4byte  i;

    for(i = 0; i < 256; ++i)
    {       
        q(0,i) = qp(0, (u1byte)i);
        q(1,i) = qp(1, (u1byte)i);
    }
};

#else

#define q(n,x)  qp(n, x)

#endif

#ifdef  M_TABLE

__device__ static u4byte  mt_gen = 0;
__device__ static u4byte  m_tab[4][256];

__device__ static void gen_mtab(void)
{   u4byte  i, f01, f5b, fef;
    
    for(i = 0; i < 256; ++i)
    {
        f01 = q(1,i); f5b = ffm_5b(f01); fef = ffm_ef(f01);
        m_tab[0][i] = f01 + (f5b << 8) + (fef << 16) + (fef << 24);
        m_tab[2][i] = f5b + (fef << 8) + (f01 << 16) + (fef << 24);

        f01 = q(0,i); f5b = ffm_5b(f01); fef = ffm_ef(f01);
        m_tab[1][i] = fef + (fef << 8) + (f5b << 16) + (f01 << 24);
        m_tab[3][i] = f5b + (f01 << 8) + (fef << 16) + (f5b << 24);
    }
};

#define mds(n,x)    m_tab[n][x]

#else

#define fm_00   ffm_01
#define fm_10   ffm_5b
#define fm_20   ffm_ef
#define fm_30   ffm_ef
#define q_0(x)  q(1,x)

#define fm_01   ffm_ef
#define fm_11   ffm_ef
#define fm_21   ffm_5b
#define fm_31   ffm_01
#define q_1(x)  q(0,x)

#define fm_02   ffm_5b
#define fm_12   ffm_ef
#define fm_22   ffm_01
#define fm_32   ffm_ef
#define q_2(x)  q(1,x)

#define fm_03   ffm_5b
#define fm_13   ffm_01
#define fm_23   ffm_ef
#define fm_33   ffm_5b
#define q_3(x)  q(0,x)

#define f_0(n,x)    ((u4byte)fm_0##n(x))
#define f_1(n,x)    ((u4byte)fm_1##n(x) << 8)
#define f_2(n,x)    ((u4byte)fm_2##n(x) << 16)
#define f_3(n,x)    ((u4byte)fm_3##n(x) << 24)

#define mds(n,x)    f_0(n,q_##n(x)) ^ f_1(n,q_##n(x)) ^ f_2(n,q_##n(x)) ^ f_3(n,q_##n(x))

#endif

__device__ static u4byte h_fun(TwofishInstance *instance, const u4byte x, const u4byte key[])
{   u4byte  b0, b1, b2, b3;

#ifndef M_TABLE
    u4byte  m5b_b0, m5b_b1, m5b_b2, m5b_b3;
    u4byte  mef_b0, mef_b1, mef_b2, mef_b3;
#endif

    b0 = extract_byte(x, 0); b1 = extract_byte(x, 1); b2 = extract_byte(x, 2); b3 = extract_byte(x, 3);

    switch(instance->k_len)
    {
    case 4: b0 = q(1, (u1byte) b0) ^ extract_byte(key[3],0);
            b1 = q(0, (u1byte) b1) ^ extract_byte(key[3],1);
            b2 = q(0, (u1byte) b2) ^ extract_byte(key[3],2);
            b3 = q(1, (u1byte) b3) ^ extract_byte(key[3],3);
    case 3: b0 = q(1, (u1byte) b0) ^ extract_byte(key[2],0);
            b1 = q(1, (u1byte) b1) ^ extract_byte(key[2],1);
            b2 = q(0, (u1byte) b2) ^ extract_byte(key[2],2);
            b3 = q(0, (u1byte) b3) ^ extract_byte(key[2],3);
    case 2: b0 = q(0, (u1byte) (q(0, (u1byte) b0) ^ extract_byte(key[1],0))) ^ extract_byte(key[0],0);
            b1 = q(0, (u1byte) (q(1, (u1byte) b1) ^ extract_byte(key[1],1))) ^ extract_byte(key[0],1);
            b2 = q(1, (u1byte) (q(0, (u1byte) b2) ^ extract_byte(key[1],2))) ^ extract_byte(key[0],2);
            b3 = q(1, (u1byte) (q(1, (u1byte) b3) ^ extract_byte(key[1],3))) ^ extract_byte(key[0],3);
    }
#ifdef  M_TABLE

    return  mds(0, b0) ^ mds(1, b1) ^ mds(2, b2) ^ mds(3, b3);

#else

    b0 = q(1, (u1byte) b0); b1 = q(0, (u1byte) b1); b2 = q(1, (u1byte) b2); b3 = q(0, (u1byte) b3);
    m5b_b0 = ffm_5b(b0); m5b_b1 = ffm_5b(b1); m5b_b2 = ffm_5b(b2); m5b_b3 = ffm_5b(b3);
    mef_b0 = ffm_ef(b0); mef_b1 = ffm_ef(b1); mef_b2 = ffm_ef(b2); mef_b3 = ffm_ef(b3);
    b0 ^= mef_b1 ^ m5b_b2 ^ m5b_b3; b3 ^= m5b_b0 ^ mef_b1 ^ mef_b2;
    b2 ^= mef_b0 ^ m5b_b1 ^ mef_b3; b1 ^= mef_b0 ^ mef_b2 ^ m5b_b3;

    return b0 | (b3 << 8) | (b2 << 16) | (b1 << 24);

#endif
};

#ifdef  MK_TABLE

#ifdef  ONE_STEP
//u4byte  mk_tab[4][256];
#else
__device__ static u1byte  sb[4][256];
#endif

#define q20(x)  q(0,q(0,x) ^ extract_byte(key[1],0)) ^ extract_byte(key[0],0)
#define q21(x)  q(0,q(1,x) ^ extract_byte(key[1],1)) ^ extract_byte(key[0],1)
#define q22(x)  q(1,q(0,x) ^ extract_byte(key[1],2)) ^ extract_byte(key[0],2)
#define q23(x)  q(1,q(1,x) ^ extract_byte(key[1],3)) ^ extract_byte(key[0],3)

#define q30(x)  q(0,q(0,q(1, x) ^ extract_byte(key[2],0)) ^ extract_byte(key[1],0)) ^ extract_byte(key[0],0)
#define q31(x)  q(0,q(1,q(1, x) ^ extract_byte(key[2],1)) ^ extract_byte(key[1],1)) ^ extract_byte(key[0],1)
#define q32(x)  q(1,q(0,q(0, x) ^ extract_byte(key[2],2)) ^ extract_byte(key[1],2)) ^ extract_byte(key[0],2)
#define q33(x)  q(1,q(1,q(0, x) ^ extract_byte(key[2],3)) ^ extract_byte(key[1],3)) ^ extract_byte(key[0],3)

#define q40(x)  q(0,q(0,q(1, q(1, x) ^ extract_byte(key[3],0)) ^ extract_byte(key[2],0)) ^ extract_byte(key[1],0)) ^ extract_byte(key[0],0)
#define q41(x)  q(0,q(1,q(1, q(0, x) ^ extract_byte(key[3],1)) ^ extract_byte(key[2],1)) ^ extract_byte(key[1],1)) ^ extract_byte(key[0],1)
#define q42(x)  q(1,q(0,q(0, q(0, x) ^ extract_byte(key[3],2)) ^ extract_byte(key[2],2)) ^ extract_byte(key[1],2)) ^ extract_byte(key[0],2)
#define q43(x)  q(1,q(1,q(0, q(1, x) ^ extract_byte(key[3],3)) ^ extract_byte(key[2],3)) ^ extract_byte(key[1],3)) ^ extract_byte(key[0],3)

__device__ static void gen_mk_tab(TwofishInstance *instance, u4byte key[])
{   u4byte  i;
    u1byte  by;

	u4byte *mk_tab = instance->mk_tab;

    switch(instance->k_len)
    {
    case 2: for(i = 0; i < 256; ++i)
            {
                by = (u1byte)i;
#ifdef ONE_STEP
                mk_tab[0 + 4*i] = mds(0, q20(by)); mk_tab[1 + 4*i] = mds(1, q21(by));
                mk_tab[2 + 4*i] = mds(2, q22(by)); mk_tab[3 + 4*i] = mds(3, q23(by));
#else
                sb[0][i] = q20(by); sb[1][i] = q21(by); 
                sb[2][i] = q22(by); sb[3][i] = q23(by);
#endif
            }
            break;
    
    case 3: for(i = 0; i < 256; ++i)
            {
                by = (u1byte)i;
#ifdef ONE_STEP
                mk_tab[0 + 4*i] = mds(0, q30(by)); mk_tab[1 + 4*i] = mds(1, q31(by));
                mk_tab[2 + 4*i] = mds(2, q32(by)); mk_tab[3 + 4*i] = mds(3, q33(by));
#else
                sb[0][i] = q30(by); sb[1][i] = q31(by); 
                sb[2][i] = q32(by); sb[3][i] = q33(by);
#endif
            }
            break;
    
    case 4: for(i = 0; i < 256; ++i)
            {
                by = (u1byte)i;
#ifdef ONE_STEP
                mk_tab[0 + 4*i] = mds(0, q40(by)); mk_tab[1 + 4*i] = mds(1, q41(by));
                mk_tab[2 + 4*i] = mds(2, q42(by)); mk_tab[3 + 4*i] = mds(3, q43(by));
#else
                sb[0][i] = q40(by); sb[1][i] = q41(by); 
                sb[2][i] = q42(by); sb[3][i] = q43(by);
#endif
            }
    }
};

#  ifdef ONE_STEP
#    define g0_fun(x) ( mk_tab[0 + 4*extract_byte(x,0)] ^ mk_tab[1 + 4*extract_byte(x,1)] \
                      ^ mk_tab[2 + 4*extract_byte(x,2)] ^ mk_tab[3 + 4*extract_byte(x,3)] )
#    define g1_fun(x) ( mk_tab[0 + 4*extract_byte(x,3)] ^ mk_tab[1 + 4*extract_byte(x,0)] \
                      ^ mk_tab[2 + 4*extract_byte(x,1)] ^ mk_tab[3 + 4*extract_byte(x,2)] )


#  else
#    define g0_fun(x) ( mds(0, sb[0][extract_byte(x,0)]) ^ mds(1, sb[1][extract_byte(x,1)]) \
                      ^ mds(2, sb[2][extract_byte(x,2)]) ^ mds(3, sb[3][extract_byte(x,3)]) )
#    define g1_fun(x) ( mds(0, sb[0][extract_byte(x,3)]) ^ mds(1, sb[1][extract_byte(x,0)]) \
                      ^ mds(2, sb[2][extract_byte(x,1)]) ^ mds(3, sb[3][extract_byte(x,2)]) )
#  endif

#else

#define g0_fun(x)   h_fun(instance, x, instance->s_key)
#define g1_fun(x)   h_fun(instance, rotl(x,8), instance->s_key)

#endif

/* The (12,8) Reed Soloman code has the generator polynomial

  g(x) = x^4 + (a + 1/a) * x^3 + a * x^2 + (a + 1/a) * x + 1

where the coefficients are in the finite field GF(2^8) with a
modular polynomial a^8 + a^6 + a^3 + a^2 + 1. To generate the
remainder we have to start with a 12th order polynomial with our
eight input bytes as the coefficients of the 4th to 11th terms. 
That is:

  m[7] * x^11 + m[6] * x^10 ... + m[0] * x^4 + 0 * x^3 +... + 0
  
We then multiply the generator polynomial by m[7] * x^7 and subtract
it - xor in GF(2^8) - from the above to eliminate the x^7 term (the 
artihmetic on the coefficients is done in GF(2^8). We then multiply 
the generator polynomial by x^6 * coeff(x^10) and use this to remove
the x^10 term. We carry on in this way until the x^4 term is removed
so that we are left with:

  r[3] * x^3 + r[2] * x^2 + r[1] 8 x^1 + r[0]

which give the resulting 4 bytes of the remainder. This is equivalent 
to the matrix multiplication in the Twofish description but much faster 
to implement.

*/

#define G_MOD   0x0000014d

__device__ static u4byte mds_rem(u4byte p0, u4byte p1)
{   u4byte  i, t, u;

    for(i = 0; i < 8; ++i)
    {
        t = p1 >> 24;   // get most significant coefficient
        
        p1 = (p1 << 8) | (p0 >> 24); p0 <<= 8;  // shift others up
            
        // multiply t by a (the primitive element - i.e. left shift)

        u = (t << 1); 
        
        if(t & 0x80)            // subtract modular polynomial on overflow
        
            u ^= G_MOD; 

        p1 ^= t ^ (u << 16);    // remove t * (a * x^2 + 1)  

        u ^= (t >> 1);          // form u = a * t + t / a = t * (a + 1 / a); 
        
        if(t & 0x01)            // add the modular polynomial on underflow
        
            u ^= G_MOD >> 1;

        p1 ^= (u << 24) | (u << 8); // remove t * (a + 1/a) * (x^3 + x)
    }

    return p1;
};

/* initialise the key schedule from the user supplied key   */

__device__ u4byte *twofish_set_key(TwofishInstance *instance, const u4byte in_key[], const u4byte key_len)
{   u4byte  i, a, b, me_key[4], mo_key[4];
	u4byte *l_key, *s_key;

	l_key = instance->l_key;
	s_key = instance->s_key;

#ifdef Q_TABLES
    if(!qt_gen)
    {
        gen_qtab(); qt_gen = 1;
    }
#endif

#ifdef M_TABLE
    if(!mt_gen)
    {
        gen_mtab(); mt_gen = 1;
    }
#endif

    instance->k_len = key_len / 64;   /* 2, 3 or 4 */

    for(i = 0; i < instance->k_len; ++i)
    {
        a = LE32(in_key[i + i]);     me_key[i] = a;
        b = LE32(in_key[i + i + 1]); mo_key[i] = b;
        s_key[instance->k_len - i - 1] = mds_rem(a, b);
    }

    for(i = 0; i < 40; i += 2)
    {
        a = 0x01010101 * i; b = a + 0x01010101;
        a = h_fun(instance, a, me_key);
        b = rotl(h_fun(instance, b, mo_key), 8);
        l_key[i] = a + b;
        l_key[i + 1] = rotl(a + 2 * b, 9);
    }

#ifdef MK_TABLE
    gen_mk_tab(instance, s_key);
#endif

    return l_key;
};

/* encrypt a block of text  */

#ifndef TC_MINIMIZE_CODE_SIZE

#define f_rnd(i)                                                    \
    t1 = g1_fun(blk[1]); t0 = g0_fun(blk[0]);                       \
    blk[2] = rotr(blk[2] ^ (t0 + t1 + l_key[4 * (i) + 8]), 1);      \
    blk[3] = rotl(blk[3], 1) ^ (t0 + 2 * t1 + l_key[4 * (i) + 9]);  \
    t1 = g1_fun(blk[3]); t0 = g0_fun(blk[2]);                       \
    blk[0] = rotr(blk[0] ^ (t0 + t1 + l_key[4 * (i) + 10]), 1);     \
    blk[1] = rotl(blk[1], 1) ^ (t0 + 2 * t1 + l_key[4 * (i) + 11])

__device__ void twofish_encrypt(TwofishInstance *instance, const u4byte in_blk[4], u4byte out_blk[])
{   u4byte  t0, t1, blk[4];

	u4byte *l_key = instance->l_key;
	u4byte *mk_tab = instance->mk_tab;

	blk[0] = LE32(in_blk[0]) ^ l_key[0];
    blk[1] = LE32(in_blk[1]) ^ l_key[1];
    blk[2] = LE32(in_blk[2]) ^ l_key[2];
    blk[3] = LE32(in_blk[3]) ^ l_key[3];

    f_rnd(0); f_rnd(1); f_rnd(2); f_rnd(3);
    f_rnd(4); f_rnd(5); f_rnd(6); f_rnd(7);

    out_blk[0] = LE32(blk[2] ^ l_key[4]);
    out_blk[1] = LE32(blk[3] ^ l_key[5]);
    out_blk[2] = LE32(blk[0] ^ l_key[6]);
    out_blk[3] = LE32(blk[1] ^ l_key[7]); 
};

#else // TC_MINIMIZE_CODE_SIZE

__device__ void twofish_encrypt(TwofishInstance *instance, const u4byte in_blk[4], u4byte out_blk[])
{   u4byte  t0, t1, blk[4];

	u4byte *l_key = instance->l_key;
#ifdef TC_WINDOWS_BOOT_TWOFISH
	u4byte *mk_tab = instance->mk_tab;
#endif
	int i;

	blk[0] = LE32(in_blk[0]) ^ l_key[0];
    blk[1] = LE32(in_blk[1]) ^ l_key[1];
    blk[2] = LE32(in_blk[2]) ^ l_key[2];
    blk[3] = LE32(in_blk[3]) ^ l_key[3];

	for (i = 0; i <= 7; ++i)
	{
		t1 = g1_fun(blk[1]); t0 = g0_fun(blk[0]);
		blk[2] = rotr(blk[2] ^ (t0 + t1 + l_key[4 * (i) + 8]), 1);
		blk[3] = rotl(blk[3], 1) ^ (t0 + 2 * t1 + l_key[4 * (i) + 9]);
		t1 = g1_fun(blk[3]); t0 = g0_fun(blk[2]);
		blk[0] = rotr(blk[0] ^ (t0 + t1 + l_key[4 * (i) + 10]), 1);
		blk[1] = rotl(blk[1], 1) ^ (t0 + 2 * t1 + l_key[4 * (i) + 11]);
	}

    out_blk[0] = LE32(blk[2] ^ l_key[4]);
    out_blk[1] = LE32(blk[3] ^ l_key[5]);
    out_blk[2] = LE32(blk[0] ^ l_key[6]);
    out_blk[3] = LE32(blk[1] ^ l_key[7]); 
};

#endif // TC_MINIMIZE_CODE_SIZE

/* decrypt a block of text  */

#ifndef TC_MINIMIZE_CODE_SIZE

#define i_rnd(i)                                                        \
        t1 = g1_fun(blk[1]); t0 = g0_fun(blk[0]);                       \
        blk[2] = rotl(blk[2], 1) ^ (t0 + t1 + l_key[4 * (i) + 10]);     \
        blk[3] = rotr(blk[3] ^ (t0 + 2 * t1 + l_key[4 * (i) + 11]), 1); \
        t1 = g1_fun(blk[3]); t0 = g0_fun(blk[2]);                       \
        blk[0] = rotl(blk[0], 1) ^ (t0 + t1 + l_key[4 * (i) +  8]);     \
        blk[1] = rotr(blk[1] ^ (t0 + 2 * t1 + l_key[4 * (i) +  9]), 1)

__device__ void twofish_decrypt(TwofishInstance *instance, const u4byte in_blk[4], u4byte out_blk[4])
{   u4byte  t0, t1, blk[4];

	u4byte *l_key = instance->l_key;
	u4byte *mk_tab = instance->mk_tab;

    blk[0] = LE32(in_blk[0]) ^ l_key[4];
    blk[1] = LE32(in_blk[1]) ^ l_key[5];
    blk[2] = LE32(in_blk[2]) ^ l_key[6];
    blk[3] = LE32(in_blk[3]) ^ l_key[7];

    i_rnd(7); i_rnd(6); i_rnd(5); i_rnd(4);
    i_rnd(3); i_rnd(2); i_rnd(1); i_rnd(0);

    out_blk[0] = LE32(blk[2] ^ l_key[0]);
    out_blk[1] = LE32(blk[3] ^ l_key[1]);
    out_blk[2] = LE32(blk[0] ^ l_key[2]);
    out_blk[3] = LE32(blk[1] ^ l_key[3]); 
};

#else // TC_MINIMIZE_CODE_SIZE

__device__ void twofish_decrypt(TwofishInstance *instance, const u4byte in_blk[4], u4byte out_blk[4])
{   u4byte  t0, t1, blk[4];

	u4byte *l_key = instance->l_key;
#ifdef TC_WINDOWS_BOOT_TWOFISH
	u4byte *mk_tab = instance->mk_tab;
#endif
	int i;

    blk[0] = LE32(in_blk[0]) ^ l_key[4];
    blk[1] = LE32(in_blk[1]) ^ l_key[5];
    blk[2] = LE32(in_blk[2]) ^ l_key[6];
    blk[3] = LE32(in_blk[3]) ^ l_key[7];

	for (i = 7; i >= 0; --i)
	{
		t1 = g1_fun(blk[1]); t0 = g0_fun(blk[0]);
		blk[2] = rotl(blk[2], 1) ^ (t0 + t1 + l_key[4 * (i) + 10]);
		blk[3] = rotr(blk[3] ^ (t0 + 2 * t1 + l_key[4 * (i) + 11]), 1);
		t1 = g1_fun(blk[3]); t0 = g0_fun(blk[2]);
		blk[0] = rotl(blk[0], 1) ^ (t0 + t1 + l_key[4 * (i) +  8]);
		blk[1] = rotr(blk[1] ^ (t0 + 2 * t1 + l_key[4 * (i) +  9]), 1);
	}

    out_blk[0] = LE32(blk[2] ^ l_key[0]);
    out_blk[1] = LE32(blk[3] ^ l_key[1]);
    out_blk[2] = LE32(blk[0] ^ l_key[2]);
    out_blk[3] = LE32(blk[1] ^ l_key[3]); 
};

#endif // TC_MINIMIZE_CODE_SIZE
/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
/*
 Legal Notice: Some portions of the source code contained in this file were
 derived from the source code of Encryption for the Masses 2.02a, which is
 Copyright (c) 1998-2000 Paul Le Roux and which is governed by the 'License
 Agreement for Encryption for the Masses'. Modifications and additions to
 the original source code (contained in this file) and all other portions
 of this file are Copyright (c) 2003-2010 TrueCrypt Developers Association
 and are governed by the TrueCrypt License 3.0 the full text of which is
 contained in the file License.txt included in TrueCrypt binary and source
 code distribution packages. */

#include "Tcdefs.h"
#include "Crypto.cuh"
#include "Xts.cuh"
#include "Crc.h"
#include "Common/Endian.h"
#include <string.h>
//#ifndef TC_WINDOWS_BOOT
//#include "EncryptionThreadPool.h"
//#endif
#include "Volumes.cuh"
#include "Twofish.cuh"


/*


// Cipher configuration
static Cipher Ciphers[] =
{
//								Block Size	Key Size	Key Schedule Size
//	  ID		Name			(Bytes)		(Bytes)		(Bytes)
	{ AES,		"AES",			16,			32,			AES_KS				},
	{ SERPENT,	"Serpent",		16,			32,			140*4				},
	{ TWOFISH,	"Twofish",		16,			32,			TWOFISH_KS			},
#ifndef TC_WINDOWS_BOOT
	{ BLOWFISH,	"Blowfish",		8,			56,			sizeof (BF_KEY)		},	// Deprecated/legacy
	{ CAST,		"CAST5",		8,			16,			sizeof (CAST_KEY)	},	// Deprecated/legacy
	{ TRIPLEDES,"Triple DES",	8,			8*3,		sizeof (TDES_KEY)	},	// Deprecated/legacy
#endif
	{ 0,		0,				0,			0,			0					}
};


// Encryption algorithm configuration
// The following modes have been deprecated (legacy): LRW, CBC, INNER_CBC, OUTER_CBC
static EncryptionAlgorithm EncryptionAlgorithms[] =
{
	//  Cipher(s)                     Modes						FormatEnabled

#ifndef TC_WINDOWS_BOOT

	{ { 0,						0 }, { 0, 0, 0, 0 },				0 },	// Must be all-zero
	{ { AES,					0 }, { XTS, LRW, CBC, 0 },			1 },
	{ { SERPENT,				0 }, { XTS, LRW, CBC, 0 },			1 },
	{ { TWOFISH,				0 }, { XTS, LRW, CBC, 0 },			1 },
	{ { TWOFISH, AES,			0 }, { XTS, LRW, OUTER_CBC, 0 },	1 },
	{ { SERPENT, TWOFISH, AES,	0 }, { XTS, LRW, OUTER_CBC, 0 },	1 },
	{ { AES, SERPENT,			0 }, { XTS, LRW, OUTER_CBC, 0 },	1 },
	{ { AES, TWOFISH, SERPENT,	0 }, { XTS, LRW, OUTER_CBC, 0 },	1 },
	{ { SERPENT, TWOFISH,		0 }, { XTS, LRW, OUTER_CBC, 0 },	1 },
	{ { BLOWFISH,				0 }, { LRW, CBC, 0, 0 },			0 },	// Deprecated/legacy
	{ { CAST,					0 }, { LRW, CBC, 0, 0 },			0 },	// Deprecated/legacy
	{ { TRIPLEDES,				0 }, { LRW, CBC, 0, 0 },			0 },	// Deprecated/legacy
	{ { BLOWFISH, AES,			0 }, { INNER_CBC, 0, 0, 0 },		0 },	// Deprecated/legacy
	{ { SERPENT, BLOWFISH, AES,	0 }, { INNER_CBC, 0, 0, 0 },		0 },	// Deprecated/legacy
	{ { 0,						0 }, { 0, 0, 0, 0 },				0 }		// Must be all-zero

#else // TC_WINDOWS_BOOT

	// Encryption algorithms available for boot drive encryption
	{ { 0,						0 }, { 0, 0 },		0 },	// Must be all-zero
	{ { AES,					0 }, { XTS, 0 },	1 },
	{ { SERPENT,				0 }, { XTS, 0 },	1 },
	{ { TWOFISH,				0 }, { XTS, 0 },	1 },
	{ { TWOFISH, AES,			0 }, { XTS, 0 },	1 },
	{ { SERPENT, TWOFISH, AES,	0 }, { XTS, 0 },	1 },
	{ { AES, SERPENT,			0 }, { XTS, 0 },	1 },
	{ { AES, TWOFISH, SERPENT,	0 }, { XTS, 0 },	1 },
	{ { SERPENT, TWOFISH,		0 }, { XTS, 0 },	1 },
	{ { 0,						0 }, { 0, 0 },		0 },	// Must be all-zero

#endif

};



// Hash algorithms
static Hash Hashes[] =
{	// ID			Name			Deprecated		System Encryption
	{ RIPEMD160,	"RIPEMD-160",	FALSE,			TRUE },
#ifndef TC_WINDOWS_BOOT
	{ SHA512,		"SHA-512",		FALSE,			FALSE },
	{ WHIRLPOOL,	"Whirlpool",	FALSE,			FALSE },
	{ SHA1,			"SHA-1",		TRUE,			FALSE },	// Deprecated/legacy
#endif
	{ 0, 0, 0 }
};
 */




/* Return values: 0 = success, ERR_CIPHER_INIT_FAILURE (fatal), ERR_CIPHER_INIT_WEAK_KEY (non-fatal) */
__device__ int cuCipherInit (int cipher, unsigned char *key, unsigned __int8 *ks)
{
    int retVal = ERR_SUCCESS;
	
    switch (cipher)
    {
		case AES:
#ifndef TC_WINDOWS_BOOT
			if (aes_encrypt_key256 (key, (aes_encrypt_ctx *) ks) != EXIT_SUCCESS)
				return ERR_CIPHER_INIT_FAILURE;
			
			if (aes_decrypt_key256 (key, (aes_decrypt_ctx *) (ks + sizeof(aes_encrypt_ctx))) != EXIT_SUCCESS)
				return ERR_CIPHER_INIT_FAILURE;
#else
			if (aes_set_key (key, (length_type) 32, (aes_context *) ks) != 0)
				return ERR_CIPHER_INIT_FAILURE;
#endif
			break;
			
		case SERPENT:
			serpent_set_key (key, 32 *8, ks);
			break;
			
		case TWOFISH:
			twofish_set_key ((TwofishInstance *)ks, (const u4byte *)key, 32 * 8);
			break;
		default:
			// Unknown/wrong cipher ID
			return ERR_CIPHER_INIT_FAILURE;
    }
	
    return retVal;
}

// Converts a 64-bit unsigned integer (passed as two 32-bit integers for compatibility with non-64-bit
// environments/platforms) into a little-endian 16-byte array.
__device__ void cuUint64ToLE16ByteArray (unsigned __int8 *byteBuf, unsigned __int32 highInt32, unsigned __int32 lowInt32)
{
    unsigned __int32 *bufPtr32 = (unsigned __int32 *) byteBuf;
	
    *bufPtr32++ = lowInt32;
    *bufPtr32++ = highInt32;
	
    // We're converting a 64-bit number into a little-endian 16-byte array so we can zero the last 8 bytes
    *bufPtr32++ = 0;
    *bufPtr32 = 0;
}

__device__ void cuEncipherBlock(int cipher, void *data, void *ks)
{
    switch (cipher)
    {
		case AES:
			// In 32-bit kernel mode, due to KeSaveFloatingPointState() overhead, AES instructions can be used only when processing the whole data unit.
			aes_encrypt ((const unsigned char*)data, (unsigned char*)data, (const aes_encrypt_ctx *)ks);
			break;
		case TWOFISH:
			twofish_encrypt ((TwofishInstance *)ks, (const unsigned int *)data, (unsigned int *)data);
			break;
		case SERPENT:
			serpent_encrypt ((const unsigned char *)data, (unsigned char *)data, (unsigned char *)ks);
			break;
		default:
			;//TC_THROW_FATAL_EXCEPTION;	// Unknown/wrong ID
    }
}

__device__ void cuDecipherBlock(int cipher, void *data, void *ks)
{
    switch (cipher)
    {
#ifndef TC_WINDOWS_BOOT
			
		case AES:
			aes_decrypt ((const unsigned char*)data, (unsigned char*)data, (const aes_decrypt_ctx *) ((char *) ks + sizeof(aes_decrypt_ctx)));
			break;
#else
		case AES:
			aes_decrypt ((unsigned char*)data, (unsigned char*)data, ((const aes_decrypt_ctx *))ks);
			break;
#endif
		case SERPENT:
			serpent_decrypt ((const unsigned char *)data, (unsigned char *)data, (unsigned char *)ks);
			break;
		case TWOFISH:
			twofish_decrypt ((TwofishInstance *)ks, (const unsigned int *)data, (unsigned int *)data);
			break;
		default:
			;//TC_THROW_FATAL_EXCEPTION;	// Unknown/wrong ID
    }
}


/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
#if BYTE_ORDER == BIG_ENDIAN
#error The TC_NO_COMPILER_INT64 version of the XTS code is not compatible with big-endian platforms
#endif
#include "Endian.h"

#if BYTE_ORDER == LITTLE_ENDIAN
#	define CUDA_BE16(x) cuda_MirrorBytes16(x)
#	define CUDA_BE32(x) cuda_MirrorBytes32(x)
#	define CUDA_BE64(x) cuda_MirrorBytes64(x)
#else
#	define CUDA_BE16(x) (x)
#	define CUDA_BE32(x) (x)
#	define CUDA_BE64(x) (x)
#endif

__device__ unsigned __int16 cuda_MirrorBytes16 (unsigned __int16 x)
{
	return (x << 8) | (x >> 8);
}


__device__ unsigned __int32 cuda_MirrorBytes32 (unsigned __int32 x)
{
	unsigned __int32 n = (unsigned __int8) x;
	n <<= 8; n |= (unsigned __int8) (x >> 8);
	n <<= 8; n |= (unsigned __int8) (x >> 16);
	return (n << 8) | (unsigned __int8) (x >> 24);
}

#define GetHeaderField16(header,offset) (CUDA_BE16(*(uint16*)(header+offset)))
#define GetHeaderField32(header,offset) (CUDA_BE32(*(uint32*)(header+offset)))

/* CRC polynomial 0x04c11db7 */
__constant__ unsigned __int32 cuda_crc_32_tab[]=
{
	0x00000000, 0x77073096, 0xee0e612c, 0x990951ba, 0x076dc419, 0x706af48f, 0xe963a535, 0x9e6495a3,
	0x0edb8832, 0x79dcb8a4, 0xe0d5e91e, 0x97d2d988, 0x09b64c2b, 0x7eb17cbd, 0xe7b82d07, 0x90bf1d91,
	0x1db71064, 0x6ab020f2, 0xf3b97148, 0x84be41de, 0x1adad47d, 0x6ddde4eb, 0xf4d4b551, 0x83d385c7,
	0x136c9856, 0x646ba8c0, 0xfd62f97a, 0x8a65c9ec, 0x14015c4f, 0x63066cd9, 0xfa0f3d63, 0x8d080df5,
	0x3b6e20c8, 0x4c69105e, 0xd56041e4, 0xa2677172, 0x3c03e4d1, 0x4b04d447, 0xd20d85fd, 0xa50ab56b,
	0x35b5a8fa, 0x42b2986c, 0xdbbbc9d6, 0xacbcf940, 0x32d86ce3, 0x45df5c75, 0xdcd60dcf, 0xabd13d59,
	0x26d930ac, 0x51de003a, 0xc8d75180, 0xbfd06116, 0x21b4f4b5, 0x56b3c423, 0xcfba9599, 0xb8bda50f,
	0x2802b89e, 0x5f058808, 0xc60cd9b2, 0xb10be924, 0x2f6f7c87, 0x58684c11, 0xc1611dab, 0xb6662d3d,
	0x76dc4190, 0x01db7106, 0x98d220bc, 0xefd5102a, 0x71b18589, 0x06b6b51f, 0x9fbfe4a5, 0xe8b8d433,
	0x7807c9a2, 0x0f00f934, 0x9609a88e, 0xe10e9818, 0x7f6a0dbb, 0x086d3d2d, 0x91646c97, 0xe6635c01,
	0x6b6b51f4, 0x1c6c6162, 0x856530d8, 0xf262004e, 0x6c0695ed, 0x1b01a57b, 0x8208f4c1, 0xf50fc457,
	0x65b0d9c6, 0x12b7e950, 0x8bbeb8ea, 0xfcb9887c, 0x62dd1ddf, 0x15da2d49, 0x8cd37cf3, 0xfbd44c65,
	0x4db26158, 0x3ab551ce, 0xa3bc0074, 0xd4bb30e2, 0x4adfa541, 0x3dd895d7, 0xa4d1c46d, 0xd3d6f4fb,
	0x4369e96a, 0x346ed9fc, 0xad678846, 0xda60b8d0, 0x44042d73, 0x33031de5, 0xaa0a4c5f, 0xdd0d7cc9,
	0x5005713c, 0x270241aa, 0xbe0b1010, 0xc90c2086, 0x5768b525, 0x206f85b3, 0xb966d409, 0xce61e49f,
	0x5edef90e, 0x29d9c998, 0xb0d09822, 0xc7d7a8b4, 0x59b33d17, 0x2eb40d81, 0xb7bd5c3b, 0xc0ba6cad,
	0xedb88320, 0x9abfb3b6, 0x03b6e20c, 0x74b1d29a, 0xead54739, 0x9dd277af, 0x04db2615, 0x73dc1683,
	0xe3630b12, 0x94643b84, 0x0d6d6a3e, 0x7a6a5aa8, 0xe40ecf0b, 0x9309ff9d, 0x0a00ae27, 0x7d079eb1,
	0xf00f9344, 0x8708a3d2, 0x1e01f268, 0x6906c2fe, 0xf762575d, 0x806567cb, 0x196c3671, 0x6e6b06e7,
	0xfed41b76, 0x89d32be0, 0x10da7a5a, 0x67dd4acc, 0xf9b9df6f, 0x8ebeeff9, 0x17b7be43, 0x60b08ed5,
	0xd6d6a3e8, 0xa1d1937e, 0x38d8c2c4, 0x4fdff252, 0xd1bb67f1, 0xa6bc5767, 0x3fb506dd, 0x48b2364b,
	0xd80d2bda, 0xaf0a1b4c, 0x36034af6, 0x41047a60, 0xdf60efc3, 0xa867df55, 0x316e8eef, 0x4669be79,
	0xcb61b38c, 0xbc66831a, 0x256fd2a0, 0x5268e236, 0xcc0c7795, 0xbb0b4703, 0x220216b9, 0x5505262f,
	0xc5ba3bbe, 0xb2bd0b28, 0x2bb45a92, 0x5cb36a04, 0xc2d7ffa7, 0xb5d0cf31, 0x2cd99e8b, 0x5bdeae1d,
	0x9b64c2b0, 0xec63f226, 0x756aa39c, 0x026d930a, 0x9c0906a9, 0xeb0e363f, 0x72076785, 0x05005713,
	0x95bf4a82, 0xe2b87a14, 0x7bb12bae, 0x0cb61b38, 0x92d28e9b, 0xe5d5be0d, 0x7cdcefb7, 0x0bdbdf21,
	0x86d3d2d4, 0xf1d4e242, 0x68ddb3f8, 0x1fda836e, 0x81be16cd, 0xf6b9265b, 0x6fb077e1, 0x18b74777,
	0x88085ae6, 0xff0f6a70, 0x66063bca, 0x11010b5c, 0x8f659eff, 0xf862ae69, 0x616bffd3, 0x166ccf45,
	0xa00ae278, 0xd70dd2ee, 0x4e048354, 0x3903b3c2, 0xa7672661, 0xd06016f7, 0x4969474d, 0x3e6e77db,
	0xaed16a4a, 0xd9d65adc, 0x40df0b66, 0x37d83bf0, 0xa9bcae53, 0xdebb9ec5, 0x47b2cf7f, 0x30b5ffe9,
	0xbdbdf21c, 0xcabac28a, 0x53b39330, 0x24b4a3a6, 0xbad03605, 0xcdd70693, 0x54de5729, 0x23d967bf,
	0xb3667a2e, 0xc4614ab8, 0x5d681b02, 0x2a6f2b94, 0xb40bbe37, 0xc30c8ea1, 0x5a05df1b, 0x2d02ef8d
};

__device__ unsigned __int32 cuGetCrc32 (unsigned char *data, int length)
{
	unsigned __int32 CRC = 0xffffffff;
	
	while (length--)
	{
		CRC = (CRC >> 8) ^ cuda_crc_32_tab[ (CRC ^ *data++) & 0xFF ];
	}
	
	return CRC ^ 0xffffffff;
}

__device__ void cuda_memcpy (unsigned char* to , unsigned char* from, int length){
	int i;
	for (i=0;i<length;i++)
		to[i]=from[i];
}




// Encrypts or decrypts all blocks in the buffer in XTS mode. For descriptions of the input parameters,
// see the 64-bit version of EncryptBufferXTS().
__device__ static void cuEncryptDecryptBufferXTS32 (const unsigned __int8 *buffer,
													   TC_LARGEST_COMPILER_UINT length,
													   const UINT64_STRUCT *startDataUnitNo,
													   unsigned int startBlock,
													   unsigned __int8 *ks,
													   unsigned __int8 *ks2,
													   int cipher,
													   BOOL decryption)
{
	
	__align__(8) unsigned __int8 byteBufUnitNo [BYTES_PER_XTS_BLOCK];
	__align__(8) unsigned __int8 whiteningValue [BYTES_PER_XTS_BLOCK];
	__align__(8) unsigned __int8 finalCarry;
	unsigned __int32 *whiteningValuePtr32;
	unsigned __int32 *finalDwordWhiteningValuePtr;
	unsigned __int32 *bufPtr32;
	
	TC_LARGEST_COMPILER_UINT blockCount;
	UINT64_STRUCT dataUnitNo;
	unsigned int block;
	unsigned int endBlock;
	
	
	bufPtr32 = (unsigned __int32 *) buffer;
	whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;
	finalDwordWhiteningValuePtr = whiteningValuePtr32 + sizeof (whiteningValue) / sizeof (*whiteningValuePtr32) - 1;
	
	
	// Store the 64-bit data unit number in a way compatible with non-64-bit environments/platforms
	dataUnitNo.HighPart = startDataUnitNo->HighPart;
	dataUnitNo.LowPart = startDataUnitNo->LowPart;
	
	blockCount = length / BYTES_PER_XTS_BLOCK;
	
	// Convert the 64-bit data unit number into a little-endian 16-byte array.
	// (Passed as two 32-bit integers for compatibility with non-64-bit environments/platforms.)
	cuUint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);
	
	// Generate whitening values for all blocks in the buffer
	while (blockCount > 0)
	{
		
		
		if (blockCount < BLOCKS_PER_XTS_DATA_UNIT)
			endBlock = startBlock + (unsigned int) blockCount;
		else
			endBlock = BLOCKS_PER_XTS_DATA_UNIT;
		
		
		// Encrypt the data unit number using the secondary key (in order to generate the first
		// whitening value for this data unit)
		cuUint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);
		cuda_memcpy (whiteningValue, byteBufUnitNo, BYTES_PER_XTS_BLOCK);
		cuEncipherBlock (cipher, whiteningValue, ks2);
		
		// Generate (and apply) subsequent whitening values for blocks in this data unit and
		// encrypt/decrypt all relevant blocks in this data unit
		for (block = 0; block < endBlock; block++)
		{
			if (block >= startBlock)
			{
				whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;
				
				// Whitening
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32 ^= *whiteningValuePtr32;
				
				bufPtr32 -= BYTES_PER_XTS_BLOCK / sizeof (*bufPtr32) - 1;
				
				// Actual encryption/decryption
				if (decryption)
					cuDecipherBlock (cipher, bufPtr32, ks);
				else
					cuEncipherBlock (cipher, bufPtr32, ks);
				
				whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;
				
				// Whitening
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32;
			}
			
			// Derive the next whitening value
			
			finalCarry = 0;
			
			for (whiteningValuePtr32 = finalDwordWhiteningValuePtr;
				 whiteningValuePtr32 >= (unsigned __int32 *) whiteningValue;
				 whiteningValuePtr32--)
			{
				if (*whiteningValuePtr32 & 0x80000000)	// If the following shift results in a carry
				{
					if (whiteningValuePtr32 != finalDwordWhiteningValuePtr)	// If not processing the highest double word
					{
						// A regular carry
						*(whiteningValuePtr32 + 1) |= 1;
					}
					else
					{
						// The highest byte shift will result in a carry
						finalCarry = 135;
					}
				}
				
				*whiteningValuePtr32 <<= 1;
			}
			
			whiteningValue[0] ^= finalCarry;
		}
		
		blockCount -= endBlock - startBlock;
		startBlock = 0;
		
		// Increase the data unit number by one
		if (!++dataUnitNo.LowPart)
		{
			dataUnitNo.HighPart++;
		}
		
		// Convert the 64-bit data unit number into a little-endian 16-byte array.
		cuUint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);
	}
	
	FAST_ERASE64 (whiteningValue, sizeof (whiteningValue));
}


// For descriptions of the input parameters, see the 64-bit version of EncryptBufferXTS().
__device__ void cuDecryptBufferXTS (unsigned __int8 *buffer,
									   TC_LARGEST_COMPILER_UINT length,
									   const UINT64_STRUCT *startDataUnitNo,
									   unsigned int startCipherBlockNo,
									   unsigned __int8 *ks,
									   unsigned __int8 *ks2,
									   int cipher)
{
	// Decrypt all ciphertext blocks in the buffer
	cuEncryptDecryptBufferXTS32 (buffer, length, startDataUnitNo, startCipherBlockNo, ks, ks2, cipher, TRUE);
}

__device__ void cuDecryptBuffer (unsigned __int8 *buf, TC_LARGEST_COMPILER_UINT len, PCRYPTO_INFO cryptoInfo)
{
	//unsigned __int8 *ks = cryptoInfo->ks;  //+ EAGetKeyScheduleSize (cryptoInfo->ea);
	//unsigned __int8 *ks2 = cryptoInfo->ks2;// + EAGetKeyScheduleSize (cryptoInfo->ea);
	UINT64_STRUCT dataUnitNo;
	//int cipher;
	
	// When encrypting/decrypting a buffer (typically a volume header) the sequential number
	// of the first XTS data unit in the buffer is always 0 and the start of the buffer is
	// always assumed to be aligned with the start of the data unit 0.
	dataUnitNo.LowPart = 0;
	dataUnitNo.HighPart = 0;
	
	//	for (cipher = EAGetLastCipher (cryptoInfo->ea);
	//		cipher != 0;
	//		cipher = EAGetPreviousCipher (cryptoInfo->ea, cipher))
	//	{
	//		ks -= CipherGetKeyScheduleSize (cipher);
	//		ks2 -= CipherGetKeyScheduleSize (cipher);
	cuDecryptBufferXTS (buf, len, &dataUnitNo, 0, cryptoInfo->ks, cryptoInfo->ks2, cryptoInfo->ea);
	//	}
}



__device__ int cuXts(int encryptionAlgorithm, unsigned char *encryptedHeader, unsigned char *headerKey, unsigned char *header) {
	
    PCRYPTO_INFO cryptoInfo;
    CRYPTO_INFO cryptoInfo_struct;
	
    uint16 headerVersion;
    int status = ERR_PARAMETER_INCORRECT;
    int primaryKeyOffset=0;
	int eaGetKeySize=32; 
	
    //int pkcs5PrfCount = LAST_PRF_ID - FIRST_PRF_ID + 1;
	
    cryptoInfo=&cryptoInfo_struct;    
    if (cryptoInfo == NULL)
        return ERR_OUT_OF_MEMORY;
    memset (cryptoInfo, 0, sizeof (CRYPTO_INFO));

    // Init objects related to the mode of operation
	// Support only XTS
    cryptoInfo->mode= XTS ;
	if (encryptionAlgorithm!=AES && encryptionAlgorithm!=SERPENT && encryptionAlgorithm!=TWOFISH)
		return UNDEFINED;
    cryptoInfo->ea=encryptionAlgorithm;
	
	// Primary key schedule
	cuda_memcpy (cryptoInfo->k2, headerKey + primaryKeyOffset, 64);
	status = cuCipherInit (cryptoInfo->ea, cryptoInfo->k2, cryptoInfo->ks);
    if (status != ERR_SUCCESS)
        return ERR_CIPHER_INIT;
        
    // Secondary key schedule
    cuda_memcpy (cryptoInfo->k2, headerKey + eaGetKeySize, eaGetKeySize);
	status = cuCipherInit (cryptoInfo->ea, cryptoInfo->k2, cryptoInfo->ks2);
    if (status != ERR_SUCCESS)
        return ERR_MODE_INIT;
    
 
    // Copy the header for decryption
    cuda_memcpy (header, encryptedHeader, 512*sizeof(unsigned char));
	
    // Try to decrypt header
    cuDecryptBuffer (header + HEADER_ENCRYPTED_DATA_OFFSET, HEADER_ENCRYPTED_DATA_SIZE, cryptoInfo);
	    
	// Magic 'TRUE'
	if (GetHeaderField32 (header, TC_HEADER_OFFSET_MAGIC) != 0x54525545)
		return ERR_MAGIC_TRUE;
	
	// Header version
	headerVersion = GetHeaderField16 (header, TC_HEADER_OFFSET_VERSION);
	if (headerVersion > VOLUME_HEADER_VERSION) {
		return ERR_VERSION_REQUIRED;
	}
	
	// Check CRC of the header fields
	if (headerVersion >= 4
		&& GetHeaderField32 (header, TC_HEADER_OFFSET_HEADER_CRC) != cuGetCrc32 (header + TC_HEADER_OFFSET_MAGIC, TC_HEADER_OFFSET_HEADER_CRC - TC_HEADER_OFFSET_MAGIC))
		//printf("Unsuccessful\n");
		return ERR_CRC_HEADER_FIELDS;
	// Required program version
	//cryptoInfo->RequiredProgramVersion = GetHeaderField16 (header, TC_HEADER_OFFSET_REQUIRED_VERSION);
	//cryptoInfo->LegacyVolume = cryptoInfo->RequiredProgramVersion < 0x600;
	
	// Check CRC of the key set
	if (GetHeaderField32 (header, TC_HEADER_OFFSET_KEY_AREA_CRC) != cuGetCrc32 (header + HEADER_MASTER_KEYDATA_OFFSET, MASTER_KEYDATA_SIZE))
		return ERR_CRC_KEY_SET;

    return SUCCESS;
}



/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
// RIPEMD-160 written and placed in the public domain by Wei Dai

/*
 * This code implements the MD4 message-digest algorithm.
 * The algorithm is due to Ron Rivest.  This code was
 * written by Colin Plumb in 1993, no copyright is claimed.
 * This code is in the public domain; do with it what you wish.
 */

/* Adapted for TrueCrypt */

#include <memory.h>
#include "Tcdefs.h"
#include "Endian.h"
#include "Rmd160.alternative.cuh"

#define F(x, y, z)    (x ^ y ^ z) 
#define G(x, y, z)    (z ^ (x & (y^z)))
#define H(x, y, z)    (z ^ (x | ~y))
#define I(x, y, z)    (y ^ (z & (x^y)))
#define J(x, y, z)    (x ^ (y | ~z))

#define PUT_64BIT_LE(cp, value) do {                                    \
	(cp)[7] = (byte) ((value) >> 56);                                        \
	(cp)[6] = (byte) ((value) >> 48);                                        \
	(cp)[5] = (byte) ((value) >> 40);                                        \
	(cp)[4] = (byte) ((value) >> 32);                                        \
	(cp)[3] = (byte) ((value) >> 24);                                        \
	(cp)[2] = (byte) ((value) >> 16);                                        \
	(cp)[1] = (byte) ((value) >> 8);                                         \
	(cp)[0] = (byte) (value); } while (0)

#define PUT_32BIT_LE(cp, value) do {                                    \
	(cp)[3] = (byte) ((value) >> 24);                                        \
	(cp)[2] = (byte) ((value) >> 16);                                        \
	(cp)[1] = (byte) ((value) >> 8);                                         \
	(cp)[0] = (byte) (value); } while (0)

#define word32 unsigned __int32

#define k0 0
#define k1 0x5a827999UL
#define k2 0x6ed9eba1UL
#define k3 0x8f1bbcdcUL
#define k4 0xa953fd4eUL
#define k5 0x50a28be6UL
#define k6 0x5c4dd124UL
#define k7 0x6d703ef3UL
#define k8 0x7a6d76e9UL
#define k9 0

#define  rrotlFixed( x, y) (word32)((x<<y) | (x>>(sizeof(word32)*8-y)))
  
//__device__ word32 rrotlFixed (word32 x, unsigned int y)
//{ 
//	return (word32)((x<<y) | (x>>(sizeof(word32)*8-y)));
//}

#define Subround(f, a, b, c, d, e, x, s, k)        \
	a += f(b, c, d) + x + k;\
	a = rrotlFixed((word32)a, s) + e;\
	c = rrotlFixed((word32)c, 10U)


/*
static byte PADDING[64]= {
	0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};
*/

__device__ void cuda_RMD160 (RMD160_CTX *ctx, const unsigned char *input1, unsigned __int32 lenArg1, const unsigned char *input2, unsigned __int32 lenArg2, unsigned char *digest){
	//global variable of the subset of functions
	uint32 padlen;
	byte count[8];
	byte PADDING[64];
	unsigned int update2_flags;
	
	if (input2==NULL || lenArg2==0)
	  update2_flags=FALSE;
	else
	  update2_flags=TRUE;
	
	// INCLUDE: void RMD160Init (RMD160_CTX *ctx)
	{
	int i;
	ctx->count = 0;
	ctx->state[0] = 0x67452301;
	ctx->state[1] = 0xefcdab89;
	ctx->state[2] = 0x98badcfe;
	ctx->state[3] = 0x10325476;
	ctx->state[4] = 0xc3d2e1f0;

	for (i=0;i<64;i++)
		PADDING[i]=0;
	PADDING[0] = 0x80;
	}
	// ENDINCLUDE: void RMD160Init (RMD160_CTX *ctx)
	
	
	// INCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input1, unsigned __int32 lenArg1)
	{
#ifndef TC_WINDOWS_BOOT
	uint32 len = lenArg1, have, need;
#else
	uint16 len = (uint16) lenArg1, have, need;
#endif
	/* Check how many bytes we already have and how many more we need. */
	have = ((ctx->count >> 3) & (RIPEMD160_BLOCK_LENGTH - 1));
	need = RIPEMD160_BLOCK_LENGTH - have;
	/* Update bitcount */
	ctx->count += len << 3;
	if (len >= need) {
		if (have != 0) {
			memcpy (ctx->buffer + have, input1, (size_t) need);
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) ctx->buffer);
			input1 += need;
			len -= need;
			have = 0;
		}
		// Process data in RIPEMD160_BLOCK_LENGTH-byte chunks. 
		while (len >= RIPEMD160_BLOCK_LENGTH) {
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) input1);
			input1 += RIPEMD160_BLOCK_LENGTH;
			len -= RIPEMD160_BLOCK_LENGTH;
		}	  
	}
	/* Handle any remaining bytes of data. */
	if (len != 0)
		memcpy (ctx->buffer + have, input1, (size_t) len);
	}
	// ENDINCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input1, unsigned __int32 lenArg1)
	
	if (update2_flags==TRUE)
	// INCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input2, unsigned __int32 lenArg2)
	{
#ifndef TC_WINDOWS_BOOT
	uint32 len = lenArg2, have, need;
#else
	uint16 len = (uint16) lenArg2, have, need;
#endif
	/* Check how many bytes we already have and how many more we need. */
	have = ((ctx->count >> 3) & (RIPEMD160_BLOCK_LENGTH - 1));
	need = RIPEMD160_BLOCK_LENGTH - have;
	/* Update bitcount */
	ctx->count += len << 3;
	if (len >= need) {
		if (have != 0) {
			memcpy (ctx->buffer + have, input2, (size_t) need);
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) ctx->buffer);
			input2 += need;
			len -= need;
			have = 0;
		}
		// Process data in RIPEMD160_BLOCK_LENGTH-byte chunks. 
		while (len >= RIPEMD160_BLOCK_LENGTH) {
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) input2);
			input2 += RIPEMD160_BLOCK_LENGTH;
			len -= RIPEMD160_BLOCK_LENGTH;
		}	  
	}
	/* Handle any remaining bytes of data. */
	if (len != 0)
		memcpy (ctx->buffer + have, input2, (size_t) len);
	}// ENDINCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input2, unsigned __int32 lenArg2)
	
	
	// INCLUDE: void RMD160Pad(RMD160_CTX *ctx)
	{
	/* Convert count to 8 bytes in little endian order. */
#ifndef TC_WINDOWS_BOOT
	PUT_64BIT_LE(count, ctx->count);
#else
	*(uint32 *) (count + 4) = 0;
	*(uint16 *) (count + 2) = 0;
	*(uint16 *) (count + 0) = ctx->count;
#endif
	/* Pad out to 56 mod 64. */
	padlen = RIPEMD160_BLOCK_LENGTH -
		(uint32)((ctx->count >> 3) & (RIPEMD160_BLOCK_LENGTH - 1));
	if (padlen < 1 + 8)
		padlen += RIPEMD160_BLOCK_LENGTH;
	}
	// ENDINCLUDE: void RMD160Pad(RMD160_CTX *ctx)
	
		
	
	
	// INCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input3, unsigned __int32 lenArg3) 
	// Call RMD160Update(ctx, PADDING, padlen - 8);
	{
	  unsigned char *input3;
	  input3=(unsigned char *)PADDING;
	  unsigned __int32 lenArg3=padlen-8;
#ifndef TC_WINDOWS_BOOT
	uint32 len = lenArg3, have, need;
#else
	uint16 len = (uint16) lenArg3, have, need;
#endif
	/* Check how many bytes we already have and how many more we need. */
	have = ((ctx->count >> 3) & (RIPEMD160_BLOCK_LENGTH - 1));
	need = RIPEMD160_BLOCK_LENGTH - have;
	/* Update bitcount */
	ctx->count += len << 3;
	if (len >= need) {
		if (have != 0) {
			memcpy (ctx->buffer + have, input3, (size_t) need);
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) ctx->buffer);
			input3 += need;
			len -= need;
			have = 0;
		}
		// Process data in RIPEMD160_BLOCK_LENGTH-byte chunks. 
		while (len >= RIPEMD160_BLOCK_LENGTH) {
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) input3);
			input3 += RIPEMD160_BLOCK_LENGTH;
			len -= RIPEMD160_BLOCK_LENGTH;
		}	  
	}
	/* Handle any remaining bytes of data. */
	if (len != 0)
		memcpy (ctx->buffer + have, input3, (size_t) len);
	}// ENDINCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input3, unsigned __int32 lenArg3)
	

	
	// INCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input4, unsigned __int32 lenArg4) 
	// Call RMD160Update(ctx, count, 8);
	{
	  unsigned char *input4;
	  input4=(unsigned char *)count;
	  unsigned __int32 lenArg4=8;
#ifndef TC_WINDOWS_BOOT
	uint32 len = lenArg4, have, need;
#else
	uint16 len = (uint16) lenArg4, have, need;
#endif
	/* Check how many bytes we already have and how many more we need. */
	have = ((ctx->count >> 3) & (RIPEMD160_BLOCK_LENGTH - 1));
	need = RIPEMD160_BLOCK_LENGTH - have;
	/* Update bitcount */
	ctx->count += len << 3;
	if (len >= need) {
		if (have != 0) {
			memcpy (ctx->buffer + have, input4, (size_t) need);
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) ctx->buffer);
			input4 += need;
			len -= need;
			have = 0;
		}
		// Process data in RIPEMD160_BLOCK_LENGTH-byte chunks. 
		while (len >= RIPEMD160_BLOCK_LENGTH) {
			cuda_RMD160Transform ((uint32 *) ctx->state, (const uint32 *) input4);
			input4 += RIPEMD160_BLOCK_LENGTH;
			len -= RIPEMD160_BLOCK_LENGTH;
		}	  
	}
	/* Handle any remaining bytes of data. */
	if (len != 0)
		memcpy (ctx->buffer + have, input4, (size_t) len);
	}// ENDINCLUDE: void RMD160Update (RMD160_CTX *ctx, const unsigned char *input4, unsigned __int32 lenArg4) 
		
		
		
	
	// INCLUDE: RMD160Final(unsigned char *digest, RMD160_CTX *ctx)	
	int i;
	if (digest) {
		for (i = 0; i < 5; i++)
			PUT_32BIT_LE(digest + i * 4, ctx->state[i]);
		memset (ctx, 0, sizeof(*ctx));
	}	
	// ENDINCLUDE: RMD160Final(unsigned char *digest, RMD160_CTX *ctx)
}



__device__ void cuda_RMD160Transform (unsigned __int32 *digest, const unsigned __int32 *data)
{
 
#if BYTE_ORDER == LITTLE_ENDIAN
	const unsigned __int32 *X = data;
#else
	unsigned __int32 X[16];
	int i;
#endif

	unsigned __int32 a1, b1, c1, d1, e1, a2, b2, c2, d2, e2;
	
	a1 = a2 = digest[0];
	b1 = b2 = digest[1];
	c1 = c2 = digest[2];
	d1 = d2 = digest[3];
	e1 = e2 = digest[4];
	

#if BYTE_ORDER == BIG_ENDIAN
	for (i = 0; i < 16; i++)
	{
		X[i] = LE32 (data[i]);
	}
#endif

	Subround(F, a1, b1, c1, d1, e1, X[ 0], 11, k0);
	Subround(F, e1, a1, b1, c1, d1, X[ 1], 14, k0);
	Subround(F, d1, e1, a1, b1, c1, X[ 2], 15, k0);
	Subround(F, c1, d1, e1, a1, b1, X[ 3], 12, k0);
	Subround(F, b1, c1, d1, e1, a1, X[ 4],  5, k0);
	Subround(F, a1, b1, c1, d1, e1, X[ 5],  8, k0);
	Subround(F, e1, a1, b1, c1, d1, X[ 6],  7, k0);
	Subround(F, d1, e1, a1, b1, c1, X[ 7],  9, k0);
	Subround(F, c1, d1, e1, a1, b1, X[ 8], 11, k0);
	Subround(F, b1, c1, d1, e1, a1, X[ 9], 13, k0);
	Subround(F, a1, b1, c1, d1, e1, X[10], 14, k0);
	Subround(F, e1, a1, b1, c1, d1, X[11], 15, k0);
	Subround(F, d1, e1, a1, b1, c1, X[12],  6, k0);
	Subround(F, c1, d1, e1, a1, b1, X[13],  7, k0);
	Subround(F, b1, c1, d1, e1, a1, X[14],  9, k0);
	Subround(F, a1, b1, c1, d1, e1, X[15],  8, k0);

	Subround(G, e1, a1, b1, c1, d1, X[ 7],  7, k1);
	Subround(G, d1, e1, a1, b1, c1, X[ 4],  6, k1);
	Subround(G, c1, d1, e1, a1, b1, X[13],  8, k1);
	Subround(G, b1, c1, d1, e1, a1, X[ 1], 13, k1);
	Subround(G, a1, b1, c1, d1, e1, X[10], 11, k1);
	Subround(G, e1, a1, b1, c1, d1, X[ 6],  9, k1);
	Subround(G, d1, e1, a1, b1, c1, X[15],  7, k1);
	Subround(G, c1, d1, e1, a1, b1, X[ 3], 15, k1);
	Subround(G, b1, c1, d1, e1, a1, X[12],  7, k1);
	Subround(G, a1, b1, c1, d1, e1, X[ 0], 12, k1);
	Subround(G, e1, a1, b1, c1, d1, X[ 9], 15, k1);
	Subround(G, d1, e1, a1, b1, c1, X[ 5],  9, k1);
	Subround(G, c1, d1, e1, a1, b1, X[ 2], 11, k1);
	Subround(G, b1, c1, d1, e1, a1, X[14],  7, k1);
	Subround(G, a1, b1, c1, d1, e1, X[11], 13, k1);
	Subround(G, e1, a1, b1, c1, d1, X[ 8], 12, k1);

	Subround(H, d1, e1, a1, b1, c1, X[ 3], 11, k2);
	Subround(H, c1, d1, e1, a1, b1, X[10], 13, k2);
	Subround(H, b1, c1, d1, e1, a1, X[14],  6, k2);
	Subround(H, a1, b1, c1, d1, e1, X[ 4],  7, k2);
	Subround(H, e1, a1, b1, c1, d1, X[ 9], 14, k2);
	Subround(H, d1, e1, a1, b1, c1, X[15],  9, k2);
	Subround(H, c1, d1, e1, a1, b1, X[ 8], 13, k2);
	Subround(H, b1, c1, d1, e1, a1, X[ 1], 15, k2);
	Subround(H, a1, b1, c1, d1, e1, X[ 2], 14, k2);
	Subround(H, e1, a1, b1, c1, d1, X[ 7],  8, k2);
	Subround(H, d1, e1, a1, b1, c1, X[ 0], 13, k2);
	Subround(H, c1, d1, e1, a1, b1, X[ 6],  6, k2);
	Subround(H, b1, c1, d1, e1, a1, X[13],  5, k2);
	Subround(H, a1, b1, c1, d1, e1, X[11], 12, k2);
	Subround(H, e1, a1, b1, c1, d1, X[ 5],  7, k2);
	Subround(H, d1, e1, a1, b1, c1, X[12],  5, k2);

	Subround(I, c1, d1, e1, a1, b1, X[ 1], 11, k3);
	Subround(I, b1, c1, d1, e1, a1, X[ 9], 12, k3);
	Subround(I, a1, b1, c1, d1, e1, X[11], 14, k3);
	Subround(I, e1, a1, b1, c1, d1, X[10], 15, k3);
	Subround(I, d1, e1, a1, b1, c1, X[ 0], 14, k3);
	Subround(I, c1, d1, e1, a1, b1, X[ 8], 15, k3);
	Subround(I, b1, c1, d1, e1, a1, X[12],  9, k3);
	Subround(I, a1, b1, c1, d1, e1, X[ 4],  8, k3);
	Subround(I, e1, a1, b1, c1, d1, X[13],  9, k3);
	Subround(I, d1, e1, a1, b1, c1, X[ 3], 14, k3);
	Subround(I, c1, d1, e1, a1, b1, X[ 7],  5, k3);
	Subround(I, b1, c1, d1, e1, a1, X[15],  6, k3);
	Subround(I, a1, b1, c1, d1, e1, X[14],  8, k3);
	Subround(I, e1, a1, b1, c1, d1, X[ 5],  6, k3);
	Subround(I, d1, e1, a1, b1, c1, X[ 6],  5, k3);
	Subround(I, c1, d1, e1, a1, b1, X[ 2], 12, k3);

	Subround(J, b1, c1, d1, e1, a1, X[ 4],  9, k4);
	Subround(J, a1, b1, c1, d1, e1, X[ 0], 15, k4);
	Subround(J, e1, a1, b1, c1, d1, X[ 5],  5, k4);
	Subround(J, d1, e1, a1, b1, c1, X[ 9], 11, k4);
	Subround(J, c1, d1, e1, a1, b1, X[ 7],  6, k4);
	Subround(J, b1, c1, d1, e1, a1, X[12],  8, k4);
	Subround(J, a1, b1, c1, d1, e1, X[ 2], 13, k4);
	Subround(J, e1, a1, b1, c1, d1, X[10], 12, k4);
	Subround(J, d1, e1, a1, b1, c1, X[14],  5, k4);
	Subround(J, c1, d1, e1, a1, b1, X[ 1], 12, k4);
	Subround(J, b1, c1, d1, e1, a1, X[ 3], 13, k4);
	Subround(J, a1, b1, c1, d1, e1, X[ 8], 14, k4);
	Subround(J, e1, a1, b1, c1, d1, X[11], 11, k4);
	Subround(J, d1, e1, a1, b1, c1, X[ 6],  8, k4);
	Subround(J, c1, d1, e1, a1, b1, X[15],  5, k4);
	Subround(J, b1, c1, d1, e1, a1, X[13],  6, k4);

	Subround(J, a2, b2, c2, d2, e2, X[ 5],  8, k5);
	Subround(J, e2, a2, b2, c2, d2, X[14],  9, k5);
	Subround(J, d2, e2, a2, b2, c2, X[ 7],  9, k5);
	Subround(J, c2, d2, e2, a2, b2, X[ 0], 11, k5);
	Subround(J, b2, c2, d2, e2, a2, X[ 9], 13, k5);
	Subround(J, a2, b2, c2, d2, e2, X[ 2], 15, k5);
	Subround(J, e2, a2, b2, c2, d2, X[11], 15, k5);
	Subround(J, d2, e2, a2, b2, c2, X[ 4],  5, k5);
	Subround(J, c2, d2, e2, a2, b2, X[13],  7, k5);
	Subround(J, b2, c2, d2, e2, a2, X[ 6],  7, k5);
	Subround(J, a2, b2, c2, d2, e2, X[15],  8, k5);
	Subround(J, e2, a2, b2, c2, d2, X[ 8], 11, k5);
	Subround(J, d2, e2, a2, b2, c2, X[ 1], 14, k5);
	Subround(J, c2, d2, e2, a2, b2, X[10], 14, k5);
	Subround(J, b2, c2, d2, e2, a2, X[ 3], 12, k5);
	Subround(J, a2, b2, c2, d2, e2, X[12],  6, k5);

	Subround(I, e2, a2, b2, c2, d2, X[ 6],  9, k6); 
	Subround(I, d2, e2, a2, b2, c2, X[11], 13, k6);
	Subround(I, c2, d2, e2, a2, b2, X[ 3], 15, k6);
	Subround(I, b2, c2, d2, e2, a2, X[ 7],  7, k6);
	Subround(I, a2, b2, c2, d2, e2, X[ 0], 12, k6);
	Subround(I, e2, a2, b2, c2, d2, X[13],  8, k6);
	Subround(I, d2, e2, a2, b2, c2, X[ 5],  9, k6);
	Subround(I, c2, d2, e2, a2, b2, X[10], 11, k6);
	Subround(I, b2, c2, d2, e2, a2, X[14],  7, k6);
	Subround(I, a2, b2, c2, d2, e2, X[15],  7, k6);
	Subround(I, e2, a2, b2, c2, d2, X[ 8], 12, k6);
	Subround(I, d2, e2, a2, b2, c2, X[12],  7, k6);
	Subround(I, c2, d2, e2, a2, b2, X[ 4],  6, k6);
	Subround(I, b2, c2, d2, e2, a2, X[ 9], 15, k6);
	Subround(I, a2, b2, c2, d2, e2, X[ 1], 13, k6);
	Subround(I, e2, a2, b2, c2, d2, X[ 2], 11, k6);

	Subround(H, d2, e2, a2, b2, c2, X[15],  9, k7);
	Subround(H, c2, d2, e2, a2, b2, X[ 5],  7, k7);
	Subround(H, b2, c2, d2, e2, a2, X[ 1], 15, k7);
	Subround(H, a2, b2, c2, d2, e2, X[ 3], 11, k7);
	Subround(H, e2, a2, b2, c2, d2, X[ 7],  8, k7);
	Subround(H, d2, e2, a2, b2, c2, X[14],  6, k7);
	Subround(H, c2, d2, e2, a2, b2, X[ 6],  6, k7);
	Subround(H, b2, c2, d2, e2, a2, X[ 9], 14, k7);
	Subround(H, a2, b2, c2, d2, e2, X[11], 12, k7);
	Subround(H, e2, a2, b2, c2, d2, X[ 8], 13, k7);
	Subround(H, d2, e2, a2, b2, c2, X[12],  5, k7);
	Subround(H, c2, d2, e2, a2, b2, X[ 2], 14, k7);
	Subround(H, b2, c2, d2, e2, a2, X[10], 13, k7);
	Subround(H, a2, b2, c2, d2, e2, X[ 0], 13, k7);
	Subround(H, e2, a2, b2, c2, d2, X[ 4],  7, k7);
	Subround(H, d2, e2, a2, b2, c2, X[13],  5, k7);

	Subround(G, c2, d2, e2, a2, b2, X[ 8], 15, k8);
	Subround(G, b2, c2, d2, e2, a2, X[ 6],  5, k8);
	Subround(G, a2, b2, c2, d2, e2, X[ 4],  8, k8);
	Subround(G, e2, a2, b2, c2, d2, X[ 1], 11, k8);
	Subround(G, d2, e2, a2, b2, c2, X[ 3], 14, k8);
	Subround(G, c2, d2, e2, a2, b2, X[11], 14, k8);
	Subround(G, b2, c2, d2, e2, a2, X[15],  6, k8);
	Subround(G, a2, b2, c2, d2, e2, X[ 0], 14, k8);
	Subround(G, e2, a2, b2, c2, d2, X[ 5],  6, k8);
	Subround(G, d2, e2, a2, b2, c2, X[12],  9, k8);
	Subround(G, c2, d2, e2, a2, b2, X[ 2], 12, k8);
	Subround(G, b2, c2, d2, e2, a2, X[13],  9, k8);
	Subround(G, a2, b2, c2, d2, e2, X[ 9], 12, k8);
	Subround(G, e2, a2, b2, c2, d2, X[ 7],  5, k8);
	Subround(G, d2, e2, a2, b2, c2, X[10], 15, k8);
	Subround(G, c2, d2, e2, a2, b2, X[14],  8, k8);

	Subround(F, b2, c2, d2, e2, a2, X[12],  8, k9);
	Subround(F, a2, b2, c2, d2, e2, X[15],  5, k9);
	Subround(F, e2, a2, b2, c2, d2, X[10], 12, k9);
	Subround(F, d2, e2, a2, b2, c2, X[ 4],  9, k9);
	Subround(F, c2, d2, e2, a2, b2, X[ 1], 12, k9);
	Subround(F, b2, c2, d2, e2, a2, X[ 5],  5, k9);
	Subround(F, a2, b2, c2, d2, e2, X[ 8], 14, k9);
	Subround(F, e2, a2, b2, c2, d2, X[ 7],  6, k9);
	Subround(F, d2, e2, a2, b2, c2, X[ 6],  8, k9);
	Subround(F, c2, d2, e2, a2, b2, X[ 2], 13, k9);
	Subround(F, b2, c2, d2, e2, a2, X[13],  6, k9);
	Subround(F, a2, b2, c2, d2, e2, X[14],  5, k9);
	Subround(F, e2, a2, b2, c2, d2, X[ 0], 15, k9);
	Subround(F, d2, e2, a2, b2, c2, X[ 3], 13, k9);
	Subround(F, c2, d2, e2, a2, b2, X[ 9], 11, k9);
	Subround(F, b2, c2, d2, e2, a2, X[11], 11, k9);

	c1        = digest[1] + c1 + d2;
	digest[1] = digest[2] + d1 + e2;
	digest[2] = digest[3] + e1 + a2;
	digest[3] = digest[4] + a1 + b2;
	digest[4] = digest[0] + b1 + c2;
	digest[0] = c1;
	
}
/*
 ---------------------------------------------------------------------------
 Copyright (c) 2002, Dr Brian Gladman, Worcester, UK.   All rights reserved.

 LICENSE TERMS

 The free distribution and use of this software is allowed (with or without
 changes) provided that:

  1. source code distributions include the above copyright notice, this
     list of conditions and the following disclaimer;

  2. binary distributions include the above copyright notice, this list
     of conditions and the following disclaimer in their documentation;

  3. the name of the copyright holder is not used to endorse products
     built using this software without specific written permission.

 DISCLAIMER

 This software is provided 'as is' with no explicit or implied warranties
 in respect of its properties, including, but not limited to, correctness
 and/or fitness for purpose.
 ---------------------------------------------------------------------------
 Issue Date: 01/08/2005

 This is a byte oriented version of SHA2 that operates on arrays of bytes
 stored in memory. This code implements sha256, sha384 and sha512 but the
 latter two functions rely on efficient 64-bit integer operations that
 may not be very efficient on 32-bit machines

 The sha256 functions use a type 'sha256_ctx' to hold details of the
 current hash state and uses the following three calls:

       void sha256_begin(sha256_ctx ctx[1])
       void sha256_hash(const unsigned char data[],
                            unsigned long len, sha256_ctx ctx[1])
       void sha_end1(unsigned char hval[], sha256_ctx ctx[1])

 The first subroutine initialises a hash computation by setting up the
 context in the sha256_ctx context. The second subroutine hashes 8-bit
 bytes from array data[] into the hash state withinh sha256_ctx context,
 the number of bytes to be hashed being given by the the unsigned long
 integer len.  The third subroutine completes the hash calculation and
 places the resulting digest value in the array of 8-bit bytes hval[].

 The sha384 and sha512 functions are similar and use the interfaces:

       void sha384_begin(sha384_ctx ctx[1]);
       void sha384_hash(const unsigned char data[],
                            unsigned long len, sha384_ctx ctx[1]);
       void sha384_end(unsigned char hval[], sha384_ctx ctx[1]);

       void sha512_begin(sha512_ctx ctx[1]);
       void sha512_hash(const unsigned char data[],
                            unsigned long len, sha512_ctx ctx[1]);
       void sha512_end(unsigned char hval[], sha512_ctx ctx[1]);

 In addition there is a function sha2 that can be used to call all these
 functions using a call with a hash length parameter as follows:

       int sha2_begin(unsigned long len, sha2_ctx ctx[1]);
       void sha2_hash(const unsigned char data[],
                            unsigned long len, sha2_ctx ctx[1]);
       void sha2_end(unsigned char hval[], sha2_ctx ctx[1]);

 My thanks to Erik Andersen <andersen@codepoet.org> for testing this code
 on big-endian systems and for his assistance with corrections
*/

#include "Common/Endian.h"
#ifndef PLATFORM_BYTE_ORDER
#define PLATFORM_BYTE_ORDER BYTE_ORDER
#endif
#ifndef IS_LITTLE_ENDIAN
#define IS_LITTLE_ENDIAN LITTLE_ENDIAN
#endif

#if 0
#define UNROLL_SHA2     /* for SHA2 loop unroll     */
#endif

#include <string.h>     /* for memcpy() etc.        */

#include "Sha2.cuh"

#if defined(__cplusplus)
extern "C"
{
#endif

#if defined( _MSC_VER ) && ( _MSC_VER > 800 )
#pragma intrinsic(memcpy)
#endif

#if 0 && defined(_MSC_VER)
#define rotl32 _lrotl
#define rotr32 _lrotr
#else
#define rotl32(x,n)   (((x) << n) | ((x) >> (32 - n)))
#define rotr32(x,n)   (((x) >> n) | ((x) << (32 - n)))
#endif

#if !defined(bswap_32)
#define bswap_32(x) ((rotr32((x), 24) & 0x00ff00ff) | (rotr32((x), 8) & 0xff00ff00))
#endif

#if (PLATFORM_BYTE_ORDER == IS_LITTLE_ENDIAN)
#define SWAP_BYTES
#else
#undef  SWAP_BYTES
#endif

#if 0

#define ch(x,y,z)       (((x) & (y)) ^ (~(x) & (z)))
#define maj(x,y,z)      (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))

#else   /* Thanks to Rich Schroeppel and Colin Plumb for the following      */

#define ch(x,y,z)       ((z) ^ ((x) & ((y) ^ (z))))
#define maj(x,y,z)      (((x) & (y)) | ((z) & ((x) ^ (y))))

#endif

/* round transforms for SHA256 and SHA512 compression functions */

#define vf(n,i) v[(n - i) & 7]

#define hf(i) (p[i & 15] += \
    g_1(p[(i + 14) & 15]) + p[(i + 9) & 15] + g_0(p[(i + 1) & 15]))

#define v_cycle(i,j)                                \
    vf(7,i) += (j ? hf(i) : p[i]) + k_0[i+j]        \
    + s_1(vf(4,i)) + ch(vf(4,i),vf(5,i),vf(6,i));   \
    vf(3,i) += vf(7,i);                             \
    vf(7,i) += s_0(vf(0,i))+ maj(vf(0,i),vf(1,i),vf(2,i))

#if defined(SHA_224) || defined(SHA_256)

#define SHA256_MASK (SHA256_BLOCK_SIZE - 1)

#if defined(SWAP_BYTES)
#define bsw_32(p,n) \
    { int _i = (n); while(_i--) ((uint_32t*)p)[_i] = bswap_32(((uint_32t*)p)[_i]); }
#else
#define bsw_32(p,n)
#endif

#define s_0(x)  (rotr32((x),  2) ^ rotr32((x), 13) ^ rotr32((x), 22))
#define s_1(x)  (rotr32((x),  6) ^ rotr32((x), 11) ^ rotr32((x), 25))
#define g_0(x)  (rotr32((x),  7) ^ rotr32((x), 18) ^ ((x) >>  3))
#define g_1(x)  (rotr32((x), 17) ^ rotr32((x), 19) ^ ((x) >> 10))
#define k_0     k256

/* rotated SHA256 round definition. Rather than swapping variables as in    */
/* FIPS-180, different variables are 'rotated' on each round, returning     */
/* to their starting positions every eight rounds                           */

#define qq(n)  v##n

#define one_cycle(a,b,c,d,e,f,g,h,k,w)  \
    qq(h) += s_1(qq(e)) + ch(qq(e), qq(f), qq(g)) + k + w; \
    qq(d) += qq(h); qq(h) += s_0(qq(a)) + maj(qq(a), qq(b), qq(c))

/* SHA256 mixing data   */

__constant__ const uint_32t k256[64] =
{   0x428a2f98ul, 0x71374491ul, 0xb5c0fbcful, 0xe9b5dba5ul,
    0x3956c25bul, 0x59f111f1ul, 0x923f82a4ul, 0xab1c5ed5ul,
    0xd807aa98ul, 0x12835b01ul, 0x243185beul, 0x550c7dc3ul,
    0x72be5d74ul, 0x80deb1feul, 0x9bdc06a7ul, 0xc19bf174ul,
    0xe49b69c1ul, 0xefbe4786ul, 0x0fc19dc6ul, 0x240ca1ccul,
    0x2de92c6ful, 0x4a7484aaul, 0x5cb0a9dcul, 0x76f988daul,
    0x983e5152ul, 0xa831c66dul, 0xb00327c8ul, 0xbf597fc7ul,
    0xc6e00bf3ul, 0xd5a79147ul, 0x06ca6351ul, 0x14292967ul,
    0x27b70a85ul, 0x2e1b2138ul, 0x4d2c6dfcul, 0x53380d13ul,
    0x650a7354ul, 0x766a0abbul, 0x81c2c92eul, 0x92722c85ul,
    0xa2bfe8a1ul, 0xa81a664bul, 0xc24b8b70ul, 0xc76c51a3ul,
    0xd192e819ul, 0xd6990624ul, 0xf40e3585ul, 0x106aa070ul,
    0x19a4c116ul, 0x1e376c08ul, 0x2748774cul, 0x34b0bcb5ul,
    0x391c0cb3ul, 0x4ed8aa4aul, 0x5b9cca4ful, 0x682e6ff3ul,
    0x748f82eeul, 0x78a5636ful, 0x84c87814ul, 0x8cc70208ul,
    0x90befffaul, 0xa4506cebul, 0xbef9a3f7ul, 0xc67178f2ul,
};

/* Compile 64 bytes of hash data into SHA256 digest value   */
/* NOTE: this routine assumes that the byte order in the    */
/* ctx->wbuf[] at this point is such that low address bytes */
/* in the ORIGINAL byte stream will go into the high end of */
/* words on BOTH big and little endian systems              */

__device__ VOID_RETURN sha256_compile(sha256_ctx ctx[1])
{
#if !defined(UNROLL_SHA2)

    uint_32t j, *p = ctx->wbuf, v[8];

    memcpy(v, ctx->hash, 8 * sizeof(uint_32t));

    for(j = 0; j < 64; j += 16)
    {
        v_cycle( 0, j); v_cycle( 1, j);
        v_cycle( 2, j); v_cycle( 3, j);
        v_cycle( 4, j); v_cycle( 5, j);
        v_cycle( 6, j); v_cycle( 7, j);
        v_cycle( 8, j); v_cycle( 9, j);
        v_cycle(10, j); v_cycle(11, j);
        v_cycle(12, j); v_cycle(13, j);
        v_cycle(14, j); v_cycle(15, j);
    }

    ctx->hash[0] += v[0]; ctx->hash[1] += v[1];
    ctx->hash[2] += v[2]; ctx->hash[3] += v[3];
    ctx->hash[4] += v[4]; ctx->hash[5] += v[5];
    ctx->hash[6] += v[6]; ctx->hash[7] += v[7];

#else

    uint_32t *p = ctx->wbuf,v0,v1,v2,v3,v4,v5,v6,v7;

    v0 = ctx->hash[0]; v1 = ctx->hash[1];
    v2 = ctx->hash[2]; v3 = ctx->hash[3];
    v4 = ctx->hash[4]; v5 = ctx->hash[5];
    v6 = ctx->hash[6]; v7 = ctx->hash[7];

    one_cycle(0,1,2,3,4,5,6,7,k256[ 0],p[ 0]);
    one_cycle(7,0,1,2,3,4,5,6,k256[ 1],p[ 1]);
    one_cycle(6,7,0,1,2,3,4,5,k256[ 2],p[ 2]);
    one_cycle(5,6,7,0,1,2,3,4,k256[ 3],p[ 3]);
    one_cycle(4,5,6,7,0,1,2,3,k256[ 4],p[ 4]);
    one_cycle(3,4,5,6,7,0,1,2,k256[ 5],p[ 5]);
    one_cycle(2,3,4,5,6,7,0,1,k256[ 6],p[ 6]);
    one_cycle(1,2,3,4,5,6,7,0,k256[ 7],p[ 7]);
    one_cycle(0,1,2,3,4,5,6,7,k256[ 8],p[ 8]);
    one_cycle(7,0,1,2,3,4,5,6,k256[ 9],p[ 9]);
    one_cycle(6,7,0,1,2,3,4,5,k256[10],p[10]);
    one_cycle(5,6,7,0,1,2,3,4,k256[11],p[11]);
    one_cycle(4,5,6,7,0,1,2,3,k256[12],p[12]);
    one_cycle(3,4,5,6,7,0,1,2,k256[13],p[13]);
    one_cycle(2,3,4,5,6,7,0,1,k256[14],p[14]);
    one_cycle(1,2,3,4,5,6,7,0,k256[15],p[15]);

    one_cycle(0,1,2,3,4,5,6,7,k256[16],hf( 0));
    one_cycle(7,0,1,2,3,4,5,6,k256[17],hf( 1));
    one_cycle(6,7,0,1,2,3,4,5,k256[18],hf( 2));
    one_cycle(5,6,7,0,1,2,3,4,k256[19],hf( 3));
    one_cycle(4,5,6,7,0,1,2,3,k256[20],hf( 4));
    one_cycle(3,4,5,6,7,0,1,2,k256[21],hf( 5));
    one_cycle(2,3,4,5,6,7,0,1,k256[22],hf( 6));
    one_cycle(1,2,3,4,5,6,7,0,k256[23],hf( 7));
    one_cycle(0,1,2,3,4,5,6,7,k256[24],hf( 8));
    one_cycle(7,0,1,2,3,4,5,6,k256[25],hf( 9));
    one_cycle(6,7,0,1,2,3,4,5,k256[26],hf(10));
    one_cycle(5,6,7,0,1,2,3,4,k256[27],hf(11));
    one_cycle(4,5,6,7,0,1,2,3,k256[28],hf(12));
    one_cycle(3,4,5,6,7,0,1,2,k256[29],hf(13));
    one_cycle(2,3,4,5,6,7,0,1,k256[30],hf(14));
    one_cycle(1,2,3,4,5,6,7,0,k256[31],hf(15));

    one_cycle(0,1,2,3,4,5,6,7,k256[32],hf( 0));
    one_cycle(7,0,1,2,3,4,5,6,k256[33],hf( 1));
    one_cycle(6,7,0,1,2,3,4,5,k256[34],hf( 2));
    one_cycle(5,6,7,0,1,2,3,4,k256[35],hf( 3));
    one_cycle(4,5,6,7,0,1,2,3,k256[36],hf( 4));
    one_cycle(3,4,5,6,7,0,1,2,k256[37],hf( 5));
    one_cycle(2,3,4,5,6,7,0,1,k256[38],hf( 6));
    one_cycle(1,2,3,4,5,6,7,0,k256[39],hf( 7));
    one_cycle(0,1,2,3,4,5,6,7,k256[40],hf( 8));
    one_cycle(7,0,1,2,3,4,5,6,k256[41],hf( 9));
    one_cycle(6,7,0,1,2,3,4,5,k256[42],hf(10));
    one_cycle(5,6,7,0,1,2,3,4,k256[43],hf(11));
    one_cycle(4,5,6,7,0,1,2,3,k256[44],hf(12));
    one_cycle(3,4,5,6,7,0,1,2,k256[45],hf(13));
    one_cycle(2,3,4,5,6,7,0,1,k256[46],hf(14));
    one_cycle(1,2,3,4,5,6,7,0,k256[47],hf(15));

    one_cycle(0,1,2,3,4,5,6,7,k256[48],hf( 0));
    one_cycle(7,0,1,2,3,4,5,6,k256[49],hf( 1));
    one_cycle(6,7,0,1,2,3,4,5,k256[50],hf( 2));
    one_cycle(5,6,7,0,1,2,3,4,k256[51],hf( 3));
    one_cycle(4,5,6,7,0,1,2,3,k256[52],hf( 4));
    one_cycle(3,4,5,6,7,0,1,2,k256[53],hf( 5));
    one_cycle(2,3,4,5,6,7,0,1,k256[54],hf( 6));
    one_cycle(1,2,3,4,5,6,7,0,k256[55],hf( 7));
    one_cycle(0,1,2,3,4,5,6,7,k256[56],hf( 8));
    one_cycle(7,0,1,2,3,4,5,6,k256[57],hf( 9));
    one_cycle(6,7,0,1,2,3,4,5,k256[58],hf(10));
    one_cycle(5,6,7,0,1,2,3,4,k256[59],hf(11));
    one_cycle(4,5,6,7,0,1,2,3,k256[60],hf(12));
    one_cycle(3,4,5,6,7,0,1,2,k256[61],hf(13));
    one_cycle(2,3,4,5,6,7,0,1,k256[62],hf(14));
    one_cycle(1,2,3,4,5,6,7,0,k256[63],hf(15));

    ctx->hash[0] += v0; ctx->hash[1] += v1;
    ctx->hash[2] += v2; ctx->hash[3] += v3;
    ctx->hash[4] += v4; ctx->hash[5] += v5;
    ctx->hash[6] += v6; ctx->hash[7] += v7;
#endif
}

/* SHA256 hash data in an array of bytes into hash buffer   */
/* and call the hash_compile function as required.          */

__device__ VOID_RETURN sha256_hash(const unsigned char data[], unsigned long len, sha256_ctx ctx[1])
{   uint_32t pos = (uint_32t)(ctx->count[0] & SHA256_MASK),
             space = SHA256_BLOCK_SIZE - pos;
    const unsigned char *sp = data;

    if((ctx->count[0] += len) < len)
        ++(ctx->count[1]);

    while(len >= space)     /* tranfer whole blocks while possible  */
    {
        memcpy(((unsigned char*)ctx->wbuf) + pos, sp, space);
        sp += space; len -= space; space = SHA256_BLOCK_SIZE; pos = 0;
        bsw_32(ctx->wbuf, SHA256_BLOCK_SIZE >> 2)
        sha256_compile(ctx);
    }

    memcpy(((unsigned char*)ctx->wbuf) + pos, sp, len);
}

/* SHA256 Final padding and digest calculation  */

__device__ static void sha_end1(unsigned char hval[], sha256_ctx ctx[1], const unsigned int hlen)
{   uint_32t    i = (uint_32t)(ctx->count[0] & SHA256_MASK);

    /* put bytes in the buffer in an order in which references to   */
    /* 32-bit words will put bytes with lower addresses into the    */
    /* top of 32 bit words on BOTH big and little endian machines   */
    bsw_32(ctx->wbuf, (i + 3) >> 2)

    /* we now need to mask valid bytes and add the padding which is */
    /* a single 1 bit and as many zero bits as necessary. Note that */
    /* we can always add the first padding byte here because the    */
    /* buffer always has at least one empty slot                    */
    ctx->wbuf[i >> 2] &= 0xffffff80 << 8 * (~i & 3);
    ctx->wbuf[i >> 2] |= 0x00000080 << 8 * (~i & 3);

    /* we need 9 or more empty positions, one for the padding byte  */
    /* (above) and eight for the length count.  If there is not     */
    /* enough space pad and empty the buffer                        */
    if(i > SHA256_BLOCK_SIZE - 9)
    {
        if(i < 60) ctx->wbuf[15] = 0;
        sha256_compile(ctx);
        i = 0;
    }
    else    /* compute a word index for the empty buffer positions  */
        i = (i >> 2) + 1;

    while(i < 14) /* and zero pad all but last two positions        */
        ctx->wbuf[i++] = 0;

    /* the following 32-bit length fields are assembled in the      */
    /* wrong byte order on little endian machines but this is       */
    /* corrected later since they are only ever used as 32-bit      */
    /* word values.                                                 */
    ctx->wbuf[14] = (ctx->count[1] << 3) | (ctx->count[0] >> 29);
    ctx->wbuf[15] = ctx->count[0] << 3;
    sha256_compile(ctx);

    /* extract the hash value as bytes in case the hash buffer is   */
    /* mislaigned for 32-bit words                                  */
    for(i = 0; i < hlen; ++i)
        hval[i] = (unsigned char)(ctx->hash[i >> 2] >> (8 * (~i & 3)));
}

#endif

#if defined(SHA_224)

__constant__ const uint_32t i224[8] =
{
    0xc1059ed8ul, 0x367cd507ul, 0x3070dd17ul, 0xf70e5939ul,
    0xffc00b31ul, 0x68581511ul, 0x64f98fa7ul, 0xbefa4fa4ul
};

__device__ VOID_RETURN sha224_begin(sha224_ctx ctx[1])
{
    ctx->count[0] = ctx->count[1] = 0;
    memcpy(ctx->hash, i224, 8 * sizeof(uint_32t));
}

__device__ VOID_RETURN sha224_end(unsigned char hval[], sha224_ctx ctx[1])
{
    sha_end1(hval, ctx, SHA224_DIGEST_SIZE);
}

__device__ VOID_RETURN sha224(unsigned char hval[], const unsigned char data[], unsigned long len)
{   sha224_ctx  cx[1];

    sha224_begin(cx);
    sha224_hash(data, len, cx);
    sha_end1(hval, cx, SHA224_DIGEST_SIZE);
}

#endif

#if defined(SHA_256)

__constant__ const uint_32t i256[8] =
{
    0x6a09e667ul, 0xbb67ae85ul, 0x3c6ef372ul, 0xa54ff53aul,
    0x510e527ful, 0x9b05688cul, 0x1f83d9abul, 0x5be0cd19ul
};

__device__ VOID_RETURN sha256_begin(sha256_ctx ctx[1])
{
    ctx->count[0] = ctx->count[1] = 0;
    memcpy(ctx->hash, i256, 8 * sizeof(uint_32t));
}

__device__ VOID_RETURN sha256_end(unsigned char hval[], sha256_ctx ctx[1])
{
    sha_end1(hval, ctx, SHA256_DIGEST_SIZE);
}

__device__ VOID_RETURN sha256(unsigned char hval[], const unsigned char data[], unsigned long len)
{   sha256_ctx  cx[1];

    sha256_begin(cx);
    sha256_hash(data, len, cx);
    sha_end1(hval, cx, SHA256_DIGEST_SIZE);
}

#endif

#if defined(SHA_384) || defined(SHA_512)

#define SHA512_MASK (SHA512_BLOCK_SIZE - 1)

#define rotr64(x,n)   (((x) >> n) | ((x) << (64 - n)))

#if !defined(bswap_64)
#define bswap_64(x) (((uint_64t)(bswap_32((uint_32t)(x)))) << 32 | bswap_32((uint_32t)((x) >> 32)))
#endif

#if defined(SWAP_BYTES)
#define bsw_64(p,n) \
    { int _i = (n); while(_i--) ((uint_64t*)p)[_i] = bswap_64(((uint_64t*)p)[_i]); }
#else
#define bsw_64(p,n)
#endif

/* SHA512 mixing function definitions   */

#ifdef   s_0
# undef  s_0
# undef  s_1
# undef  g_0
# undef  g_1
# undef  k_0
#endif

#define s_0(x)  (rotr64((x), 28) ^ rotr64((x), 34) ^ rotr64((x), 39))
#define s_1(x)  (rotr64((x), 14) ^ rotr64((x), 18) ^ rotr64((x), 41))
#define g_0(x)  (rotr64((x),  1) ^ rotr64((x),  8) ^ ((x) >>  7))
#define g_1(x)  (rotr64((x), 19) ^ rotr64((x), 61) ^ ((x) >>  6))
#define k_0     k512

/* SHA384/SHA512 mixing data    */

__constant__ const uint_64t  k512[80] =
{
    li_64(428a2f98d728ae22), li_64(7137449123ef65cd),
    li_64(b5c0fbcfec4d3b2f), li_64(e9b5dba58189dbbc),
    li_64(3956c25bf348b538), li_64(59f111f1b605d019),
    li_64(923f82a4af194f9b), li_64(ab1c5ed5da6d8118),
    li_64(d807aa98a3030242), li_64(12835b0145706fbe),
    li_64(243185be4ee4b28c), li_64(550c7dc3d5ffb4e2),
    li_64(72be5d74f27b896f), li_64(80deb1fe3b1696b1),
    li_64(9bdc06a725c71235), li_64(c19bf174cf692694),
    li_64(e49b69c19ef14ad2), li_64(efbe4786384f25e3),
    li_64(0fc19dc68b8cd5b5), li_64(240ca1cc77ac9c65),
    li_64(2de92c6f592b0275), li_64(4a7484aa6ea6e483),
    li_64(5cb0a9dcbd41fbd4), li_64(76f988da831153b5),
    li_64(983e5152ee66dfab), li_64(a831c66d2db43210),
    li_64(b00327c898fb213f), li_64(bf597fc7beef0ee4),
    li_64(c6e00bf33da88fc2), li_64(d5a79147930aa725),
    li_64(06ca6351e003826f), li_64(142929670a0e6e70),
    li_64(27b70a8546d22ffc), li_64(2e1b21385c26c926),
    li_64(4d2c6dfc5ac42aed), li_64(53380d139d95b3df),
    li_64(650a73548baf63de), li_64(766a0abb3c77b2a8),
    li_64(81c2c92e47edaee6), li_64(92722c851482353b),
    li_64(a2bfe8a14cf10364), li_64(a81a664bbc423001),
    li_64(c24b8b70d0f89791), li_64(c76c51a30654be30),
    li_64(d192e819d6ef5218), li_64(d69906245565a910),
    li_64(f40e35855771202a), li_64(106aa07032bbd1b8),
    li_64(19a4c116b8d2d0c8), li_64(1e376c085141ab53),
    li_64(2748774cdf8eeb99), li_64(34b0bcb5e19b48a8),
    li_64(391c0cb3c5c95a63), li_64(4ed8aa4ae3418acb),
    li_64(5b9cca4f7763e373), li_64(682e6ff3d6b2b8a3),
    li_64(748f82ee5defb2fc), li_64(78a5636f43172f60),
    li_64(84c87814a1f0ab72), li_64(8cc702081a6439ec),
    li_64(90befffa23631e28), li_64(a4506cebde82bde9),
    li_64(bef9a3f7b2c67915), li_64(c67178f2e372532b),
    li_64(ca273eceea26619c), li_64(d186b8c721c0c207),
    li_64(eada7dd6cde0eb1e), li_64(f57d4f7fee6ed178),
    li_64(06f067aa72176fba), li_64(0a637dc5a2c898a6),
    li_64(113f9804bef90dae), li_64(1b710b35131c471b),
    li_64(28db77f523047d84), li_64(32caab7b40c72493),
    li_64(3c9ebe0a15c9bebc), li_64(431d67c49c100d4c),
    li_64(4cc5d4becb3e42b6), li_64(597f299cfc657e2a),
    li_64(5fcb6fab3ad6faec), li_64(6c44198c4a475817)
};

/* Compile 128 bytes of hash data into SHA384/512 digest    */
/* NOTE: this routine assumes that the byte order in the    */
/* ctx->wbuf[] at this point is such that low address bytes */
/* in the ORIGINAL byte stream will go into the high end of */
/* words on BOTH big and little endian systems              */

__device__ VOID_RETURN sha512_compile(sha512_ctx ctx[1])
{   uint_64t    v[8], *p = ctx->wbuf;
    uint_32t    j;

    memcpy(v, ctx->hash, 8 * sizeof(uint_64t));

    for(j = 0; j < 80; j += 16)
    {
        v_cycle( 0, j); v_cycle( 1, j);
        v_cycle( 2, j); v_cycle( 3, j);
        v_cycle( 4, j); v_cycle( 5, j);
        v_cycle( 6, j); v_cycle( 7, j);
        v_cycle( 8, j); v_cycle( 9, j);
        v_cycle(10, j); v_cycle(11, j);
        v_cycle(12, j); v_cycle(13, j);
        v_cycle(14, j); v_cycle(15, j);
    }

    ctx->hash[0] += v[0]; ctx->hash[1] += v[1];
    ctx->hash[2] += v[2]; ctx->hash[3] += v[3];
    ctx->hash[4] += v[4]; ctx->hash[5] += v[5];
    ctx->hash[6] += v[6]; ctx->hash[7] += v[7];
}

/* Compile 128 bytes of hash data into SHA256 digest value  */
/* NOTE: this routine assumes that the byte order in the    */
/* ctx->wbuf[] at this point is in such an order that low   */
/* address bytes in the ORIGINAL byte stream placed in this */
/* buffer will now go to the high end of words on BOTH big  */
/* and little endian systems                                */

__device__ VOID_RETURN sha512_hash(const unsigned char data[], unsigned long len, sha512_ctx ctx[1])
{   uint_32t pos = (uint_32t)(ctx->count[0] & SHA512_MASK),
             space = SHA512_BLOCK_SIZE - pos;
    const unsigned char *sp = data;

    if((ctx->count[0] += len) < len)
        ++(ctx->count[1]);

    while(len >= space)     /* tranfer whole blocks while possible  */
    {
        memcpy(((unsigned char*)ctx->wbuf) + pos, sp, space);
        sp += space; len -= space; space = SHA512_BLOCK_SIZE; pos = 0;
        bsw_64(ctx->wbuf, SHA512_BLOCK_SIZE >> 3);
        sha512_compile(ctx);
    }

    memcpy(((unsigned char*)ctx->wbuf) + pos, sp, len);
}

/* SHA384/512 Final padding and digest calculation  */

__device__ static void sha_end2(unsigned char hval[], sha512_ctx ctx[1], const unsigned int hlen)
{   uint_32t    i = (uint_32t)(ctx->count[0] & SHA512_MASK);

    /* put bytes in the buffer in an order in which references to   */
    /* 32-bit words will put bytes with lower addresses into the    */
    /* top of 32 bit words on BOTH big and little endian machines   */
    bsw_64(ctx->wbuf, (i + 7) >> 3);

    /* we now need to mask valid bytes and add the padding which is */
    /* a single 1 bit and as many zero bits as necessary. Note that */
    /* we can always add the first padding byte here because the    */
    /* buffer always has at least one empty slot                    */
    ctx->wbuf[i >> 3] &= li_64(ffffffffffffff00) << 8 * (~i & 7);
    ctx->wbuf[i >> 3] |= li_64(0000000000000080) << 8 * (~i & 7);

    /* we need 17 or more empty byte positions, one for the padding */
    /* byte (above) and sixteen for the length count.  If there is  */
    /* not enough space pad and empty the buffer                    */
    if(i > SHA512_BLOCK_SIZE - 17)
    {
        if(i < 120) ctx->wbuf[15] = 0;
        sha512_compile(ctx);
        i = 0;
    }
    else
        i = (i >> 3) + 1;

    while(i < 14)
        ctx->wbuf[i++] = 0;

    /* the following 64-bit length fields are assembled in the      */
    /* wrong byte order on little endian machines but this is       */
    /* corrected later since they are only ever used as 64-bit      */
    /* word values.                                                 */
    ctx->wbuf[14] = (ctx->count[1] << 3) | (ctx->count[0] >> 61);
    ctx->wbuf[15] = ctx->count[0] << 3;
    sha512_compile(ctx);

    /* extract the hash value as bytes in case the hash buffer is   */
    /* misaligned for 32-bit words                                  */
    for(i = 0; i < hlen; ++i)
        hval[i] = (unsigned char)(ctx->hash[i >> 3] >> (8 * (~i & 7)));
}

#endif

#if defined(SHA_384)

/* SHA384 initialisation data   */

__constant__ const uint_64t  i384[80] =
{
    li_64(cbbb9d5dc1059ed8), li_64(629a292a367cd507),
    li_64(9159015a3070dd17), li_64(152fecd8f70e5939),
    li_64(67332667ffc00b31), li_64(8eb44a8768581511),
    li_64(db0c2e0d64f98fa7), li_64(47b5481dbefa4fa4)
};

__device__ VOID_RETURN sha384_begin(sha384_ctx ctx[1])
{
    ctx->count[0] = ctx->count[1] = 0;
    memcpy(ctx->hash, i384, 8 * sizeof(uint_64t));
}

__device__ VOID_RETURN sha384_end(unsigned char hval[], sha384_ctx ctx[1])
{
    sha_end2(hval, ctx, SHA384_DIGEST_SIZE);
}

__device__ VOID_RETURN sha384(unsigned char hval[], const unsigned char data[], unsigned long len)
{   sha384_ctx  cx[1];

    sha384_begin(cx);
    sha384_hash(data, len, cx);
    sha_end2(hval, cx, SHA384_DIGEST_SIZE);
}

#endif

#if defined(SHA_512)

/* SHA512 initialisation data   */

__constant__ const uint_64t  i512[80] =
{
    li_64(6a09e667f3bcc908), li_64(bb67ae8584caa73b),
    li_64(3c6ef372fe94f82b), li_64(a54ff53a5f1d36f1),
    li_64(510e527fade682d1), li_64(9b05688c2b3e6c1f),
    li_64(1f83d9abfb41bd6b), li_64(5be0cd19137e2179)
};

__device__ VOID_RETURN sha512_begin(sha512_ctx ctx[1])
{
    ctx->count[0] = ctx->count[1] = 0;
    memcpy(ctx->hash, i512, 8 * sizeof(uint_64t));
}

__device__ VOID_RETURN sha512_end(unsigned char hval[], sha512_ctx ctx[1])
{
    sha_end2(hval, ctx, SHA512_DIGEST_SIZE);
}

__device__ VOID_RETURN sha512(unsigned char hval[], const unsigned char data[], unsigned long len)
{   sha512_ctx  cx[1];

    sha512_begin(cx);
    sha512_hash(data, len, cx);
    sha_end2(hval, cx, SHA512_DIGEST_SIZE);
}

#endif

#if defined(SHA_2)

#define CTX_224(x)  ((x)->uu->ctx256)
#define CTX_256(x)  ((x)->uu->ctx256)
#define CTX_384(x)  ((x)->uu->ctx512)
#define CTX_512(x)  ((x)->uu->ctx512)

/* SHA2 initialisation */

__device__ INT_RETURN sha2_begin(unsigned long len, sha2_ctx ctx[1])
{
    switch(len)
    {
#if defined(SHA_224)
        case 224:
        case  28:   CTX_256(ctx)->count[0] = CTX_256(ctx)->count[1] = 0;
                    memcpy(CTX_256(ctx)->hash, i224, 32);
                    ctx->sha2_len = 28; return EXIT_SUCCESS;
#endif
#if defined(SHA_256)
        case 256:
        case  32:   CTX_256(ctx)->count[0] = CTX_256(ctx)->count[1] = 0;
                    memcpy(CTX_256(ctx)->hash, i256, 32);
                    ctx->sha2_len = 32; return EXIT_SUCCESS;
#endif
#if defined(SHA_384)
        case 384:
        case  48:   CTX_384(ctx)->count[0] = CTX_384(ctx)->count[1] = 0;
                    memcpy(CTX_384(ctx)->hash, i384, 64);
                    ctx->sha2_len = 48; return EXIT_SUCCESS;
#endif
#if defined(SHA_512)
        case 512:
        case  64:   CTX_512(ctx)->count[0] = CTX_512(ctx)->count[1] = 0;
                    memcpy(CTX_512(ctx)->hash, i512, 64);
                    ctx->sha2_len = 64; return EXIT_SUCCESS;
#endif
        default:    return EXIT_FAILURE;
    }
}

__device__ VOID_RETURN sha2_hash(const unsigned char data[], unsigned long len, sha2_ctx ctx[1])
{
    switch(ctx->sha2_len)
    {
#if defined(SHA_224)
        case 28: sha224_hash(data, len, CTX_224(ctx)); return;
#endif
#if defined(SHA_256)
        case 32: sha256_hash(data, len, CTX_256(ctx)); return;
#endif
#if defined(SHA_384)
        case 48: sha384_hash(data, len, CTX_384(ctx)); return;
#endif
#if defined(SHA_512)
        case 64: sha512_hash(data, len, CTX_512(ctx)); return;
#endif
    }
}

__device__ VOID_RETURN sha2_end(unsigned char hval[], sha2_ctx ctx[1])
{
    switch(ctx->sha2_len)
    {
#if defined(SHA_224)
        case 28: sha_end1(hval, CTX_224(ctx), SHA224_DIGEST_SIZE); return;
#endif
#if defined(SHA_256)
        case 32: sha_end1(hval, CTX_256(ctx), SHA256_DIGEST_SIZE); return;
#endif
#if defined(SHA_384)
        case 48: sha_end2(hval, CTX_384(ctx), SHA384_DIGEST_SIZE); return;
#endif
#if defined(SHA_512)
        case 64: sha_end2(hval, CTX_512(ctx), SHA512_DIGEST_SIZE); return;
#endif
    }
}

__device__ INT_RETURN sha2(unsigned char hval[], unsigned long size,
                                const unsigned char data[], unsigned long len)
{   sha2_ctx    cx[1];

    if(sha2_begin(size, cx) == EXIT_SUCCESS)
    {
        sha2_hash(data, len, cx); sha2_end(hval, cx); return EXIT_SUCCESS;
    }
    else
        return EXIT_FAILURE;
}

#endif

#if defined(__cplusplus)
}
#endif
/**
 * The Whirlpool hashing function.
 *
 * <P>
 * <b>References</b>
 *
 * <P>
 * The Whirlpool algorithm was developed by
 * <a href="mailto:pbarreto@scopus.com.br">Paulo S. L. M. Barreto</a> and
 * <a href="mailto:vincent.rijmen@cryptomathic.com">Vincent Rijmen</a>.
 *
 * See
 *      P.S.L.M. Barreto, V. Rijmen,
 *      ``The Whirlpool hashing function,''
 *      NESSIE submission, 2000 (tweaked version, 2001),
 *      <https://www.cosic.esat.kuleuven.ac.be/nessie/workshop/submissions/whirlpool.zip>
 * 
 * @author  Paulo S.L.M. Barreto
 * @author  Vincent Rijmen.
 * Adapted for TrueCrypt.
 *
 * @version 3.0 (2003.03.12)
 *
 * =============================================================================
 *
 * Differences from version 2.1:
 *
 * - Suboptimal diffusion matrix replaced by cir(1, 1, 4, 1, 8, 5, 2, 9).
 *
 * =============================================================================
 *
 * Differences from version 2.0:
 *
 * - Generation of ISO/IEC 10118-3 test vectors.
 * - Bug fix: nonzero carry was ignored when tallying the data length
 *      (this bug apparently only manifested itself when feeding data
 *      in pieces rather than in a single chunk at once).
 * - Support for MS Visual C++ 64-bit integer arithmetic.
 *
 * Differences from version 1.0:
 *
 * - Original S-box replaced by the tweaked, hardware-efficient version.
 *
 * =============================================================================
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */
 /* The code contained in this file (Whirlpool.c) is in the public domain. */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "Whirlpool.cuh"

/* #define TRACE_INTERMEDIATE_VALUES */

/*
 * The number of rounds of the internal dedicated block cipher.
 */
#define R 10

/*
 * Though Whirlpool is endianness-neutral, the encryption tables are listed
 * in BIG-ENDIAN format, which is adopted throughout this implementation
 * (but little-endian notation would be equally suitable if consistently
 * employed).
 */

__constant__ static const u64 C0[256] = {
    LL(0x18186018c07830d8), LL(0x23238c2305af4626), LL(0xc6c63fc67ef991b8), LL(0xe8e887e8136fcdfb),
    LL(0x878726874ca113cb), LL(0xb8b8dab8a9626d11), LL(0x0101040108050209), LL(0x4f4f214f426e9e0d),
    LL(0x3636d836adee6c9b), LL(0xa6a6a2a6590451ff), LL(0xd2d26fd2debdb90c), LL(0xf5f5f3f5fb06f70e),
    LL(0x7979f979ef80f296), LL(0x6f6fa16f5fcede30), LL(0x91917e91fcef3f6d), LL(0x52525552aa07a4f8),
    LL(0x60609d6027fdc047), LL(0xbcbccabc89766535), LL(0x9b9b569baccd2b37), LL(0x8e8e028e048c018a),
    LL(0xa3a3b6a371155bd2), LL(0x0c0c300c603c186c), LL(0x7b7bf17bff8af684), LL(0x3535d435b5e16a80),
    LL(0x1d1d741de8693af5), LL(0xe0e0a7e05347ddb3), LL(0xd7d77bd7f6acb321), LL(0xc2c22fc25eed999c),
    LL(0x2e2eb82e6d965c43), LL(0x4b4b314b627a9629), LL(0xfefedffea321e15d), LL(0x575741578216aed5),
    LL(0x15155415a8412abd), LL(0x7777c1779fb6eee8), LL(0x3737dc37a5eb6e92), LL(0xe5e5b3e57b56d79e),
    LL(0x9f9f469f8cd92313), LL(0xf0f0e7f0d317fd23), LL(0x4a4a354a6a7f9420), LL(0xdada4fda9e95a944),
    LL(0x58587d58fa25b0a2), LL(0xc9c903c906ca8fcf), LL(0x2929a429558d527c), LL(0x0a0a280a5022145a),
    LL(0xb1b1feb1e14f7f50), LL(0xa0a0baa0691a5dc9), LL(0x6b6bb16b7fdad614), LL(0x85852e855cab17d9),
    LL(0xbdbdcebd8173673c), LL(0x5d5d695dd234ba8f), LL(0x1010401080502090), LL(0xf4f4f7f4f303f507),
    LL(0xcbcb0bcb16c08bdd), LL(0x3e3ef83eedc67cd3), LL(0x0505140528110a2d), LL(0x676781671fe6ce78),
    LL(0xe4e4b7e47353d597), LL(0x27279c2725bb4e02), LL(0x4141194132588273), LL(0x8b8b168b2c9d0ba7),
    LL(0xa7a7a6a7510153f6), LL(0x7d7de97dcf94fab2), LL(0x95956e95dcfb3749), LL(0xd8d847d88e9fad56),
    LL(0xfbfbcbfb8b30eb70), LL(0xeeee9fee2371c1cd), LL(0x7c7ced7cc791f8bb), LL(0x6666856617e3cc71),
    LL(0xdddd53dda68ea77b), LL(0x17175c17b84b2eaf), LL(0x4747014702468e45), LL(0x9e9e429e84dc211a),
    LL(0xcaca0fca1ec589d4), LL(0x2d2db42d75995a58), LL(0xbfbfc6bf9179632e), LL(0x07071c07381b0e3f),
    LL(0xadad8ead012347ac), LL(0x5a5a755aea2fb4b0), LL(0x838336836cb51bef), LL(0x3333cc3385ff66b6),
    LL(0x636391633ff2c65c), LL(0x02020802100a0412), LL(0xaaaa92aa39384993), LL(0x7171d971afa8e2de),
    LL(0xc8c807c80ecf8dc6), LL(0x19196419c87d32d1), LL(0x494939497270923b), LL(0xd9d943d9869aaf5f),
    LL(0xf2f2eff2c31df931), LL(0xe3e3abe34b48dba8), LL(0x5b5b715be22ab6b9), LL(0x88881a8834920dbc),
    LL(0x9a9a529aa4c8293e), LL(0x262698262dbe4c0b), LL(0x3232c8328dfa64bf), LL(0xb0b0fab0e94a7d59),
    LL(0xe9e983e91b6acff2), LL(0x0f0f3c0f78331e77), LL(0xd5d573d5e6a6b733), LL(0x80803a8074ba1df4),
    LL(0xbebec2be997c6127), LL(0xcdcd13cd26de87eb), LL(0x3434d034bde46889), LL(0x48483d487a759032),
    LL(0xffffdbffab24e354), LL(0x7a7af57af78ff48d), LL(0x90907a90f4ea3d64), LL(0x5f5f615fc23ebe9d),
    LL(0x202080201da0403d), LL(0x6868bd6867d5d00f), LL(0x1a1a681ad07234ca), LL(0xaeae82ae192c41b7),
    LL(0xb4b4eab4c95e757d), LL(0x54544d549a19a8ce), LL(0x93937693ece53b7f), LL(0x222288220daa442f),
    LL(0x64648d6407e9c863), LL(0xf1f1e3f1db12ff2a), LL(0x7373d173bfa2e6cc), LL(0x12124812905a2482),
    LL(0x40401d403a5d807a), LL(0x0808200840281048), LL(0xc3c32bc356e89b95), LL(0xecec97ec337bc5df),
    LL(0xdbdb4bdb9690ab4d), LL(0xa1a1bea1611f5fc0), LL(0x8d8d0e8d1c830791), LL(0x3d3df43df5c97ac8),
    LL(0x97976697ccf1335b), LL(0x0000000000000000), LL(0xcfcf1bcf36d483f9), LL(0x2b2bac2b4587566e),
    LL(0x7676c57697b3ece1), LL(0x8282328264b019e6), LL(0xd6d67fd6fea9b128), LL(0x1b1b6c1bd87736c3),
    LL(0xb5b5eeb5c15b7774), LL(0xafaf86af112943be), LL(0x6a6ab56a77dfd41d), LL(0x50505d50ba0da0ea),
    LL(0x45450945124c8a57), LL(0xf3f3ebf3cb18fb38), LL(0x3030c0309df060ad), LL(0xefef9bef2b74c3c4),
    LL(0x3f3ffc3fe5c37eda), LL(0x55554955921caac7), LL(0xa2a2b2a2791059db), LL(0xeaea8fea0365c9e9),
    LL(0x656589650fecca6a), LL(0xbabad2bab9686903), LL(0x2f2fbc2f65935e4a), LL(0xc0c027c04ee79d8e),
    LL(0xdede5fdebe81a160), LL(0x1c1c701ce06c38fc), LL(0xfdfdd3fdbb2ee746), LL(0x4d4d294d52649a1f),
    LL(0x92927292e4e03976), LL(0x7575c9758fbceafa), LL(0x06061806301e0c36), LL(0x8a8a128a249809ae),
    LL(0xb2b2f2b2f940794b), LL(0xe6e6bfe66359d185), LL(0x0e0e380e70361c7e), LL(0x1f1f7c1ff8633ee7),
    LL(0x6262956237f7c455), LL(0xd4d477d4eea3b53a), LL(0xa8a89aa829324d81), LL(0x96966296c4f43152),
    LL(0xf9f9c3f99b3aef62), LL(0xc5c533c566f697a3), LL(0x2525942535b14a10), LL(0x59597959f220b2ab),
    LL(0x84842a8454ae15d0), LL(0x7272d572b7a7e4c5), LL(0x3939e439d5dd72ec), LL(0x4c4c2d4c5a619816),
    LL(0x5e5e655eca3bbc94), LL(0x7878fd78e785f09f), LL(0x3838e038ddd870e5), LL(0x8c8c0a8c14860598),
    LL(0xd1d163d1c6b2bf17), LL(0xa5a5aea5410b57e4), LL(0xe2e2afe2434dd9a1), LL(0x616199612ff8c24e),
    LL(0xb3b3f6b3f1457b42), LL(0x2121842115a54234), LL(0x9c9c4a9c94d62508), LL(0x1e1e781ef0663cee),
    LL(0x4343114322528661), LL(0xc7c73bc776fc93b1), LL(0xfcfcd7fcb32be54f), LL(0x0404100420140824),
    LL(0x51515951b208a2e3), LL(0x99995e99bcc72f25), LL(0x6d6da96d4fc4da22), LL(0x0d0d340d68391a65),
    LL(0xfafacffa8335e979), LL(0xdfdf5bdfb684a369), LL(0x7e7ee57ed79bfca9), LL(0x242490243db44819),
    LL(0x3b3bec3bc5d776fe), LL(0xabab96ab313d4b9a), LL(0xcece1fce3ed181f0), LL(0x1111441188552299),
    LL(0x8f8f068f0c890383), LL(0x4e4e254e4a6b9c04), LL(0xb7b7e6b7d1517366), LL(0xebeb8beb0b60cbe0),
    LL(0x3c3cf03cfdcc78c1), LL(0x81813e817cbf1ffd), LL(0x94946a94d4fe3540), LL(0xf7f7fbf7eb0cf31c),
    LL(0xb9b9deb9a1676f18), LL(0x13134c13985f268b), LL(0x2c2cb02c7d9c5851), LL(0xd3d36bd3d6b8bb05),
    LL(0xe7e7bbe76b5cd38c), LL(0x6e6ea56e57cbdc39), LL(0xc4c437c46ef395aa), LL(0x03030c03180f061b),
    LL(0x565645568a13acdc), LL(0x44440d441a49885e), LL(0x7f7fe17fdf9efea0), LL(0xa9a99ea921374f88),
    LL(0x2a2aa82a4d825467), LL(0xbbbbd6bbb16d6b0a), LL(0xc1c123c146e29f87), LL(0x53535153a202a6f1),
    LL(0xdcdc57dcae8ba572), LL(0x0b0b2c0b58271653), LL(0x9d9d4e9d9cd32701), LL(0x6c6cad6c47c1d82b),
    LL(0x3131c43195f562a4), LL(0x7474cd7487b9e8f3), LL(0xf6f6fff6e309f115), LL(0x464605460a438c4c),
    LL(0xacac8aac092645a5), LL(0x89891e893c970fb5), LL(0x14145014a04428b4), LL(0xe1e1a3e15b42dfba),
    LL(0x16165816b04e2ca6), LL(0x3a3ae83acdd274f7), LL(0x6969b9696fd0d206), LL(0x09092409482d1241),
    LL(0x7070dd70a7ade0d7), LL(0xb6b6e2b6d954716f), LL(0xd0d067d0ceb7bd1e), LL(0xeded93ed3b7ec7d6),
    LL(0xcccc17cc2edb85e2), LL(0x424215422a578468), LL(0x98985a98b4c22d2c), LL(0xa4a4aaa4490e55ed),
    LL(0x2828a0285d885075), LL(0x5c5c6d5cda31b886), LL(0xf8f8c7f8933fed6b), LL(0x8686228644a411c2),
};

__constant__ static const u64 C1[256] = {
    LL(0xd818186018c07830), LL(0x2623238c2305af46), LL(0xb8c6c63fc67ef991), LL(0xfbe8e887e8136fcd),
    LL(0xcb878726874ca113), LL(0x11b8b8dab8a9626d), LL(0x0901010401080502), LL(0x0d4f4f214f426e9e),
    LL(0x9b3636d836adee6c), LL(0xffa6a6a2a6590451), LL(0x0cd2d26fd2debdb9), LL(0x0ef5f5f3f5fb06f7),
    LL(0x967979f979ef80f2), LL(0x306f6fa16f5fcede), LL(0x6d91917e91fcef3f), LL(0xf852525552aa07a4),
    LL(0x4760609d6027fdc0), LL(0x35bcbccabc897665), LL(0x379b9b569baccd2b), LL(0x8a8e8e028e048c01),
    LL(0xd2a3a3b6a371155b), LL(0x6c0c0c300c603c18), LL(0x847b7bf17bff8af6), LL(0x803535d435b5e16a),
    LL(0xf51d1d741de8693a), LL(0xb3e0e0a7e05347dd), LL(0x21d7d77bd7f6acb3), LL(0x9cc2c22fc25eed99),
    LL(0x432e2eb82e6d965c), LL(0x294b4b314b627a96), LL(0x5dfefedffea321e1), LL(0xd5575741578216ae),
    LL(0xbd15155415a8412a), LL(0xe87777c1779fb6ee), LL(0x923737dc37a5eb6e), LL(0x9ee5e5b3e57b56d7),
    LL(0x139f9f469f8cd923), LL(0x23f0f0e7f0d317fd), LL(0x204a4a354a6a7f94), LL(0x44dada4fda9e95a9),
    LL(0xa258587d58fa25b0), LL(0xcfc9c903c906ca8f), LL(0x7c2929a429558d52), LL(0x5a0a0a280a502214),
    LL(0x50b1b1feb1e14f7f), LL(0xc9a0a0baa0691a5d), LL(0x146b6bb16b7fdad6), LL(0xd985852e855cab17),
    LL(0x3cbdbdcebd817367), LL(0x8f5d5d695dd234ba), LL(0x9010104010805020), LL(0x07f4f4f7f4f303f5),
    LL(0xddcbcb0bcb16c08b), LL(0xd33e3ef83eedc67c), LL(0x2d0505140528110a), LL(0x78676781671fe6ce),
    LL(0x97e4e4b7e47353d5), LL(0x0227279c2725bb4e), LL(0x7341411941325882), LL(0xa78b8b168b2c9d0b),
    LL(0xf6a7a7a6a7510153), LL(0xb27d7de97dcf94fa), LL(0x4995956e95dcfb37), LL(0x56d8d847d88e9fad),
    LL(0x70fbfbcbfb8b30eb), LL(0xcdeeee9fee2371c1), LL(0xbb7c7ced7cc791f8), LL(0x716666856617e3cc),
    LL(0x7bdddd53dda68ea7), LL(0xaf17175c17b84b2e), LL(0x454747014702468e), LL(0x1a9e9e429e84dc21),
    LL(0xd4caca0fca1ec589), LL(0x582d2db42d75995a), LL(0x2ebfbfc6bf917963), LL(0x3f07071c07381b0e),
    LL(0xacadad8ead012347), LL(0xb05a5a755aea2fb4), LL(0xef838336836cb51b), LL(0xb63333cc3385ff66),
    LL(0x5c636391633ff2c6), LL(0x1202020802100a04), LL(0x93aaaa92aa393849), LL(0xde7171d971afa8e2),
    LL(0xc6c8c807c80ecf8d), LL(0xd119196419c87d32), LL(0x3b49493949727092), LL(0x5fd9d943d9869aaf),
    LL(0x31f2f2eff2c31df9), LL(0xa8e3e3abe34b48db), LL(0xb95b5b715be22ab6), LL(0xbc88881a8834920d),
    LL(0x3e9a9a529aa4c829), LL(0x0b262698262dbe4c), LL(0xbf3232c8328dfa64), LL(0x59b0b0fab0e94a7d),
    LL(0xf2e9e983e91b6acf), LL(0x770f0f3c0f78331e), LL(0x33d5d573d5e6a6b7), LL(0xf480803a8074ba1d),
    LL(0x27bebec2be997c61), LL(0xebcdcd13cd26de87), LL(0x893434d034bde468), LL(0x3248483d487a7590),
    LL(0x54ffffdbffab24e3), LL(0x8d7a7af57af78ff4), LL(0x6490907a90f4ea3d), LL(0x9d5f5f615fc23ebe),
    LL(0x3d202080201da040), LL(0x0f6868bd6867d5d0), LL(0xca1a1a681ad07234), LL(0xb7aeae82ae192c41),
    LL(0x7db4b4eab4c95e75), LL(0xce54544d549a19a8), LL(0x7f93937693ece53b), LL(0x2f222288220daa44),
    LL(0x6364648d6407e9c8), LL(0x2af1f1e3f1db12ff), LL(0xcc7373d173bfa2e6), LL(0x8212124812905a24),
    LL(0x7a40401d403a5d80), LL(0x4808082008402810), LL(0x95c3c32bc356e89b), LL(0xdfecec97ec337bc5),
    LL(0x4ddbdb4bdb9690ab), LL(0xc0a1a1bea1611f5f), LL(0x918d8d0e8d1c8307), LL(0xc83d3df43df5c97a),
    LL(0x5b97976697ccf133), LL(0x0000000000000000), LL(0xf9cfcf1bcf36d483), LL(0x6e2b2bac2b458756),
    LL(0xe17676c57697b3ec), LL(0xe68282328264b019), LL(0x28d6d67fd6fea9b1), LL(0xc31b1b6c1bd87736),
    LL(0x74b5b5eeb5c15b77), LL(0xbeafaf86af112943), LL(0x1d6a6ab56a77dfd4), LL(0xea50505d50ba0da0),
    LL(0x5745450945124c8a), LL(0x38f3f3ebf3cb18fb), LL(0xad3030c0309df060), LL(0xc4efef9bef2b74c3),
    LL(0xda3f3ffc3fe5c37e), LL(0xc755554955921caa), LL(0xdba2a2b2a2791059), LL(0xe9eaea8fea0365c9),
    LL(0x6a656589650fecca), LL(0x03babad2bab96869), LL(0x4a2f2fbc2f65935e), LL(0x8ec0c027c04ee79d),
    LL(0x60dede5fdebe81a1), LL(0xfc1c1c701ce06c38), LL(0x46fdfdd3fdbb2ee7), LL(0x1f4d4d294d52649a),
    LL(0x7692927292e4e039), LL(0xfa7575c9758fbcea), LL(0x3606061806301e0c), LL(0xae8a8a128a249809),
    LL(0x4bb2b2f2b2f94079), LL(0x85e6e6bfe66359d1), LL(0x7e0e0e380e70361c), LL(0xe71f1f7c1ff8633e),
    LL(0x556262956237f7c4), LL(0x3ad4d477d4eea3b5), LL(0x81a8a89aa829324d), LL(0x5296966296c4f431),
    LL(0x62f9f9c3f99b3aef), LL(0xa3c5c533c566f697), LL(0x102525942535b14a), LL(0xab59597959f220b2),
    LL(0xd084842a8454ae15), LL(0xc57272d572b7a7e4), LL(0xec3939e439d5dd72), LL(0x164c4c2d4c5a6198),
    LL(0x945e5e655eca3bbc), LL(0x9f7878fd78e785f0), LL(0xe53838e038ddd870), LL(0x988c8c0a8c148605),
    LL(0x17d1d163d1c6b2bf), LL(0xe4a5a5aea5410b57), LL(0xa1e2e2afe2434dd9), LL(0x4e616199612ff8c2),
    LL(0x42b3b3f6b3f1457b), LL(0x342121842115a542), LL(0x089c9c4a9c94d625), LL(0xee1e1e781ef0663c),
    LL(0x6143431143225286), LL(0xb1c7c73bc776fc93), LL(0x4ffcfcd7fcb32be5), LL(0x2404041004201408),
    LL(0xe351515951b208a2), LL(0x2599995e99bcc72f), LL(0x226d6da96d4fc4da), LL(0x650d0d340d68391a),
    LL(0x79fafacffa8335e9), LL(0x69dfdf5bdfb684a3), LL(0xa97e7ee57ed79bfc), LL(0x19242490243db448),
    LL(0xfe3b3bec3bc5d776), LL(0x9aabab96ab313d4b), LL(0xf0cece1fce3ed181), LL(0x9911114411885522),
    LL(0x838f8f068f0c8903), LL(0x044e4e254e4a6b9c), LL(0x66b7b7e6b7d15173), LL(0xe0ebeb8beb0b60cb),
    LL(0xc13c3cf03cfdcc78), LL(0xfd81813e817cbf1f), LL(0x4094946a94d4fe35), LL(0x1cf7f7fbf7eb0cf3),
    LL(0x18b9b9deb9a1676f), LL(0x8b13134c13985f26), LL(0x512c2cb02c7d9c58), LL(0x05d3d36bd3d6b8bb),
    LL(0x8ce7e7bbe76b5cd3), LL(0x396e6ea56e57cbdc), LL(0xaac4c437c46ef395), LL(0x1b03030c03180f06),
    LL(0xdc565645568a13ac), LL(0x5e44440d441a4988), LL(0xa07f7fe17fdf9efe), LL(0x88a9a99ea921374f),
    LL(0x672a2aa82a4d8254), LL(0x0abbbbd6bbb16d6b), LL(0x87c1c123c146e29f), LL(0xf153535153a202a6),
    LL(0x72dcdc57dcae8ba5), LL(0x530b0b2c0b582716), LL(0x019d9d4e9d9cd327), LL(0x2b6c6cad6c47c1d8),
    LL(0xa43131c43195f562), LL(0xf37474cd7487b9e8), LL(0x15f6f6fff6e309f1), LL(0x4c464605460a438c),
    LL(0xa5acac8aac092645), LL(0xb589891e893c970f), LL(0xb414145014a04428), LL(0xbae1e1a3e15b42df),
    LL(0xa616165816b04e2c), LL(0xf73a3ae83acdd274), LL(0x066969b9696fd0d2), LL(0x4109092409482d12),
    LL(0xd77070dd70a7ade0), LL(0x6fb6b6e2b6d95471), LL(0x1ed0d067d0ceb7bd), LL(0xd6eded93ed3b7ec7),
    LL(0xe2cccc17cc2edb85), LL(0x68424215422a5784), LL(0x2c98985a98b4c22d), LL(0xeda4a4aaa4490e55),
    LL(0x752828a0285d8850), LL(0x865c5c6d5cda31b8), LL(0x6bf8f8c7f8933fed), LL(0xc28686228644a411),
};

__constant__ static const u64 C2[256] = {
    LL(0x30d818186018c078), LL(0x462623238c2305af), LL(0x91b8c6c63fc67ef9), LL(0xcdfbe8e887e8136f),
    LL(0x13cb878726874ca1), LL(0x6d11b8b8dab8a962), LL(0x0209010104010805), LL(0x9e0d4f4f214f426e),
    LL(0x6c9b3636d836adee), LL(0x51ffa6a6a2a65904), LL(0xb90cd2d26fd2debd), LL(0xf70ef5f5f3f5fb06),
    LL(0xf2967979f979ef80), LL(0xde306f6fa16f5fce), LL(0x3f6d91917e91fcef), LL(0xa4f852525552aa07),
    LL(0xc04760609d6027fd), LL(0x6535bcbccabc8976), LL(0x2b379b9b569baccd), LL(0x018a8e8e028e048c),
    LL(0x5bd2a3a3b6a37115), LL(0x186c0c0c300c603c), LL(0xf6847b7bf17bff8a), LL(0x6a803535d435b5e1),
    LL(0x3af51d1d741de869), LL(0xddb3e0e0a7e05347), LL(0xb321d7d77bd7f6ac), LL(0x999cc2c22fc25eed),
    LL(0x5c432e2eb82e6d96), LL(0x96294b4b314b627a), LL(0xe15dfefedffea321), LL(0xaed5575741578216),
    LL(0x2abd15155415a841), LL(0xeee87777c1779fb6), LL(0x6e923737dc37a5eb), LL(0xd79ee5e5b3e57b56),
    LL(0x23139f9f469f8cd9), LL(0xfd23f0f0e7f0d317), LL(0x94204a4a354a6a7f), LL(0xa944dada4fda9e95),
    LL(0xb0a258587d58fa25), LL(0x8fcfc9c903c906ca), LL(0x527c2929a429558d), LL(0x145a0a0a280a5022),
    LL(0x7f50b1b1feb1e14f), LL(0x5dc9a0a0baa0691a), LL(0xd6146b6bb16b7fda), LL(0x17d985852e855cab),
    LL(0x673cbdbdcebd8173), LL(0xba8f5d5d695dd234), LL(0x2090101040108050), LL(0xf507f4f4f7f4f303),
    LL(0x8bddcbcb0bcb16c0), LL(0x7cd33e3ef83eedc6), LL(0x0a2d050514052811), LL(0xce78676781671fe6),
    LL(0xd597e4e4b7e47353), LL(0x4e0227279c2725bb), LL(0x8273414119413258), LL(0x0ba78b8b168b2c9d),
    LL(0x53f6a7a7a6a75101), LL(0xfab27d7de97dcf94), LL(0x374995956e95dcfb), LL(0xad56d8d847d88e9f),
    LL(0xeb70fbfbcbfb8b30), LL(0xc1cdeeee9fee2371), LL(0xf8bb7c7ced7cc791), LL(0xcc716666856617e3),
    LL(0xa77bdddd53dda68e), LL(0x2eaf17175c17b84b), LL(0x8e45474701470246), LL(0x211a9e9e429e84dc),
    LL(0x89d4caca0fca1ec5), LL(0x5a582d2db42d7599), LL(0x632ebfbfc6bf9179), LL(0x0e3f07071c07381b),
    LL(0x47acadad8ead0123), LL(0xb4b05a5a755aea2f), LL(0x1bef838336836cb5), LL(0x66b63333cc3385ff),
    LL(0xc65c636391633ff2), LL(0x041202020802100a), LL(0x4993aaaa92aa3938), LL(0xe2de7171d971afa8),
    LL(0x8dc6c8c807c80ecf), LL(0x32d119196419c87d), LL(0x923b494939497270), LL(0xaf5fd9d943d9869a),
    LL(0xf931f2f2eff2c31d), LL(0xdba8e3e3abe34b48), LL(0xb6b95b5b715be22a), LL(0x0dbc88881a883492),
    LL(0x293e9a9a529aa4c8), LL(0x4c0b262698262dbe), LL(0x64bf3232c8328dfa), LL(0x7d59b0b0fab0e94a),
    LL(0xcff2e9e983e91b6a), LL(0x1e770f0f3c0f7833), LL(0xb733d5d573d5e6a6), LL(0x1df480803a8074ba),
    LL(0x6127bebec2be997c), LL(0x87ebcdcd13cd26de), LL(0x68893434d034bde4), LL(0x903248483d487a75),
    LL(0xe354ffffdbffab24), LL(0xf48d7a7af57af78f), LL(0x3d6490907a90f4ea), LL(0xbe9d5f5f615fc23e),
    LL(0x403d202080201da0), LL(0xd00f6868bd6867d5), LL(0x34ca1a1a681ad072), LL(0x41b7aeae82ae192c),
    LL(0x757db4b4eab4c95e), LL(0xa8ce54544d549a19), LL(0x3b7f93937693ece5), LL(0x442f222288220daa),
    LL(0xc86364648d6407e9), LL(0xff2af1f1e3f1db12), LL(0xe6cc7373d173bfa2), LL(0x248212124812905a),
    LL(0x807a40401d403a5d), LL(0x1048080820084028), LL(0x9b95c3c32bc356e8), LL(0xc5dfecec97ec337b),
    LL(0xab4ddbdb4bdb9690), LL(0x5fc0a1a1bea1611f), LL(0x07918d8d0e8d1c83), LL(0x7ac83d3df43df5c9),
    LL(0x335b97976697ccf1), LL(0x0000000000000000), LL(0x83f9cfcf1bcf36d4), LL(0x566e2b2bac2b4587),
    LL(0xece17676c57697b3), LL(0x19e68282328264b0), LL(0xb128d6d67fd6fea9), LL(0x36c31b1b6c1bd877),
    LL(0x7774b5b5eeb5c15b), LL(0x43beafaf86af1129), LL(0xd41d6a6ab56a77df), LL(0xa0ea50505d50ba0d),
    LL(0x8a5745450945124c), LL(0xfb38f3f3ebf3cb18), LL(0x60ad3030c0309df0), LL(0xc3c4efef9bef2b74),
    LL(0x7eda3f3ffc3fe5c3), LL(0xaac755554955921c), LL(0x59dba2a2b2a27910), LL(0xc9e9eaea8fea0365),
    LL(0xca6a656589650fec), LL(0x6903babad2bab968), LL(0x5e4a2f2fbc2f6593), LL(0x9d8ec0c027c04ee7),
    LL(0xa160dede5fdebe81), LL(0x38fc1c1c701ce06c), LL(0xe746fdfdd3fdbb2e), LL(0x9a1f4d4d294d5264),
    LL(0x397692927292e4e0), LL(0xeafa7575c9758fbc), LL(0x0c3606061806301e), LL(0x09ae8a8a128a2498),
    LL(0x794bb2b2f2b2f940), LL(0xd185e6e6bfe66359), LL(0x1c7e0e0e380e7036), LL(0x3ee71f1f7c1ff863),
    LL(0xc4556262956237f7), LL(0xb53ad4d477d4eea3), LL(0x4d81a8a89aa82932), LL(0x315296966296c4f4),
    LL(0xef62f9f9c3f99b3a), LL(0x97a3c5c533c566f6), LL(0x4a102525942535b1), LL(0xb2ab59597959f220),
    LL(0x15d084842a8454ae), LL(0xe4c57272d572b7a7), LL(0x72ec3939e439d5dd), LL(0x98164c4c2d4c5a61),
    LL(0xbc945e5e655eca3b), LL(0xf09f7878fd78e785), LL(0x70e53838e038ddd8), LL(0x05988c8c0a8c1486),
    LL(0xbf17d1d163d1c6b2), LL(0x57e4a5a5aea5410b), LL(0xd9a1e2e2afe2434d), LL(0xc24e616199612ff8),
    LL(0x7b42b3b3f6b3f145), LL(0x42342121842115a5), LL(0x25089c9c4a9c94d6), LL(0x3cee1e1e781ef066),
    LL(0x8661434311432252), LL(0x93b1c7c73bc776fc), LL(0xe54ffcfcd7fcb32b), LL(0x0824040410042014),
    LL(0xa2e351515951b208), LL(0x2f2599995e99bcc7), LL(0xda226d6da96d4fc4), LL(0x1a650d0d340d6839),
    LL(0xe979fafacffa8335), LL(0xa369dfdf5bdfb684), LL(0xfca97e7ee57ed79b), LL(0x4819242490243db4),
    LL(0x76fe3b3bec3bc5d7), LL(0x4b9aabab96ab313d), LL(0x81f0cece1fce3ed1), LL(0x2299111144118855),
    LL(0x03838f8f068f0c89), LL(0x9c044e4e254e4a6b), LL(0x7366b7b7e6b7d151), LL(0xcbe0ebeb8beb0b60),
    LL(0x78c13c3cf03cfdcc), LL(0x1ffd81813e817cbf), LL(0x354094946a94d4fe), LL(0xf31cf7f7fbf7eb0c),
    LL(0x6f18b9b9deb9a167), LL(0x268b13134c13985f), LL(0x58512c2cb02c7d9c), LL(0xbb05d3d36bd3d6b8),
    LL(0xd38ce7e7bbe76b5c), LL(0xdc396e6ea56e57cb), LL(0x95aac4c437c46ef3), LL(0x061b03030c03180f),
    LL(0xacdc565645568a13), LL(0x885e44440d441a49), LL(0xfea07f7fe17fdf9e), LL(0x4f88a9a99ea92137),
    LL(0x54672a2aa82a4d82), LL(0x6b0abbbbd6bbb16d), LL(0x9f87c1c123c146e2), LL(0xa6f153535153a202),
    LL(0xa572dcdc57dcae8b), LL(0x16530b0b2c0b5827), LL(0x27019d9d4e9d9cd3), LL(0xd82b6c6cad6c47c1),
    LL(0x62a43131c43195f5), LL(0xe8f37474cd7487b9), LL(0xf115f6f6fff6e309), LL(0x8c4c464605460a43),
    LL(0x45a5acac8aac0926), LL(0x0fb589891e893c97), LL(0x28b414145014a044), LL(0xdfbae1e1a3e15b42),
    LL(0x2ca616165816b04e), LL(0x74f73a3ae83acdd2), LL(0xd2066969b9696fd0), LL(0x124109092409482d),
    LL(0xe0d77070dd70a7ad), LL(0x716fb6b6e2b6d954), LL(0xbd1ed0d067d0ceb7), LL(0xc7d6eded93ed3b7e),
    LL(0x85e2cccc17cc2edb), LL(0x8468424215422a57), LL(0x2d2c98985a98b4c2), LL(0x55eda4a4aaa4490e),
    LL(0x50752828a0285d88), LL(0xb8865c5c6d5cda31), LL(0xed6bf8f8c7f8933f), LL(0x11c28686228644a4),
};

__constant__ static const u64 C3[256] = {
    LL(0x7830d818186018c0), LL(0xaf462623238c2305), LL(0xf991b8c6c63fc67e), LL(0x6fcdfbe8e887e813),
    LL(0xa113cb878726874c), LL(0x626d11b8b8dab8a9), LL(0x0502090101040108), LL(0x6e9e0d4f4f214f42),
    LL(0xee6c9b3636d836ad), LL(0x0451ffa6a6a2a659), LL(0xbdb90cd2d26fd2de), LL(0x06f70ef5f5f3f5fb),
    LL(0x80f2967979f979ef), LL(0xcede306f6fa16f5f), LL(0xef3f6d91917e91fc), LL(0x07a4f852525552aa),
    LL(0xfdc04760609d6027), LL(0x766535bcbccabc89), LL(0xcd2b379b9b569bac), LL(0x8c018a8e8e028e04),
    LL(0x155bd2a3a3b6a371), LL(0x3c186c0c0c300c60), LL(0x8af6847b7bf17bff), LL(0xe16a803535d435b5),
    LL(0x693af51d1d741de8), LL(0x47ddb3e0e0a7e053), LL(0xacb321d7d77bd7f6), LL(0xed999cc2c22fc25e),
    LL(0x965c432e2eb82e6d), LL(0x7a96294b4b314b62), LL(0x21e15dfefedffea3), LL(0x16aed55757415782),
    LL(0x412abd15155415a8), LL(0xb6eee87777c1779f), LL(0xeb6e923737dc37a5), LL(0x56d79ee5e5b3e57b),
    LL(0xd923139f9f469f8c), LL(0x17fd23f0f0e7f0d3), LL(0x7f94204a4a354a6a), LL(0x95a944dada4fda9e),
    LL(0x25b0a258587d58fa), LL(0xca8fcfc9c903c906), LL(0x8d527c2929a42955), LL(0x22145a0a0a280a50),
    LL(0x4f7f50b1b1feb1e1), LL(0x1a5dc9a0a0baa069), LL(0xdad6146b6bb16b7f), LL(0xab17d985852e855c),
    LL(0x73673cbdbdcebd81), LL(0x34ba8f5d5d695dd2), LL(0x5020901010401080), LL(0x03f507f4f4f7f4f3),
    LL(0xc08bddcbcb0bcb16), LL(0xc67cd33e3ef83eed), LL(0x110a2d0505140528), LL(0xe6ce78676781671f),
    LL(0x53d597e4e4b7e473), LL(0xbb4e0227279c2725), LL(0x5882734141194132), LL(0x9d0ba78b8b168b2c),
    LL(0x0153f6a7a7a6a751), LL(0x94fab27d7de97dcf), LL(0xfb374995956e95dc), LL(0x9fad56d8d847d88e),
    LL(0x30eb70fbfbcbfb8b), LL(0x71c1cdeeee9fee23), LL(0x91f8bb7c7ced7cc7), LL(0xe3cc716666856617),
    LL(0x8ea77bdddd53dda6), LL(0x4b2eaf17175c17b8), LL(0x468e454747014702), LL(0xdc211a9e9e429e84),
    LL(0xc589d4caca0fca1e), LL(0x995a582d2db42d75), LL(0x79632ebfbfc6bf91), LL(0x1b0e3f07071c0738),
    LL(0x2347acadad8ead01), LL(0x2fb4b05a5a755aea), LL(0xb51bef838336836c), LL(0xff66b63333cc3385),
    LL(0xf2c65c636391633f), LL(0x0a04120202080210), LL(0x384993aaaa92aa39), LL(0xa8e2de7171d971af),
    LL(0xcf8dc6c8c807c80e), LL(0x7d32d119196419c8), LL(0x70923b4949394972), LL(0x9aaf5fd9d943d986),
    LL(0x1df931f2f2eff2c3), LL(0x48dba8e3e3abe34b), LL(0x2ab6b95b5b715be2), LL(0x920dbc88881a8834),
    LL(0xc8293e9a9a529aa4), LL(0xbe4c0b262698262d), LL(0xfa64bf3232c8328d), LL(0x4a7d59b0b0fab0e9),
    LL(0x6acff2e9e983e91b), LL(0x331e770f0f3c0f78), LL(0xa6b733d5d573d5e6), LL(0xba1df480803a8074),
    LL(0x7c6127bebec2be99), LL(0xde87ebcdcd13cd26), LL(0xe468893434d034bd), LL(0x75903248483d487a),
    LL(0x24e354ffffdbffab), LL(0x8ff48d7a7af57af7), LL(0xea3d6490907a90f4), LL(0x3ebe9d5f5f615fc2),
    LL(0xa0403d202080201d), LL(0xd5d00f6868bd6867), LL(0x7234ca1a1a681ad0), LL(0x2c41b7aeae82ae19),
    LL(0x5e757db4b4eab4c9), LL(0x19a8ce54544d549a), LL(0xe53b7f93937693ec), LL(0xaa442f222288220d),
    LL(0xe9c86364648d6407), LL(0x12ff2af1f1e3f1db), LL(0xa2e6cc7373d173bf), LL(0x5a24821212481290),
    LL(0x5d807a40401d403a), LL(0x2810480808200840), LL(0xe89b95c3c32bc356), LL(0x7bc5dfecec97ec33),
    LL(0x90ab4ddbdb4bdb96), LL(0x1f5fc0a1a1bea161), LL(0x8307918d8d0e8d1c), LL(0xc97ac83d3df43df5),
    LL(0xf1335b97976697cc), LL(0x0000000000000000), LL(0xd483f9cfcf1bcf36), LL(0x87566e2b2bac2b45),
    LL(0xb3ece17676c57697), LL(0xb019e68282328264), LL(0xa9b128d6d67fd6fe), LL(0x7736c31b1b6c1bd8),
    LL(0x5b7774b5b5eeb5c1), LL(0x2943beafaf86af11), LL(0xdfd41d6a6ab56a77), LL(0x0da0ea50505d50ba),
    LL(0x4c8a574545094512), LL(0x18fb38f3f3ebf3cb), LL(0xf060ad3030c0309d), LL(0x74c3c4efef9bef2b),
    LL(0xc37eda3f3ffc3fe5), LL(0x1caac75555495592), LL(0x1059dba2a2b2a279), LL(0x65c9e9eaea8fea03),
    LL(0xecca6a656589650f), LL(0x686903babad2bab9), LL(0x935e4a2f2fbc2f65), LL(0xe79d8ec0c027c04e),
    LL(0x81a160dede5fdebe), LL(0x6c38fc1c1c701ce0), LL(0x2ee746fdfdd3fdbb), LL(0x649a1f4d4d294d52),
    LL(0xe0397692927292e4), LL(0xbceafa7575c9758f), LL(0x1e0c360606180630), LL(0x9809ae8a8a128a24),
    LL(0x40794bb2b2f2b2f9), LL(0x59d185e6e6bfe663), LL(0x361c7e0e0e380e70), LL(0x633ee71f1f7c1ff8),
    LL(0xf7c4556262956237), LL(0xa3b53ad4d477d4ee), LL(0x324d81a8a89aa829), LL(0xf4315296966296c4),
    LL(0x3aef62f9f9c3f99b), LL(0xf697a3c5c533c566), LL(0xb14a102525942535), LL(0x20b2ab59597959f2),
    LL(0xae15d084842a8454), LL(0xa7e4c57272d572b7), LL(0xdd72ec3939e439d5), LL(0x6198164c4c2d4c5a),
    LL(0x3bbc945e5e655eca), LL(0x85f09f7878fd78e7), LL(0xd870e53838e038dd), LL(0x8605988c8c0a8c14),
    LL(0xb2bf17d1d163d1c6), LL(0x0b57e4a5a5aea541), LL(0x4dd9a1e2e2afe243), LL(0xf8c24e616199612f),
    LL(0x457b42b3b3f6b3f1), LL(0xa542342121842115), LL(0xd625089c9c4a9c94), LL(0x663cee1e1e781ef0),
    LL(0x5286614343114322), LL(0xfc93b1c7c73bc776), LL(0x2be54ffcfcd7fcb3), LL(0x1408240404100420),
    LL(0x08a2e351515951b2), LL(0xc72f2599995e99bc), LL(0xc4da226d6da96d4f), LL(0x391a650d0d340d68),
    LL(0x35e979fafacffa83), LL(0x84a369dfdf5bdfb6), LL(0x9bfca97e7ee57ed7), LL(0xb44819242490243d),
    LL(0xd776fe3b3bec3bc5), LL(0x3d4b9aabab96ab31), LL(0xd181f0cece1fce3e), LL(0x5522991111441188),
    LL(0x8903838f8f068f0c), LL(0x6b9c044e4e254e4a), LL(0x517366b7b7e6b7d1), LL(0x60cbe0ebeb8beb0b),
    LL(0xcc78c13c3cf03cfd), LL(0xbf1ffd81813e817c), LL(0xfe354094946a94d4), LL(0x0cf31cf7f7fbf7eb),
    LL(0x676f18b9b9deb9a1), LL(0x5f268b13134c1398), LL(0x9c58512c2cb02c7d), LL(0xb8bb05d3d36bd3d6),
    LL(0x5cd38ce7e7bbe76b), LL(0xcbdc396e6ea56e57), LL(0xf395aac4c437c46e), LL(0x0f061b03030c0318),
    LL(0x13acdc565645568a), LL(0x49885e44440d441a), LL(0x9efea07f7fe17fdf), LL(0x374f88a9a99ea921),
    LL(0x8254672a2aa82a4d), LL(0x6d6b0abbbbd6bbb1), LL(0xe29f87c1c123c146), LL(0x02a6f153535153a2),
    LL(0x8ba572dcdc57dcae), LL(0x2716530b0b2c0b58), LL(0xd327019d9d4e9d9c), LL(0xc1d82b6c6cad6c47),
    LL(0xf562a43131c43195), LL(0xb9e8f37474cd7487), LL(0x09f115f6f6fff6e3), LL(0x438c4c464605460a),
    LL(0x2645a5acac8aac09), LL(0x970fb589891e893c), LL(0x4428b414145014a0), LL(0x42dfbae1e1a3e15b),
    LL(0x4e2ca616165816b0), LL(0xd274f73a3ae83acd), LL(0xd0d2066969b9696f), LL(0x2d12410909240948),
    LL(0xade0d77070dd70a7), LL(0x54716fb6b6e2b6d9), LL(0xb7bd1ed0d067d0ce), LL(0x7ec7d6eded93ed3b),
    LL(0xdb85e2cccc17cc2e), LL(0x578468424215422a), LL(0xc22d2c98985a98b4), LL(0x0e55eda4a4aaa449),
    LL(0x8850752828a0285d), LL(0x31b8865c5c6d5cda), LL(0x3fed6bf8f8c7f893), LL(0xa411c28686228644),
};

__constant__ static const u64 C4[256] = {
    LL(0xc07830d818186018), LL(0x05af462623238c23), LL(0x7ef991b8c6c63fc6), LL(0x136fcdfbe8e887e8),
    LL(0x4ca113cb87872687), LL(0xa9626d11b8b8dab8), LL(0x0805020901010401), LL(0x426e9e0d4f4f214f),
    LL(0xadee6c9b3636d836), LL(0x590451ffa6a6a2a6), LL(0xdebdb90cd2d26fd2), LL(0xfb06f70ef5f5f3f5),
    LL(0xef80f2967979f979), LL(0x5fcede306f6fa16f), LL(0xfcef3f6d91917e91), LL(0xaa07a4f852525552),
    LL(0x27fdc04760609d60), LL(0x89766535bcbccabc), LL(0xaccd2b379b9b569b), LL(0x048c018a8e8e028e),
    LL(0x71155bd2a3a3b6a3), LL(0x603c186c0c0c300c), LL(0xff8af6847b7bf17b), LL(0xb5e16a803535d435),
    LL(0xe8693af51d1d741d), LL(0x5347ddb3e0e0a7e0), LL(0xf6acb321d7d77bd7), LL(0x5eed999cc2c22fc2),
    LL(0x6d965c432e2eb82e), LL(0x627a96294b4b314b), LL(0xa321e15dfefedffe), LL(0x8216aed557574157),
    LL(0xa8412abd15155415), LL(0x9fb6eee87777c177), LL(0xa5eb6e923737dc37), LL(0x7b56d79ee5e5b3e5),
    LL(0x8cd923139f9f469f), LL(0xd317fd23f0f0e7f0), LL(0x6a7f94204a4a354a), LL(0x9e95a944dada4fda),
    LL(0xfa25b0a258587d58), LL(0x06ca8fcfc9c903c9), LL(0x558d527c2929a429), LL(0x5022145a0a0a280a),
    LL(0xe14f7f50b1b1feb1), LL(0x691a5dc9a0a0baa0), LL(0x7fdad6146b6bb16b), LL(0x5cab17d985852e85),
    LL(0x8173673cbdbdcebd), LL(0xd234ba8f5d5d695d), LL(0x8050209010104010), LL(0xf303f507f4f4f7f4),
    LL(0x16c08bddcbcb0bcb), LL(0xedc67cd33e3ef83e), LL(0x28110a2d05051405), LL(0x1fe6ce7867678167),
    LL(0x7353d597e4e4b7e4), LL(0x25bb4e0227279c27), LL(0x3258827341411941), LL(0x2c9d0ba78b8b168b),
    LL(0x510153f6a7a7a6a7), LL(0xcf94fab27d7de97d), LL(0xdcfb374995956e95), LL(0x8e9fad56d8d847d8),
    LL(0x8b30eb70fbfbcbfb), LL(0x2371c1cdeeee9fee), LL(0xc791f8bb7c7ced7c), LL(0x17e3cc7166668566),
    LL(0xa68ea77bdddd53dd), LL(0xb84b2eaf17175c17), LL(0x02468e4547470147), LL(0x84dc211a9e9e429e),
    LL(0x1ec589d4caca0fca), LL(0x75995a582d2db42d), LL(0x9179632ebfbfc6bf), LL(0x381b0e3f07071c07),
    LL(0x012347acadad8ead), LL(0xea2fb4b05a5a755a), LL(0x6cb51bef83833683), LL(0x85ff66b63333cc33),
    LL(0x3ff2c65c63639163), LL(0x100a041202020802), LL(0x39384993aaaa92aa), LL(0xafa8e2de7171d971),
    LL(0x0ecf8dc6c8c807c8), LL(0xc87d32d119196419), LL(0x7270923b49493949), LL(0x869aaf5fd9d943d9),
    LL(0xc31df931f2f2eff2), LL(0x4b48dba8e3e3abe3), LL(0xe22ab6b95b5b715b), LL(0x34920dbc88881a88),
    LL(0xa4c8293e9a9a529a), LL(0x2dbe4c0b26269826), LL(0x8dfa64bf3232c832), LL(0xe94a7d59b0b0fab0),
    LL(0x1b6acff2e9e983e9), LL(0x78331e770f0f3c0f), LL(0xe6a6b733d5d573d5), LL(0x74ba1df480803a80),
    LL(0x997c6127bebec2be), LL(0x26de87ebcdcd13cd), LL(0xbde468893434d034), LL(0x7a75903248483d48),
    LL(0xab24e354ffffdbff), LL(0xf78ff48d7a7af57a), LL(0xf4ea3d6490907a90), LL(0xc23ebe9d5f5f615f),
    LL(0x1da0403d20208020), LL(0x67d5d00f6868bd68), LL(0xd07234ca1a1a681a), LL(0x192c41b7aeae82ae),
    LL(0xc95e757db4b4eab4), LL(0x9a19a8ce54544d54), LL(0xece53b7f93937693), LL(0x0daa442f22228822),
    LL(0x07e9c86364648d64), LL(0xdb12ff2af1f1e3f1), LL(0xbfa2e6cc7373d173), LL(0x905a248212124812),
    LL(0x3a5d807a40401d40), LL(0x4028104808082008), LL(0x56e89b95c3c32bc3), LL(0x337bc5dfecec97ec),
    LL(0x9690ab4ddbdb4bdb), LL(0x611f5fc0a1a1bea1), LL(0x1c8307918d8d0e8d), LL(0xf5c97ac83d3df43d),
    LL(0xccf1335b97976697), LL(0x0000000000000000), LL(0x36d483f9cfcf1bcf), LL(0x4587566e2b2bac2b),
    LL(0x97b3ece17676c576), LL(0x64b019e682823282), LL(0xfea9b128d6d67fd6), LL(0xd87736c31b1b6c1b),
    LL(0xc15b7774b5b5eeb5), LL(0x112943beafaf86af), LL(0x77dfd41d6a6ab56a), LL(0xba0da0ea50505d50),
    LL(0x124c8a5745450945), LL(0xcb18fb38f3f3ebf3), LL(0x9df060ad3030c030), LL(0x2b74c3c4efef9bef),
    LL(0xe5c37eda3f3ffc3f), LL(0x921caac755554955), LL(0x791059dba2a2b2a2), LL(0x0365c9e9eaea8fea),
    LL(0x0fecca6a65658965), LL(0xb9686903babad2ba), LL(0x65935e4a2f2fbc2f), LL(0x4ee79d8ec0c027c0),
    LL(0xbe81a160dede5fde), LL(0xe06c38fc1c1c701c), LL(0xbb2ee746fdfdd3fd), LL(0x52649a1f4d4d294d),
    LL(0xe4e0397692927292), LL(0x8fbceafa7575c975), LL(0x301e0c3606061806), LL(0x249809ae8a8a128a),
    LL(0xf940794bb2b2f2b2), LL(0x6359d185e6e6bfe6), LL(0x70361c7e0e0e380e), LL(0xf8633ee71f1f7c1f),
    LL(0x37f7c45562629562), LL(0xeea3b53ad4d477d4), LL(0x29324d81a8a89aa8), LL(0xc4f4315296966296),
    LL(0x9b3aef62f9f9c3f9), LL(0x66f697a3c5c533c5), LL(0x35b14a1025259425), LL(0xf220b2ab59597959),
    LL(0x54ae15d084842a84), LL(0xb7a7e4c57272d572), LL(0xd5dd72ec3939e439), LL(0x5a6198164c4c2d4c),
    LL(0xca3bbc945e5e655e), LL(0xe785f09f7878fd78), LL(0xddd870e53838e038), LL(0x148605988c8c0a8c),
    LL(0xc6b2bf17d1d163d1), LL(0x410b57e4a5a5aea5), LL(0x434dd9a1e2e2afe2), LL(0x2ff8c24e61619961),
    LL(0xf1457b42b3b3f6b3), LL(0x15a5423421218421), LL(0x94d625089c9c4a9c), LL(0xf0663cee1e1e781e),
    LL(0x2252866143431143), LL(0x76fc93b1c7c73bc7), LL(0xb32be54ffcfcd7fc), LL(0x2014082404041004),
    LL(0xb208a2e351515951), LL(0xbcc72f2599995e99), LL(0x4fc4da226d6da96d), LL(0x68391a650d0d340d),
    LL(0x8335e979fafacffa), LL(0xb684a369dfdf5bdf), LL(0xd79bfca97e7ee57e), LL(0x3db4481924249024),
    LL(0xc5d776fe3b3bec3b), LL(0x313d4b9aabab96ab), LL(0x3ed181f0cece1fce), LL(0x8855229911114411),
    LL(0x0c8903838f8f068f), LL(0x4a6b9c044e4e254e), LL(0xd1517366b7b7e6b7), LL(0x0b60cbe0ebeb8beb),
    LL(0xfdcc78c13c3cf03c), LL(0x7cbf1ffd81813e81), LL(0xd4fe354094946a94), LL(0xeb0cf31cf7f7fbf7),
    LL(0xa1676f18b9b9deb9), LL(0x985f268b13134c13), LL(0x7d9c58512c2cb02c), LL(0xd6b8bb05d3d36bd3),
    LL(0x6b5cd38ce7e7bbe7), LL(0x57cbdc396e6ea56e), LL(0x6ef395aac4c437c4), LL(0x180f061b03030c03),
    LL(0x8a13acdc56564556), LL(0x1a49885e44440d44), LL(0xdf9efea07f7fe17f), LL(0x21374f88a9a99ea9),
    LL(0x4d8254672a2aa82a), LL(0xb16d6b0abbbbd6bb), LL(0x46e29f87c1c123c1), LL(0xa202a6f153535153),
    LL(0xae8ba572dcdc57dc), LL(0x582716530b0b2c0b), LL(0x9cd327019d9d4e9d), LL(0x47c1d82b6c6cad6c),
    LL(0x95f562a43131c431), LL(0x87b9e8f37474cd74), LL(0xe309f115f6f6fff6), LL(0x0a438c4c46460546),
    LL(0x092645a5acac8aac), LL(0x3c970fb589891e89), LL(0xa04428b414145014), LL(0x5b42dfbae1e1a3e1),
    LL(0xb04e2ca616165816), LL(0xcdd274f73a3ae83a), LL(0x6fd0d2066969b969), LL(0x482d124109092409),
    LL(0xa7ade0d77070dd70), LL(0xd954716fb6b6e2b6), LL(0xceb7bd1ed0d067d0), LL(0x3b7ec7d6eded93ed),
    LL(0x2edb85e2cccc17cc), LL(0x2a57846842421542), LL(0xb4c22d2c98985a98), LL(0x490e55eda4a4aaa4),
    LL(0x5d8850752828a028), LL(0xda31b8865c5c6d5c), LL(0x933fed6bf8f8c7f8), LL(0x44a411c286862286),
};

__constant__ static const u64 C5[256] = {
    LL(0x18c07830d8181860), LL(0x2305af462623238c), LL(0xc67ef991b8c6c63f), LL(0xe8136fcdfbe8e887),
    LL(0x874ca113cb878726), LL(0xb8a9626d11b8b8da), LL(0x0108050209010104), LL(0x4f426e9e0d4f4f21),
    LL(0x36adee6c9b3636d8), LL(0xa6590451ffa6a6a2), LL(0xd2debdb90cd2d26f), LL(0xf5fb06f70ef5f5f3),
    LL(0x79ef80f2967979f9), LL(0x6f5fcede306f6fa1), LL(0x91fcef3f6d91917e), LL(0x52aa07a4f8525255),
    LL(0x6027fdc04760609d), LL(0xbc89766535bcbcca), LL(0x9baccd2b379b9b56), LL(0x8e048c018a8e8e02),
    LL(0xa371155bd2a3a3b6), LL(0x0c603c186c0c0c30), LL(0x7bff8af6847b7bf1), LL(0x35b5e16a803535d4),
    LL(0x1de8693af51d1d74), LL(0xe05347ddb3e0e0a7), LL(0xd7f6acb321d7d77b), LL(0xc25eed999cc2c22f),
    LL(0x2e6d965c432e2eb8), LL(0x4b627a96294b4b31), LL(0xfea321e15dfefedf), LL(0x578216aed5575741),
    LL(0x15a8412abd151554), LL(0x779fb6eee87777c1), LL(0x37a5eb6e923737dc), LL(0xe57b56d79ee5e5b3),
    LL(0x9f8cd923139f9f46), LL(0xf0d317fd23f0f0e7), LL(0x4a6a7f94204a4a35), LL(0xda9e95a944dada4f),
    LL(0x58fa25b0a258587d), LL(0xc906ca8fcfc9c903), LL(0x29558d527c2929a4), LL(0x0a5022145a0a0a28),
    LL(0xb1e14f7f50b1b1fe), LL(0xa0691a5dc9a0a0ba), LL(0x6b7fdad6146b6bb1), LL(0x855cab17d985852e),
    LL(0xbd8173673cbdbdce), LL(0x5dd234ba8f5d5d69), LL(0x1080502090101040), LL(0xf4f303f507f4f4f7),
    LL(0xcb16c08bddcbcb0b), LL(0x3eedc67cd33e3ef8), LL(0x0528110a2d050514), LL(0x671fe6ce78676781),
    LL(0xe47353d597e4e4b7), LL(0x2725bb4e0227279c), LL(0x4132588273414119), LL(0x8b2c9d0ba78b8b16),
    LL(0xa7510153f6a7a7a6), LL(0x7dcf94fab27d7de9), LL(0x95dcfb374995956e), LL(0xd88e9fad56d8d847),
    LL(0xfb8b30eb70fbfbcb), LL(0xee2371c1cdeeee9f), LL(0x7cc791f8bb7c7ced), LL(0x6617e3cc71666685),
    LL(0xdda68ea77bdddd53), LL(0x17b84b2eaf17175c), LL(0x4702468e45474701), LL(0x9e84dc211a9e9e42),
    LL(0xca1ec589d4caca0f), LL(0x2d75995a582d2db4), LL(0xbf9179632ebfbfc6), LL(0x07381b0e3f07071c),
    LL(0xad012347acadad8e), LL(0x5aea2fb4b05a5a75), LL(0x836cb51bef838336), LL(0x3385ff66b63333cc),
    LL(0x633ff2c65c636391), LL(0x02100a0412020208), LL(0xaa39384993aaaa92), LL(0x71afa8e2de7171d9),
    LL(0xc80ecf8dc6c8c807), LL(0x19c87d32d1191964), LL(0x497270923b494939), LL(0xd9869aaf5fd9d943),
    LL(0xf2c31df931f2f2ef), LL(0xe34b48dba8e3e3ab), LL(0x5be22ab6b95b5b71), LL(0x8834920dbc88881a),
    LL(0x9aa4c8293e9a9a52), LL(0x262dbe4c0b262698), LL(0x328dfa64bf3232c8), LL(0xb0e94a7d59b0b0fa),
    LL(0xe91b6acff2e9e983), LL(0x0f78331e770f0f3c), LL(0xd5e6a6b733d5d573), LL(0x8074ba1df480803a),
    LL(0xbe997c6127bebec2), LL(0xcd26de87ebcdcd13), LL(0x34bde468893434d0), LL(0x487a75903248483d),
    LL(0xffab24e354ffffdb), LL(0x7af78ff48d7a7af5), LL(0x90f4ea3d6490907a), LL(0x5fc23ebe9d5f5f61),
    LL(0x201da0403d202080), LL(0x6867d5d00f6868bd), LL(0x1ad07234ca1a1a68), LL(0xae192c41b7aeae82),
    LL(0xb4c95e757db4b4ea), LL(0x549a19a8ce54544d), LL(0x93ece53b7f939376), LL(0x220daa442f222288),
    LL(0x6407e9c86364648d), LL(0xf1db12ff2af1f1e3), LL(0x73bfa2e6cc7373d1), LL(0x12905a2482121248),
    LL(0x403a5d807a40401d), LL(0x0840281048080820), LL(0xc356e89b95c3c32b), LL(0xec337bc5dfecec97),
    LL(0xdb9690ab4ddbdb4b), LL(0xa1611f5fc0a1a1be), LL(0x8d1c8307918d8d0e), LL(0x3df5c97ac83d3df4),
    LL(0x97ccf1335b979766), LL(0x0000000000000000), LL(0xcf36d483f9cfcf1b), LL(0x2b4587566e2b2bac),
    LL(0x7697b3ece17676c5), LL(0x8264b019e6828232), LL(0xd6fea9b128d6d67f), LL(0x1bd87736c31b1b6c),
    LL(0xb5c15b7774b5b5ee), LL(0xaf112943beafaf86), LL(0x6a77dfd41d6a6ab5), LL(0x50ba0da0ea50505d),
    LL(0x45124c8a57454509), LL(0xf3cb18fb38f3f3eb), LL(0x309df060ad3030c0), LL(0xef2b74c3c4efef9b),
    LL(0x3fe5c37eda3f3ffc), LL(0x55921caac7555549), LL(0xa2791059dba2a2b2), LL(0xea0365c9e9eaea8f),
    LL(0x650fecca6a656589), LL(0xbab9686903babad2), LL(0x2f65935e4a2f2fbc), LL(0xc04ee79d8ec0c027),
    LL(0xdebe81a160dede5f), LL(0x1ce06c38fc1c1c70), LL(0xfdbb2ee746fdfdd3), LL(0x4d52649a1f4d4d29),
    LL(0x92e4e03976929272), LL(0x758fbceafa7575c9), LL(0x06301e0c36060618), LL(0x8a249809ae8a8a12),
    LL(0xb2f940794bb2b2f2), LL(0xe66359d185e6e6bf), LL(0x0e70361c7e0e0e38), LL(0x1ff8633ee71f1f7c),
    LL(0x6237f7c455626295), LL(0xd4eea3b53ad4d477), LL(0xa829324d81a8a89a), LL(0x96c4f43152969662),
    LL(0xf99b3aef62f9f9c3), LL(0xc566f697a3c5c533), LL(0x2535b14a10252594), LL(0x59f220b2ab595979),
    LL(0x8454ae15d084842a), LL(0x72b7a7e4c57272d5), LL(0x39d5dd72ec3939e4), LL(0x4c5a6198164c4c2d),
    LL(0x5eca3bbc945e5e65), LL(0x78e785f09f7878fd), LL(0x38ddd870e53838e0), LL(0x8c148605988c8c0a),
    LL(0xd1c6b2bf17d1d163), LL(0xa5410b57e4a5a5ae), LL(0xe2434dd9a1e2e2af), LL(0x612ff8c24e616199),
    LL(0xb3f1457b42b3b3f6), LL(0x2115a54234212184), LL(0x9c94d625089c9c4a), LL(0x1ef0663cee1e1e78),
    LL(0x4322528661434311), LL(0xc776fc93b1c7c73b), LL(0xfcb32be54ffcfcd7), LL(0x0420140824040410),
    LL(0x51b208a2e3515159), LL(0x99bcc72f2599995e), LL(0x6d4fc4da226d6da9), LL(0x0d68391a650d0d34),
    LL(0xfa8335e979fafacf), LL(0xdfb684a369dfdf5b), LL(0x7ed79bfca97e7ee5), LL(0x243db44819242490),
    LL(0x3bc5d776fe3b3bec), LL(0xab313d4b9aabab96), LL(0xce3ed181f0cece1f), LL(0x1188552299111144),
    LL(0x8f0c8903838f8f06), LL(0x4e4a6b9c044e4e25), LL(0xb7d1517366b7b7e6), LL(0xeb0b60cbe0ebeb8b),
    LL(0x3cfdcc78c13c3cf0), LL(0x817cbf1ffd81813e), LL(0x94d4fe354094946a), LL(0xf7eb0cf31cf7f7fb),
    LL(0xb9a1676f18b9b9de), LL(0x13985f268b13134c), LL(0x2c7d9c58512c2cb0), LL(0xd3d6b8bb05d3d36b),
    LL(0xe76b5cd38ce7e7bb), LL(0x6e57cbdc396e6ea5), LL(0xc46ef395aac4c437), LL(0x03180f061b03030c),
    LL(0x568a13acdc565645), LL(0x441a49885e44440d), LL(0x7fdf9efea07f7fe1), LL(0xa921374f88a9a99e),
    LL(0x2a4d8254672a2aa8), LL(0xbbb16d6b0abbbbd6), LL(0xc146e29f87c1c123), LL(0x53a202a6f1535351),
    LL(0xdcae8ba572dcdc57), LL(0x0b582716530b0b2c), LL(0x9d9cd327019d9d4e), LL(0x6c47c1d82b6c6cad),
    LL(0x3195f562a43131c4), LL(0x7487b9e8f37474cd), LL(0xf6e309f115f6f6ff), LL(0x460a438c4c464605),
    LL(0xac092645a5acac8a), LL(0x893c970fb589891e), LL(0x14a04428b4141450), LL(0xe15b42dfbae1e1a3),
    LL(0x16b04e2ca6161658), LL(0x3acdd274f73a3ae8), LL(0x696fd0d2066969b9), LL(0x09482d1241090924),
    LL(0x70a7ade0d77070dd), LL(0xb6d954716fb6b6e2), LL(0xd0ceb7bd1ed0d067), LL(0xed3b7ec7d6eded93),
    LL(0xcc2edb85e2cccc17), LL(0x422a578468424215), LL(0x98b4c22d2c98985a), LL(0xa4490e55eda4a4aa),
    LL(0x285d8850752828a0), LL(0x5cda31b8865c5c6d), LL(0xf8933fed6bf8f8c7), LL(0x8644a411c2868622),
};

__constant__ static const u64 C6[256] = {
    LL(0x6018c07830d81818), LL(0x8c2305af46262323), LL(0x3fc67ef991b8c6c6), LL(0x87e8136fcdfbe8e8),
    LL(0x26874ca113cb8787), LL(0xdab8a9626d11b8b8), LL(0x0401080502090101), LL(0x214f426e9e0d4f4f),
    LL(0xd836adee6c9b3636), LL(0xa2a6590451ffa6a6), LL(0x6fd2debdb90cd2d2), LL(0xf3f5fb06f70ef5f5),
    LL(0xf979ef80f2967979), LL(0xa16f5fcede306f6f), LL(0x7e91fcef3f6d9191), LL(0x5552aa07a4f85252),
    LL(0x9d6027fdc0476060), LL(0xcabc89766535bcbc), LL(0x569baccd2b379b9b), LL(0x028e048c018a8e8e),
    LL(0xb6a371155bd2a3a3), LL(0x300c603c186c0c0c), LL(0xf17bff8af6847b7b), LL(0xd435b5e16a803535),
    LL(0x741de8693af51d1d), LL(0xa7e05347ddb3e0e0), LL(0x7bd7f6acb321d7d7), LL(0x2fc25eed999cc2c2),
    LL(0xb82e6d965c432e2e), LL(0x314b627a96294b4b), LL(0xdffea321e15dfefe), LL(0x41578216aed55757),
    LL(0x5415a8412abd1515), LL(0xc1779fb6eee87777), LL(0xdc37a5eb6e923737), LL(0xb3e57b56d79ee5e5),
    LL(0x469f8cd923139f9f), LL(0xe7f0d317fd23f0f0), LL(0x354a6a7f94204a4a), LL(0x4fda9e95a944dada),
    LL(0x7d58fa25b0a25858), LL(0x03c906ca8fcfc9c9), LL(0xa429558d527c2929), LL(0x280a5022145a0a0a),
    LL(0xfeb1e14f7f50b1b1), LL(0xbaa0691a5dc9a0a0), LL(0xb16b7fdad6146b6b), LL(0x2e855cab17d98585),
    LL(0xcebd8173673cbdbd), LL(0x695dd234ba8f5d5d), LL(0x4010805020901010), LL(0xf7f4f303f507f4f4),
    LL(0x0bcb16c08bddcbcb), LL(0xf83eedc67cd33e3e), LL(0x140528110a2d0505), LL(0x81671fe6ce786767),
    LL(0xb7e47353d597e4e4), LL(0x9c2725bb4e022727), LL(0x1941325882734141), LL(0x168b2c9d0ba78b8b),
    LL(0xa6a7510153f6a7a7), LL(0xe97dcf94fab27d7d), LL(0x6e95dcfb37499595), LL(0x47d88e9fad56d8d8),
    LL(0xcbfb8b30eb70fbfb), LL(0x9fee2371c1cdeeee), LL(0xed7cc791f8bb7c7c), LL(0x856617e3cc716666),
    LL(0x53dda68ea77bdddd), LL(0x5c17b84b2eaf1717), LL(0x014702468e454747), LL(0x429e84dc211a9e9e),
    LL(0x0fca1ec589d4caca), LL(0xb42d75995a582d2d), LL(0xc6bf9179632ebfbf), LL(0x1c07381b0e3f0707),
    LL(0x8ead012347acadad), LL(0x755aea2fb4b05a5a), LL(0x36836cb51bef8383), LL(0xcc3385ff66b63333),
    LL(0x91633ff2c65c6363), LL(0x0802100a04120202), LL(0x92aa39384993aaaa), LL(0xd971afa8e2de7171),
    LL(0x07c80ecf8dc6c8c8), LL(0x6419c87d32d11919), LL(0x39497270923b4949), LL(0x43d9869aaf5fd9d9),
    LL(0xeff2c31df931f2f2), LL(0xabe34b48dba8e3e3), LL(0x715be22ab6b95b5b), LL(0x1a8834920dbc8888),
    LL(0x529aa4c8293e9a9a), LL(0x98262dbe4c0b2626), LL(0xc8328dfa64bf3232), LL(0xfab0e94a7d59b0b0),
    LL(0x83e91b6acff2e9e9), LL(0x3c0f78331e770f0f), LL(0x73d5e6a6b733d5d5), LL(0x3a8074ba1df48080),
    LL(0xc2be997c6127bebe), LL(0x13cd26de87ebcdcd), LL(0xd034bde468893434), LL(0x3d487a7590324848),
    LL(0xdbffab24e354ffff), LL(0xf57af78ff48d7a7a), LL(0x7a90f4ea3d649090), LL(0x615fc23ebe9d5f5f),
    LL(0x80201da0403d2020), LL(0xbd6867d5d00f6868), LL(0x681ad07234ca1a1a), LL(0x82ae192c41b7aeae),
    LL(0xeab4c95e757db4b4), LL(0x4d549a19a8ce5454), LL(0x7693ece53b7f9393), LL(0x88220daa442f2222),
    LL(0x8d6407e9c8636464), LL(0xe3f1db12ff2af1f1), LL(0xd173bfa2e6cc7373), LL(0x4812905a24821212),
    LL(0x1d403a5d807a4040), LL(0x2008402810480808), LL(0x2bc356e89b95c3c3), LL(0x97ec337bc5dfecec),
    LL(0x4bdb9690ab4ddbdb), LL(0xbea1611f5fc0a1a1), LL(0x0e8d1c8307918d8d), LL(0xf43df5c97ac83d3d),
    LL(0x6697ccf1335b9797), LL(0x0000000000000000), LL(0x1bcf36d483f9cfcf), LL(0xac2b4587566e2b2b),
    LL(0xc57697b3ece17676), LL(0x328264b019e68282), LL(0x7fd6fea9b128d6d6), LL(0x6c1bd87736c31b1b),
    LL(0xeeb5c15b7774b5b5), LL(0x86af112943beafaf), LL(0xb56a77dfd41d6a6a), LL(0x5d50ba0da0ea5050),
    LL(0x0945124c8a574545), LL(0xebf3cb18fb38f3f3), LL(0xc0309df060ad3030), LL(0x9bef2b74c3c4efef),
    LL(0xfc3fe5c37eda3f3f), LL(0x4955921caac75555), LL(0xb2a2791059dba2a2), LL(0x8fea0365c9e9eaea),
    LL(0x89650fecca6a6565), LL(0xd2bab9686903baba), LL(0xbc2f65935e4a2f2f), LL(0x27c04ee79d8ec0c0),
    LL(0x5fdebe81a160dede), LL(0x701ce06c38fc1c1c), LL(0xd3fdbb2ee746fdfd), LL(0x294d52649a1f4d4d),
    LL(0x7292e4e039769292), LL(0xc9758fbceafa7575), LL(0x1806301e0c360606), LL(0x128a249809ae8a8a),
    LL(0xf2b2f940794bb2b2), LL(0xbfe66359d185e6e6), LL(0x380e70361c7e0e0e), LL(0x7c1ff8633ee71f1f),
    LL(0x956237f7c4556262), LL(0x77d4eea3b53ad4d4), LL(0x9aa829324d81a8a8), LL(0x6296c4f431529696),
    LL(0xc3f99b3aef62f9f9), LL(0x33c566f697a3c5c5), LL(0x942535b14a102525), LL(0x7959f220b2ab5959),
    LL(0x2a8454ae15d08484), LL(0xd572b7a7e4c57272), LL(0xe439d5dd72ec3939), LL(0x2d4c5a6198164c4c),
    LL(0x655eca3bbc945e5e), LL(0xfd78e785f09f7878), LL(0xe038ddd870e53838), LL(0x0a8c148605988c8c),
    LL(0x63d1c6b2bf17d1d1), LL(0xaea5410b57e4a5a5), LL(0xafe2434dd9a1e2e2), LL(0x99612ff8c24e6161),
    LL(0xf6b3f1457b42b3b3), LL(0x842115a542342121), LL(0x4a9c94d625089c9c), LL(0x781ef0663cee1e1e),
    LL(0x1143225286614343), LL(0x3bc776fc93b1c7c7), LL(0xd7fcb32be54ffcfc), LL(0x1004201408240404),
    LL(0x5951b208a2e35151), LL(0x5e99bcc72f259999), LL(0xa96d4fc4da226d6d), LL(0x340d68391a650d0d),
    LL(0xcffa8335e979fafa), LL(0x5bdfb684a369dfdf), LL(0xe57ed79bfca97e7e), LL(0x90243db448192424),
    LL(0xec3bc5d776fe3b3b), LL(0x96ab313d4b9aabab), LL(0x1fce3ed181f0cece), LL(0x4411885522991111),
    LL(0x068f0c8903838f8f), LL(0x254e4a6b9c044e4e), LL(0xe6b7d1517366b7b7), LL(0x8beb0b60cbe0ebeb),
    LL(0xf03cfdcc78c13c3c), LL(0x3e817cbf1ffd8181), LL(0x6a94d4fe35409494), LL(0xfbf7eb0cf31cf7f7),
    LL(0xdeb9a1676f18b9b9), LL(0x4c13985f268b1313), LL(0xb02c7d9c58512c2c), LL(0x6bd3d6b8bb05d3d3),
    LL(0xbbe76b5cd38ce7e7), LL(0xa56e57cbdc396e6e), LL(0x37c46ef395aac4c4), LL(0x0c03180f061b0303),
    LL(0x45568a13acdc5656), LL(0x0d441a49885e4444), LL(0xe17fdf9efea07f7f), LL(0x9ea921374f88a9a9),
    LL(0xa82a4d8254672a2a), LL(0xd6bbb16d6b0abbbb), LL(0x23c146e29f87c1c1), LL(0x5153a202a6f15353),
    LL(0x57dcae8ba572dcdc), LL(0x2c0b582716530b0b), LL(0x4e9d9cd327019d9d), LL(0xad6c47c1d82b6c6c),
    LL(0xc43195f562a43131), LL(0xcd7487b9e8f37474), LL(0xfff6e309f115f6f6), LL(0x05460a438c4c4646),
    LL(0x8aac092645a5acac), LL(0x1e893c970fb58989), LL(0x5014a04428b41414), LL(0xa3e15b42dfbae1e1),
    LL(0x5816b04e2ca61616), LL(0xe83acdd274f73a3a), LL(0xb9696fd0d2066969), LL(0x2409482d12410909),
    LL(0xdd70a7ade0d77070), LL(0xe2b6d954716fb6b6), LL(0x67d0ceb7bd1ed0d0), LL(0x93ed3b7ec7d6eded),
    LL(0x17cc2edb85e2cccc), LL(0x15422a5784684242), LL(0x5a98b4c22d2c9898), LL(0xaaa4490e55eda4a4),
    LL(0xa0285d8850752828), LL(0x6d5cda31b8865c5c), LL(0xc7f8933fed6bf8f8), LL(0x228644a411c28686),
};

__constant__ static const u64 C7[256] = {
    LL(0x186018c07830d818), LL(0x238c2305af462623), LL(0xc63fc67ef991b8c6), LL(0xe887e8136fcdfbe8),
    LL(0x8726874ca113cb87), LL(0xb8dab8a9626d11b8), LL(0x0104010805020901), LL(0x4f214f426e9e0d4f),
    LL(0x36d836adee6c9b36), LL(0xa6a2a6590451ffa6), LL(0xd26fd2debdb90cd2), LL(0xf5f3f5fb06f70ef5),
    LL(0x79f979ef80f29679), LL(0x6fa16f5fcede306f), LL(0x917e91fcef3f6d91), LL(0x525552aa07a4f852),
    LL(0x609d6027fdc04760), LL(0xbccabc89766535bc), LL(0x9b569baccd2b379b), LL(0x8e028e048c018a8e),
    LL(0xa3b6a371155bd2a3), LL(0x0c300c603c186c0c), LL(0x7bf17bff8af6847b), LL(0x35d435b5e16a8035),
    LL(0x1d741de8693af51d), LL(0xe0a7e05347ddb3e0), LL(0xd77bd7f6acb321d7), LL(0xc22fc25eed999cc2),
    LL(0x2eb82e6d965c432e), LL(0x4b314b627a96294b), LL(0xfedffea321e15dfe), LL(0x5741578216aed557),
    LL(0x155415a8412abd15), LL(0x77c1779fb6eee877), LL(0x37dc37a5eb6e9237), LL(0xe5b3e57b56d79ee5),
    LL(0x9f469f8cd923139f), LL(0xf0e7f0d317fd23f0), LL(0x4a354a6a7f94204a), LL(0xda4fda9e95a944da),
    LL(0x587d58fa25b0a258), LL(0xc903c906ca8fcfc9), LL(0x29a429558d527c29), LL(0x0a280a5022145a0a),
    LL(0xb1feb1e14f7f50b1), LL(0xa0baa0691a5dc9a0), LL(0x6bb16b7fdad6146b), LL(0x852e855cab17d985),
    LL(0xbdcebd8173673cbd), LL(0x5d695dd234ba8f5d), LL(0x1040108050209010), LL(0xf4f7f4f303f507f4),
    LL(0xcb0bcb16c08bddcb), LL(0x3ef83eedc67cd33e), LL(0x05140528110a2d05), LL(0x6781671fe6ce7867),
    LL(0xe4b7e47353d597e4), LL(0x279c2725bb4e0227), LL(0x4119413258827341), LL(0x8b168b2c9d0ba78b),
    LL(0xa7a6a7510153f6a7), LL(0x7de97dcf94fab27d), LL(0x956e95dcfb374995), LL(0xd847d88e9fad56d8),
    LL(0xfbcbfb8b30eb70fb), LL(0xee9fee2371c1cdee), LL(0x7ced7cc791f8bb7c), LL(0x66856617e3cc7166),
    LL(0xdd53dda68ea77bdd), LL(0x175c17b84b2eaf17), LL(0x47014702468e4547), LL(0x9e429e84dc211a9e),
    LL(0xca0fca1ec589d4ca), LL(0x2db42d75995a582d), LL(0xbfc6bf9179632ebf), LL(0x071c07381b0e3f07),
    LL(0xad8ead012347acad), LL(0x5a755aea2fb4b05a), LL(0x8336836cb51bef83), LL(0x33cc3385ff66b633),
    LL(0x6391633ff2c65c63), LL(0x020802100a041202), LL(0xaa92aa39384993aa), LL(0x71d971afa8e2de71),
    LL(0xc807c80ecf8dc6c8), LL(0x196419c87d32d119), LL(0x4939497270923b49), LL(0xd943d9869aaf5fd9),
    LL(0xf2eff2c31df931f2), LL(0xe3abe34b48dba8e3), LL(0x5b715be22ab6b95b), LL(0x881a8834920dbc88),
    LL(0x9a529aa4c8293e9a), LL(0x2698262dbe4c0b26), LL(0x32c8328dfa64bf32), LL(0xb0fab0e94a7d59b0),
    LL(0xe983e91b6acff2e9), LL(0x0f3c0f78331e770f), LL(0xd573d5e6a6b733d5), LL(0x803a8074ba1df480),
    LL(0xbec2be997c6127be), LL(0xcd13cd26de87ebcd), LL(0x34d034bde4688934), LL(0x483d487a75903248),
    LL(0xffdbffab24e354ff), LL(0x7af57af78ff48d7a), LL(0x907a90f4ea3d6490), LL(0x5f615fc23ebe9d5f),
    LL(0x2080201da0403d20), LL(0x68bd6867d5d00f68), LL(0x1a681ad07234ca1a), LL(0xae82ae192c41b7ae),
    LL(0xb4eab4c95e757db4), LL(0x544d549a19a8ce54), LL(0x937693ece53b7f93), LL(0x2288220daa442f22),
    LL(0x648d6407e9c86364), LL(0xf1e3f1db12ff2af1), LL(0x73d173bfa2e6cc73), LL(0x124812905a248212),
    LL(0x401d403a5d807a40), LL(0x0820084028104808), LL(0xc32bc356e89b95c3), LL(0xec97ec337bc5dfec),
    LL(0xdb4bdb9690ab4ddb), LL(0xa1bea1611f5fc0a1), LL(0x8d0e8d1c8307918d), LL(0x3df43df5c97ac83d),
    LL(0x976697ccf1335b97), LL(0x0000000000000000), LL(0xcf1bcf36d483f9cf), LL(0x2bac2b4587566e2b),
    LL(0x76c57697b3ece176), LL(0x82328264b019e682), LL(0xd67fd6fea9b128d6), LL(0x1b6c1bd87736c31b),
    LL(0xb5eeb5c15b7774b5), LL(0xaf86af112943beaf), LL(0x6ab56a77dfd41d6a), LL(0x505d50ba0da0ea50),
    LL(0x450945124c8a5745), LL(0xf3ebf3cb18fb38f3), LL(0x30c0309df060ad30), LL(0xef9bef2b74c3c4ef),
    LL(0x3ffc3fe5c37eda3f), LL(0x554955921caac755), LL(0xa2b2a2791059dba2), LL(0xea8fea0365c9e9ea),
    LL(0x6589650fecca6a65), LL(0xbad2bab9686903ba), LL(0x2fbc2f65935e4a2f), LL(0xc027c04ee79d8ec0),
    LL(0xde5fdebe81a160de), LL(0x1c701ce06c38fc1c), LL(0xfdd3fdbb2ee746fd), LL(0x4d294d52649a1f4d),
    LL(0x927292e4e0397692), LL(0x75c9758fbceafa75), LL(0x061806301e0c3606), LL(0x8a128a249809ae8a),
    LL(0xb2f2b2f940794bb2), LL(0xe6bfe66359d185e6), LL(0x0e380e70361c7e0e), LL(0x1f7c1ff8633ee71f),
    LL(0x62956237f7c45562), LL(0xd477d4eea3b53ad4), LL(0xa89aa829324d81a8), LL(0x966296c4f4315296),
    LL(0xf9c3f99b3aef62f9), LL(0xc533c566f697a3c5), LL(0x25942535b14a1025), LL(0x597959f220b2ab59),
    LL(0x842a8454ae15d084), LL(0x72d572b7a7e4c572), LL(0x39e439d5dd72ec39), LL(0x4c2d4c5a6198164c),
    LL(0x5e655eca3bbc945e), LL(0x78fd78e785f09f78), LL(0x38e038ddd870e538), LL(0x8c0a8c148605988c),
    LL(0xd163d1c6b2bf17d1), LL(0xa5aea5410b57e4a5), LL(0xe2afe2434dd9a1e2), LL(0x6199612ff8c24e61),
    LL(0xb3f6b3f1457b42b3), LL(0x21842115a5423421), LL(0x9c4a9c94d625089c), LL(0x1e781ef0663cee1e),
    LL(0x4311432252866143), LL(0xc73bc776fc93b1c7), LL(0xfcd7fcb32be54ffc), LL(0x0410042014082404),
    LL(0x515951b208a2e351), LL(0x995e99bcc72f2599), LL(0x6da96d4fc4da226d), LL(0x0d340d68391a650d),
    LL(0xfacffa8335e979fa), LL(0xdf5bdfb684a369df), LL(0x7ee57ed79bfca97e), LL(0x2490243db4481924),
    LL(0x3bec3bc5d776fe3b), LL(0xab96ab313d4b9aab), LL(0xce1fce3ed181f0ce), LL(0x1144118855229911),
    LL(0x8f068f0c8903838f), LL(0x4e254e4a6b9c044e), LL(0xb7e6b7d1517366b7), LL(0xeb8beb0b60cbe0eb),
    LL(0x3cf03cfdcc78c13c), LL(0x813e817cbf1ffd81), LL(0x946a94d4fe354094), LL(0xf7fbf7eb0cf31cf7),
    LL(0xb9deb9a1676f18b9), LL(0x134c13985f268b13), LL(0x2cb02c7d9c58512c), LL(0xd36bd3d6b8bb05d3),
    LL(0xe7bbe76b5cd38ce7), LL(0x6ea56e57cbdc396e), LL(0xc437c46ef395aac4), LL(0x030c03180f061b03),
    LL(0x5645568a13acdc56), LL(0x440d441a49885e44), LL(0x7fe17fdf9efea07f), LL(0xa99ea921374f88a9),
    LL(0x2aa82a4d8254672a), LL(0xbbd6bbb16d6b0abb), LL(0xc123c146e29f87c1), LL(0x535153a202a6f153),
    LL(0xdc57dcae8ba572dc), LL(0x0b2c0b582716530b), LL(0x9d4e9d9cd327019d), LL(0x6cad6c47c1d82b6c),
    LL(0x31c43195f562a431), LL(0x74cd7487b9e8f374), LL(0xf6fff6e309f115f6), LL(0x4605460a438c4c46),
    LL(0xac8aac092645a5ac), LL(0x891e893c970fb589), LL(0x145014a04428b414), LL(0xe1a3e15b42dfbae1),
    LL(0x165816b04e2ca616), LL(0x3ae83acdd274f73a), LL(0x69b9696fd0d20669), LL(0x092409482d124109),
    LL(0x70dd70a7ade0d770), LL(0xb6e2b6d954716fb6), LL(0xd067d0ceb7bd1ed0), LL(0xed93ed3b7ec7d6ed),
    LL(0xcc17cc2edb85e2cc), LL(0x4215422a57846842), LL(0x985a98b4c22d2c98), LL(0xa4aaa4490e55eda4),
    LL(0x28a0285d88507528), LL(0x5c6d5cda31b8865c), LL(0xf8c7f8933fed6bf8), LL(0x86228644a411c286),
};

__constant__ static const u64 rc[R + 1] = {
    LL(0x0000000000000000),
    LL(0x1823c6e887b8014f),
    LL(0x36a6d2f5796f9152),
    LL(0x60bc9b8ea30c7b35),
    LL(0x1de0d7c22e4bfe57),
    LL(0x157737e59ff04ada),
    LL(0x58c9290ab1a06b85),
    LL(0xbd5d10f4cb3e0567),
    LL(0xe427418ba77d95d8),
    LL(0xfbee7c66dd17479e),
    LL(0xca2dbf07ad5a8333),
};

/**
 * The core Whirlpool transform.
 */
__device__ static void processBuffer(struct NESSIEstruct * const structpointer) {
    int i, r;
    u64 K[8];        /* the round key */
    u64 block[8];    /* mu(buffer) */
    u64 state[8];    /* the cipher state */
    u64 L[8];
    u8 *buffer = structpointer->buffer;
    /*
     * map the buffer to a block:
     */
    for (i = 0; i < 8; i++, buffer += 8) {
        block[i] =
            (((u64)buffer[0]        ) << 56) ^
            (((u64)buffer[1] & 0xffL) << 48) ^
            (((u64)buffer[2] & 0xffL) << 40) ^
            (((u64)buffer[3] & 0xffL) << 32) ^
            (((u64)buffer[4] & 0xffL) << 24) ^
            (((u64)buffer[5] & 0xffL) << 16) ^
            (((u64)buffer[6] & 0xffL) <<  8) ^
            (((u64)buffer[7] & 0xffL)      );
    }
    /*
     * compute and apply K^0 to the cipher state:
     */
    state[0] = block[0] ^ (K[0] = structpointer->hash[0]);
    state[1] = block[1] ^ (K[1] = structpointer->hash[1]);
    state[2] = block[2] ^ (K[2] = structpointer->hash[2]);
    state[3] = block[3] ^ (K[3] = structpointer->hash[3]);
    state[4] = block[4] ^ (K[4] = structpointer->hash[4]);
    state[5] = block[5] ^ (K[5] = structpointer->hash[5]);
    state[6] = block[6] ^ (K[6] = structpointer->hash[6]);
    state[7] = block[7] ^ (K[7] = structpointer->hash[7]);
    /*
     * iterate over all rounds:
     */
    for (r = 1; r <= R; r++) {
        /*
         * compute K^r from K^{r-1}:
         */
        L[0] =
            C0[(int)(K[0] >> 56)       ] ^
            C1[(int)(K[7] >> 48) & 0xff] ^
            C2[(int)(K[6] >> 40) & 0xff] ^
            C3[(int)(K[5] >> 32) & 0xff] ^
            C4[(int)(K[4] >> 24) & 0xff] ^
            C5[(int)(K[3] >> 16) & 0xff] ^
            C6[(int)(K[2] >>  8) & 0xff] ^
            C7[(int)(K[1]      ) & 0xff] ^
            rc[r];
        L[1] =
            C0[(int)(K[1] >> 56)       ] ^
            C1[(int)(K[0] >> 48) & 0xff] ^
            C2[(int)(K[7] >> 40) & 0xff] ^
            C3[(int)(K[6] >> 32) & 0xff] ^
            C4[(int)(K[5] >> 24) & 0xff] ^
            C5[(int)(K[4] >> 16) & 0xff] ^
            C6[(int)(K[3] >>  8) & 0xff] ^
            C7[(int)(K[2]      ) & 0xff];
        L[2] =
            C0[(int)(K[2] >> 56)       ] ^
            C1[(int)(K[1] >> 48) & 0xff] ^
            C2[(int)(K[0] >> 40) & 0xff] ^
            C3[(int)(K[7] >> 32) & 0xff] ^
            C4[(int)(K[6] >> 24) & 0xff] ^
            C5[(int)(K[5] >> 16) & 0xff] ^
            C6[(int)(K[4] >>  8) & 0xff] ^
            C7[(int)(K[3]      ) & 0xff];
        L[3] =
            C0[(int)(K[3] >> 56)       ] ^
            C1[(int)(K[2] >> 48) & 0xff] ^
            C2[(int)(K[1] >> 40) & 0xff] ^
            C3[(int)(K[0] >> 32) & 0xff] ^
            C4[(int)(K[7] >> 24) & 0xff] ^
            C5[(int)(K[6] >> 16) & 0xff] ^
            C6[(int)(K[5] >>  8) & 0xff] ^
            C7[(int)(K[4]      ) & 0xff];
        L[4] =
            C0[(int)(K[4] >> 56)       ] ^
            C1[(int)(K[3] >> 48) & 0xff] ^
            C2[(int)(K[2] >> 40) & 0xff] ^
            C3[(int)(K[1] >> 32) & 0xff] ^
            C4[(int)(K[0] >> 24) & 0xff] ^
            C5[(int)(K[7] >> 16) & 0xff] ^
            C6[(int)(K[6] >>  8) & 0xff] ^
            C7[(int)(K[5]      ) & 0xff];
        L[5] =
            C0[(int)(K[5] >> 56)       ] ^
            C1[(int)(K[4] >> 48) & 0xff] ^
            C2[(int)(K[3] >> 40) & 0xff] ^
            C3[(int)(K[2] >> 32) & 0xff] ^
            C4[(int)(K[1] >> 24) & 0xff] ^
            C5[(int)(K[0] >> 16) & 0xff] ^
            C6[(int)(K[7] >>  8) & 0xff] ^
            C7[(int)(K[6]      ) & 0xff];
        L[6] =
            C0[(int)(K[6] >> 56)       ] ^
            C1[(int)(K[5] >> 48) & 0xff] ^
            C2[(int)(K[4] >> 40) & 0xff] ^
            C3[(int)(K[3] >> 32) & 0xff] ^
            C4[(int)(K[2] >> 24) & 0xff] ^
            C5[(int)(K[1] >> 16) & 0xff] ^
            C6[(int)(K[0] >>  8) & 0xff] ^
            C7[(int)(K[7]      ) & 0xff];
        L[7] =
            C0[(int)(K[7] >> 56)       ] ^
            C1[(int)(K[6] >> 48) & 0xff] ^
            C2[(int)(K[5] >> 40) & 0xff] ^
            C3[(int)(K[4] >> 32) & 0xff] ^
            C4[(int)(K[3] >> 24) & 0xff] ^
            C5[(int)(K[2] >> 16) & 0xff] ^
            C6[(int)(K[1] >>  8) & 0xff] ^
            C7[(int)(K[0]      ) & 0xff];
        K[0] = L[0];
        K[1] = L[1];
        K[2] = L[2];
        K[3] = L[3];
        K[4] = L[4];
        K[5] = L[5];
        K[6] = L[6];
        K[7] = L[7];
        /*
         * apply the r-th round transformation:
         */
        L[0] =
            C0[(int)(state[0] >> 56)       ] ^
            C1[(int)(state[7] >> 48) & 0xff] ^
            C2[(int)(state[6] >> 40) & 0xff] ^
            C3[(int)(state[5] >> 32) & 0xff] ^
            C4[(int)(state[4] >> 24) & 0xff] ^
            C5[(int)(state[3] >> 16) & 0xff] ^
            C6[(int)(state[2] >>  8) & 0xff] ^
            C7[(int)(state[1]      ) & 0xff] ^
            K[0];
        L[1] =
            C0[(int)(state[1] >> 56)       ] ^
            C1[(int)(state[0] >> 48) & 0xff] ^
            C2[(int)(state[7] >> 40) & 0xff] ^
            C3[(int)(state[6] >> 32) & 0xff] ^
            C4[(int)(state[5] >> 24) & 0xff] ^
            C5[(int)(state[4] >> 16) & 0xff] ^
            C6[(int)(state[3] >>  8) & 0xff] ^
            C7[(int)(state[2]      ) & 0xff] ^
            K[1];
        L[2] =
            C0[(int)(state[2] >> 56)       ] ^
            C1[(int)(state[1] >> 48) & 0xff] ^
            C2[(int)(state[0] >> 40) & 0xff] ^
            C3[(int)(state[7] >> 32) & 0xff] ^
            C4[(int)(state[6] >> 24) & 0xff] ^
            C5[(int)(state[5] >> 16) & 0xff] ^
            C6[(int)(state[4] >>  8) & 0xff] ^
            C7[(int)(state[3]      ) & 0xff] ^
            K[2];
        L[3] =
            C0[(int)(state[3] >> 56)       ] ^
            C1[(int)(state[2] >> 48) & 0xff] ^
            C2[(int)(state[1] >> 40) & 0xff] ^
            C3[(int)(state[0] >> 32) & 0xff] ^
            C4[(int)(state[7] >> 24) & 0xff] ^
            C5[(int)(state[6] >> 16) & 0xff] ^
            C6[(int)(state[5] >>  8) & 0xff] ^
            C7[(int)(state[4]      ) & 0xff] ^
            K[3];
        L[4] =
            C0[(int)(state[4] >> 56)       ] ^
            C1[(int)(state[3] >> 48) & 0xff] ^
            C2[(int)(state[2] >> 40) & 0xff] ^
            C3[(int)(state[1] >> 32) & 0xff] ^
            C4[(int)(state[0] >> 24) & 0xff] ^
            C5[(int)(state[7] >> 16) & 0xff] ^
            C6[(int)(state[6] >>  8) & 0xff] ^
            C7[(int)(state[5]      ) & 0xff] ^
            K[4];
        L[5] =
            C0[(int)(state[5] >> 56)       ] ^
            C1[(int)(state[4] >> 48) & 0xff] ^
            C2[(int)(state[3] >> 40) & 0xff] ^
            C3[(int)(state[2] >> 32) & 0xff] ^
            C4[(int)(state[1] >> 24) & 0xff] ^
            C5[(int)(state[0] >> 16) & 0xff] ^
            C6[(int)(state[7] >>  8) & 0xff] ^
            C7[(int)(state[6]      ) & 0xff] ^
            K[5];
        L[6] =
            C0[(int)(state[6] >> 56)       ] ^
            C1[(int)(state[5] >> 48) & 0xff] ^
            C2[(int)(state[4] >> 40) & 0xff] ^
            C3[(int)(state[3] >> 32) & 0xff] ^
            C4[(int)(state[2] >> 24) & 0xff] ^
            C5[(int)(state[1] >> 16) & 0xff] ^
            C6[(int)(state[0] >>  8) & 0xff] ^
            C7[(int)(state[7]      ) & 0xff] ^
            K[6];
        L[7] =
            C0[(int)(state[7] >> 56)       ] ^
            C1[(int)(state[6] >> 48) & 0xff] ^
            C2[(int)(state[5] >> 40) & 0xff] ^
            C3[(int)(state[4] >> 32) & 0xff] ^
            C4[(int)(state[3] >> 24) & 0xff] ^
            C5[(int)(state[2] >> 16) & 0xff] ^
            C6[(int)(state[1] >>  8) & 0xff] ^
            C7[(int)(state[0]      ) & 0xff] ^
            K[7];
        state[0] = L[0];
        state[1] = L[1];
        state[2] = L[2];
        state[3] = L[3];
        state[4] = L[4];
        state[5] = L[5];
        state[6] = L[6];
        state[7] = L[7];
    }
    /*
     * apply the Miyaguchi-Preneel compression function:
     */
    structpointer->hash[0] ^= state[0] ^ block[0];
    structpointer->hash[1] ^= state[1] ^ block[1];
    structpointer->hash[2] ^= state[2] ^ block[2];
    structpointer->hash[3] ^= state[3] ^ block[3];
    structpointer->hash[4] ^= state[4] ^ block[4];
    structpointer->hash[5] ^= state[5] ^ block[5];
    structpointer->hash[6] ^= state[6] ^ block[6];
    structpointer->hash[7] ^= state[7] ^ block[7];
}

/**
 * Initialize the hashing state.
 */
__device__ void WHIRLPOOL_init(struct NESSIEstruct * const structpointer) {
    int i;

    memset(structpointer->bitLength, 0, 32);
    structpointer->bufferBits = structpointer->bufferPos = 0;
    structpointer->buffer[0] = 0; /* it's only necessary to cleanup buffer[bufferPos] */
    for (i = 0; i < 8; i++) {
        structpointer->hash[i] = 0L; /* initial value */
    }
}

/**
 * Delivers input data to the hashing algorithm.
 *
 * @param    source        plaintext data to hash.
 * @param    sourceBits    how many bits of plaintext to process.
 *
 * This method maintains the invariant: bufferBits < DIGESTBITS
 */
__device__ void WHIRLPOOL_add(const unsigned char * const source,
               unsigned __int32 sourceBits,
               struct NESSIEstruct * const structpointer) {
    /*
                       sourcePos
                       |
                       +-------+-------+-------
                          ||||||||||||||||||||| source
                       +-------+-------+-------
    +-------+-------+-------+-------+-------+-------
    ||||||||||||||||||||||                           buffer
    +-------+-------+-------+-------+-------+-------
                    |
                    bufferPos
    */
    int sourcePos    = 0; /* index of leftmost source u8 containing data (1 to 8 bits). */
    int sourceGap    = (8 - ((int)sourceBits & 7)) & 7; /* space on source[sourcePos]. */
    int bufferRem    = structpointer->bufferBits & 7; /* occupied bits on buffer[bufferPos]. */
    int i;
    u32 b, carry;
    u8 *buffer       = structpointer->buffer;
    u8 *bitLength    = structpointer->bitLength;
    int bufferBits   = structpointer->bufferBits;
    int bufferPos    = structpointer->bufferPos;

    /*
     * tally the length of the added data:
     */
    u64 value = sourceBits;
    for (i = 31, carry = 0; i >= 0 && (carry != 0 || value != LL(0)); i--) {
        carry += bitLength[i] + ((u32)value & 0xff);
        bitLength[i] = (u8)carry;
        carry >>= 8;
        value >>= 8;
    }
    /*
     * process data in chunks of 8 bits (a more efficient approach would be to take whole-word chunks):
     */
    while (sourceBits > 8) {
        /* N.B. at least source[sourcePos] and source[sourcePos+1] contain data. */
        /*
         * take a byte from the source:
         */
        b = ((source[sourcePos] << sourceGap) & 0xff) |
            ((source[sourcePos + 1] & 0xff) >> (8 - sourceGap));
        /*
         * process this byte:
         */
        buffer[bufferPos++] |= (u8)(b >> bufferRem);
        bufferBits += 8 - bufferRem; /* bufferBits = 8*bufferPos; */
        if (bufferBits == DIGESTBITS) {
            /*
             * process data block:
             */
            processBuffer(structpointer);
            /*
             * reset buffer:
             */
            bufferBits = bufferPos = 0;
        }
        buffer[bufferPos] = (u8) (b << (8 - bufferRem));
        bufferBits += bufferRem;
        /*
         * proceed to remaining data:
         */
        sourceBits -= 8;
        sourcePos++;
    }
    /* now 0 <= sourceBits <= 8;
     * furthermore, all data (if any is left) is in source[sourcePos].
     */
    if (sourceBits > 0) {
        b = (source[sourcePos] << sourceGap) & 0xff; /* bits are left-justified on b. */
        /*
         * process the remaining bits:
         */
        buffer[bufferPos] |= b >> bufferRem;
    } else {
        b = 0;
    }
    if (bufferRem + sourceBits < 8) {
        /*
         * all remaining data fits on buffer[bufferPos],
         * and there still remains some space.
         */
        bufferBits += sourceBits;
    } else {
        /*
         * buffer[bufferPos] is full:
         */
        bufferPos++;
        bufferBits += 8 - bufferRem; /* bufferBits = 8*bufferPos; */
        sourceBits -= 8 - bufferRem;
        /* now 0 <= sourceBits < 8;
         * furthermore, all data (if any is left) is in source[sourcePos].
         */
        if (bufferBits == DIGESTBITS) {
            /*
             * process data block:
             */
            processBuffer(structpointer);
            /*
             * reset buffer:
             */
            bufferBits = bufferPos = 0;
        }
        buffer[bufferPos] = (u8) (b << (8 - bufferRem));
        bufferBits += (int)sourceBits;
    }
    structpointer->bufferBits   = bufferBits;
    structpointer->bufferPos    = bufferPos;
}

/**
 * Get the hash value from the hashing state.
 * 
 * This method uses the invariant: bufferBits < DIGESTBITS
 */
__device__ void WHIRLPOOL_finalize(struct NESSIEstruct * const structpointer,
                    unsigned char * const result) {
    int i;
    u8 *buffer      = structpointer->buffer;
    u8 *bitLength   = structpointer->bitLength;
    int bufferBits  = structpointer->bufferBits;
    int bufferPos   = structpointer->bufferPos;
    u8 *digest      = result;

    /*
     * append a '1'-bit:
     */
    buffer[bufferPos] |= 0x80U >> (bufferBits & 7);
    bufferPos++; /* all remaining bits on the current u8 are set to zero. */
    /*
     * pad with zero bits to complete (N*WBLOCKBITS - LENGTHBITS) bits:
     */
    if (bufferPos > WBLOCKBYTES - LENGTHBYTES) {
        if (bufferPos < WBLOCKBYTES) {
            memset(&buffer[bufferPos], 0, WBLOCKBYTES - bufferPos);
        }
        /*
         * process data block:
         */
        processBuffer(structpointer);
        /*
         * reset buffer:
         */
        bufferPos = 0;
    }
    if (bufferPos < WBLOCKBYTES - LENGTHBYTES) {
        memset(&buffer[bufferPos], 0, (WBLOCKBYTES - LENGTHBYTES) - bufferPos);
    }
    bufferPos = WBLOCKBYTES - LENGTHBYTES;
    /*
     * append bit length of hashed data:
     */
    memcpy(&buffer[WBLOCKBYTES - LENGTHBYTES], bitLength, LENGTHBYTES);
    /*
     * process data block:
     */
    processBuffer(structpointer);
    /*
     * return the completed message digest:
     */
    for (i = 0; i < DIGESTBYTES/8; i++) {
        digest[0] = (u8)(structpointer->hash[i] >> 56);
        digest[1] = (u8)(structpointer->hash[i] >> 48);
        digest[2] = (u8)(structpointer->hash[i] >> 40);
        digest[3] = (u8)(structpointer->hash[i] >> 32);
        digest[4] = (u8)(structpointer->hash[i] >> 24);
        digest[5] = (u8)(structpointer->hash[i] >> 16);
        digest[6] = (u8)(structpointer->hash[i] >>  8);
        digest[7] = (u8)(structpointer->hash[i]      );
        digest += 8;
    }
    structpointer->bufferBits   = bufferBits;
    structpointer->bufferPos    = bufferPos;
}
